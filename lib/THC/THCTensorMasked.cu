#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"

#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

template <typename T, typename MaskT>
struct TensorMaskedFillOp {
  TensorMaskedFillOp(T v) : value(v) {}
  __device__ inline void operator()(T* t, MaskT* mask) {
    if (*mask) {
      *t = value;
    }
  }

  T value;
};

template <typename T, typename MaskT, typename MaskPrefixSumT>
struct TensorMaskedCopyOp {
  TensorMaskedCopyOp(T* s) : in(s) {}

  __device__ inline void operator()(T* out,
                                    MaskT* mask,
                                    MaskPrefixSumT* maskPrefixSum) {
    if (*mask) {
      *out = in[*maskPrefixSum];
    }
  }

  // Where we are copying from
  T* in;
};

template <typename T, typename MaskT, typename MaskPrefixSumT>
struct TensorMaskedSelectOp {
  TensorMaskedSelectOp(T* t) : out(t) {}
  __device__ inline void operator()(MaskT* mask,
                                    MaskPrefixSumT* maskPrefixSum,
                                    T* in) {
    if (*mask) {
      out[*maskPrefixSum] = *in;
    }
  }

  T* out;
};

#include "generic/THCTensorMasked.cu"
#include "THCGenerateAllTypes.h"
