#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCApply.cuh"
#include "THCNumerics.cuh"
#include "THCReduce.cuh"

template <typename T, typename TOut>
struct TensorLTOp {
  __device__ inline void operator()(TOut* out, T* a, T* b) {
    *out = ScalarConvert<bool, TOut>::to(THCNumerics<T>::lt(*a, *b));
  }
};

template <typename T, typename TOut>
struct TensorGTOp {
  __device__ inline void operator()(TOut* out, T* a, T* b) {
    *out = ScalarConvert<bool, TOut>::to(THCNumerics<T>::gt(*a, *b));
  }
};

template <typename T, typename TOut>
struct TensorLEOp {
  __device__ inline void operator()(TOut* out, T* a, T* b) {
    *out = ScalarConvert<bool, TOut>::to(THCNumerics<T>::le(*a, *b));
  }
};

template <typename T, typename TOut>
struct TensorGEOp {
  __device__ inline void operator()(TOut* out, T* a, T* b) {
    *out = ScalarConvert<bool, TOut>::to(THCNumerics<T>::ge(*a, *b));
  }
};

template <typename T, typename TOut>
struct TensorEQOp {
  __device__ inline void operator()(TOut* out, T* a, T* b) {
    *out = ScalarConvert<bool, TOut>::to(THCNumerics<T>::eq(*a, *b));
  }
};

template <typename T, typename TOut>
struct TensorNEOp {
  __device__ inline void operator()(TOut* out, T* a, T* b) {
    *out = ScalarConvert<bool, TOut>::to(THCNumerics<T>::ne(*a, *b));
  }
};

template<typename TensorType, typename TensorTypeOut, typename Op>
void THC_logicalTensor(THCState *state,
                       TensorTypeOut *self_,
                       TensorType *src1,
                       TensorType *src2,
                       Op op) {
  THLongStorage* st = TensorUtils<TensorType>::newSizeOf(state, src1);
  TensorUtils<TensorTypeOut>::resize(state, self_, st, NULL);
  THLongStorage_free(st);

  THArgCheck(TensorUtils<TensorType>::getNumElements(state, src1) ==
             TensorUtils<TensorType>::getNumElements(state, src2), 3,
             "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, op)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

#include "generic/THCTensorMathCompareT.cu"
#include "THCGenerateAllTypes.h"
