#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCTensorRandom.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

/* Perform an inclusive scan along an outer dimension of a tensor.
 *
 * - num_orows is the size of the flattened outer dimensions;
 * - num_irows is the size of the flattened inner dimensions;
 * - row_size is the size of the dimension along which to compute the variance;
 *
 * The dimensions to the outside and inside of the specified dimension are considered as flattened.
 * Thread blocks with the same blockIdx.y process an "outer row" (i.e. an element of the flattened
 * outer dimensions, which contains several "inner rows").
 * Each thread processes a single inner row at a time.
 */
template<class BinaryOp>
__global__ void THCudaTensor_kernel_scanOuterDim(float *tgt_, float *src_,
                                                 unsigned num_orows, unsigned num_irows, unsigned row_size,
                                                 float init, BinaryOp binary_op)
{
  for (unsigned orow = blockIdx.x; orow < num_orows; orow += gridDim.x) {
    for (unsigned irow = blockIdx.y * blockDim.x + threadIdx.x; irow < num_irows; irow += gridDim.y * blockDim.x) {
      float *src = src_ + orow * row_size * num_irows + irow;
      float *tgt = tgt_ + orow * row_size * num_irows + irow;
      float acc = init;

      for (unsigned col = 0; col < row_size; ++col) {
        acc = binary_op(acc, *src);
        *tgt = acc;

        src += num_irows;
        tgt += num_irows;
      }
    }
  }
}

template<class BinaryOp>
__host__ void THCudaTensor_scanOuterDim(THCState *state, THCudaTensor *tgt, THCudaTensor *src, long dimension,
                                        float init, BinaryOp binary_op)
{
  unsigned ndim = THCudaTensor_nDimension(state, src);
  // Treat all outer dimensions (i.e. dim < dimension) as one.
  unsigned num_orows = 1;
  for (unsigned dim = 0; dim < dimension; dim++) {
    num_orows *= THCudaTensor_size(state, src, dim);
  }
  unsigned row_size = THCudaTensor_size(state, src, dimension);
  // Treat all inner dimensions (i.e. dim > dimension) as one.
  unsigned num_irows = 1;
  for (unsigned dim = dimension + 1; dim < ndim; dim++) {
    num_irows *= THCudaTensor_size(state, src, dim);
  }

  dim3 threads(min(512, num_irows));
  unsigned maxGridDim = 1024;
  dim3 grid(min(maxGridDim, num_orows), min(maxGridDim, THCCeilDiv(num_irows, threads.x)));

  THCudaTensor_kernel_scanOuterDim<<<grid, threads, 0, THCState_getCurrentStream(state)>>>(
      THCudaTensor_data(state, tgt), THCudaTensor_data(state, src), num_orows, num_irows, row_size, init, binary_op);
  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
}


/* Perform an inclusive scan along the innermost dimension of a tensor.
 *
 * - num_rows is the size of the flattened outer dimensions;
 * - row_size is the size of the innermost dimension;
 *
 * The outer dimensions of the tensor are considered as a single dimension, i.e. the tensor is
 * considered as having 'num_rows' rows of size 'row_size'.
 * Each thread block processes one or more sets of contiguous rows (processing multiple rows
 * per thread block is quicker than processing a single row, especially for short rows).
 */
template<int num_threads_x, int num_threads_y, class BinaryFunction>
__global__ void THCudaTensor_kernel_scanInnermostDim(float *tgt_, float *src_,
                                                     unsigned num_rows, unsigned row_size,
                                                     float init, BinaryFunction binary_op)
{
  __shared__ float sbuf[num_threads_y][2 * num_threads_x];

  float* row_buf = sbuf[threadIdx.y];

  for (unsigned block_row = blockIdx.x * blockDim.y;
       block_row < num_rows;
       block_row += blockDim.y * gridDim.x) {
    unsigned row = block_row + threadIdx.y;
    float block_total = init;

    float *row_src = src_ + row * row_size;
    float *row_tgt = tgt_ + row * row_size;

    // Perform scan on one block at a time, keeping track of the total value of
    // all blocks processed so far.
    for (unsigned block_col = 0; block_col < row_size; block_col += 2 * num_threads_x) {
      // Load data into shared memory (two values per thread).
      unsigned col1 = block_col + threadIdx.x;
      unsigned col2 = block_col + num_threads_x + threadIdx.x;
      if (row < num_rows) {
        if (col1 < row_size) {
          row_buf[threadIdx.x] = row_src[col1];
        } else {
          row_buf[threadIdx.x] = init;
        }

        if (col2 < row_size) {
          row_buf[num_threads_x + threadIdx.x] = row_src[col2];
        } else {
          row_buf[num_threads_x + threadIdx.x] = init;
        }

        // Add the total value of all previous blocks to the first value of this block.
        if (threadIdx.x == 0) {
          row_buf[0] = binary_op(row_buf[0], block_total);
        }
      }
      __syncthreads();

      // Parallel reduction (up-sweep).
      for (unsigned s = num_threads_x, d = 1; s >= 1; s >>= 1, d <<= 1) {
        if (row < num_rows && threadIdx.x < s) {
          unsigned offset = (2 * threadIdx.x + 1) * d - 1;
          row_buf[offset + d] = binary_op(row_buf[offset], row_buf[offset + d]);
        }
        __syncthreads();
      }

      // Down-sweep.
      for (unsigned s = 2, d = num_threads_x / 2; d >= 1; s <<= 1, d >>= 1) {
        if (row < num_rows && threadIdx.x < s - 1) {
          unsigned offset = 2 * (threadIdx.x + 1) * d - 1;
          row_buf[offset + d] = binary_op(row_buf[offset], row_buf[offset + d]);
        }
        __syncthreads();
      }

      // Write back to output.
      if (row < num_rows) {
        if (col1 < row_size) row_tgt[col1] = row_buf[threadIdx.x];
        if (col2 < row_size) row_tgt[col2] = row_buf[num_threads_x + threadIdx.x];
      }
      block_total = row_buf[2 * num_threads_x - 1];
      __syncthreads();
    }
  }
}

template<class BinaryFunction>
__host__ void THCudaTensor_scanInnermostDim(THCState *state, THCudaTensor *tgt, THCudaTensor *src, float init, BinaryFunction binary_op)
{
  unsigned ndim = THCudaTensor_nDimension(state, src);
  // Treat all outer dimensions as a single dimension.
  unsigned num_rows = 1;
  for (unsigned dim = 0; dim < ndim - 1; dim++) {
    num_rows *= THCudaTensor_size(state, src, dim);
  }
  unsigned row_size = THCudaTensor_size(state, src, ndim - 1);

  dim3 threads(16, 32);
  dim3 grid(min(1024, THCCeilDiv(num_rows, threads.y)));

  THCudaTensor_kernel_scanInnermostDim<16, 32><<<grid, threads, 0, THCState_getCurrentStream(state)>>>(
      THCudaTensor_data(state, tgt), THCudaTensor_data(state, src), num_rows, row_size, init, binary_op);
  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
}

template<class BinaryFunction>
void THCudaTensor_scanDim(THCState *state, THCudaTensor *self_, THCudaTensor *src, long dimension, float init, BinaryFunction binary_op)
{
  THCudaTensor_resizeAs(state, self_, src);

  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  src = THCudaTensor_newContiguous(state, src);

  if (dimension == THCudaTensor_nDimension(state, src) - 1) {
    THCudaTensor_scanInnermostDim(state, self, src, init, binary_op);
  } else {
    THCudaTensor_scanOuterDim(state, self, src, dimension, init, binary_op);
  }

  THCudaTensor_free(state, src);
  THCudaTensor_freeCopyTo(state, self, self_);
}

void THCudaTensor_cumsum(THCState *state, THCudaTensor *self, THCudaTensor *src, long dimension)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self, src));
  return THCudaTensor_scanDim(state, self, src, dimension, 0.0f, thrust::plus<float>());
}

void THCudaTensor_cumprod(THCState *state, THCudaTensor *self, THCudaTensor *src, long dimension)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self, src));
  return THCudaTensor_scanDim(state, self, src, dimension, 1.0f, thrust::multiplies<float>());
}
