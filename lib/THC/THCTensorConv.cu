#include "hip/hip_runtime.h"
#include "THCTensorConv.h"
#include "THCGeneral.h"
#include <stdio.h>

/*
 * Description:
 *   This code provides convolutions and xcorrelations that are API compatible with
 *   the ones in THLabConv.
 *
 * History:
 *   Sept 11, 2011, 11:59PM  -  Clement Farabet  -  Optimized RevConv by a good x2
 *   July 22, 2011, 8:38PM   -  Clement Farabet  -  All Valid/Full/XCORR/CONV implemented
 *   July 22, 2011, 4:00PM   -  Clement Farabet  -  Rewrote for loop to insure memory coalescing
 *   July 21, 2011, 11:21PM  -  Clement Farabet  -  Creation, based conv2d routine
 */

#define CUDA_SHARED_MEM_SIZE (12*1024-32) // this is given by nVidia: max shared mem per block

/*
 * Description:
 *   base conv2D routine: 3D input, 3D output, 4D kernel
 *
 *   - all chunks of data should be contiguous
 *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
 *   - the templated kernel size is useful to generate code that's 2x faster
 *     but can be set to 0 to allow arbitrary kernel sizes
 */
template <bool swapkernel, int T_kernel_h, int T_kernel_w>
  __global__ void conv2generic(float *input, float *kernel, float *output,
                               int input_n, int input_h, int input_w,
                               int kernel_n, int kernel_h, int kernel_w,
                               int stride_h, int stride_w)
{
  // output dimensions
  int output_h = (input_h - kernel_h) / stride_h + 1;
  int output_w = (input_w - kernel_w) / stride_w + 1;

  // xcorr or conv
  int koffset = swapkernel ? kernel_w*kernel_h-1 : 0;

  // nb outputs
  int output_n = kernel_n / input_n;

  // generate offsets according to block/thread ids
  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  int oo_start = blockIdx.x;
  int oo_end = oo_start+1;

  int ii_start = (blockIdx.x / output_n) * input_n;
  int ii_end = ii_start + input_n;

  // nb threads, unique thread id
  int tid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;
  int nthreads = blockDim.x * blockDim.y * blockDim.z;

  // iterators
  int oo, ii, xx, yy, kx, ky, kk;

  // do the kernels fit in shared mem ?
  if (input_n*kernel_w*kernel_h <= CUDA_SHARED_MEM_SIZE) {

    // put the kernel in shared memory
    __shared__ float shared_kernel[CUDA_SHARED_MEM_SIZE];

    // first thread of each block does the copy
    for (kk = tid; kk < kernel_w*kernel_h*input_n; kk += nthreads) {
      shared_kernel[kk] = kernel[input_n*kernel_w*kernel_h*(oo_start % output_n) + kk];
    }
    __syncthreads();

    // templated kernel size
    if ((T_kernel_w > 0) && (T_kernel_h > 0)) {
      // unrolled convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for(ii = ii_start; ii < ii_end; ii++) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              float *kernel_p = shared_kernel + (ii % input_n)*kernel_w*kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
#pragma unroll
                for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                  for(kx = 0; kx < T_kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
#pragma unroll
                for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                  for(kx = 0; kx < T_kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    } else {
      // default convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for(ii = ii_start; ii < ii_end; ii++) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              float *kernel_p = shared_kernel + (ii % input_n) * kernel_w * kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    }

  } else { // not enough shared mem for kernels, simply stream them

    // convolution loop
    for(oo = oo_start; oo < oo_end; oo++) {
      for(ii = ii_start; ii < ii_end; ii++) {
        for(yy = yy_start; yy < yy_end; yy+=yy_step) {
          for(xx = xx_start; xx < xx_end; xx+=xx_step) {
            // Dot product in two dimensions... (between input image and the mask)
            float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
            float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
            float *kernel_p = kernel + ((oo % output_n) * input_n + (ii % input_n))*kernel_w*kernel_h + koffset;
            float sum = 0;
            if (swapkernel) {
              for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                for(kx = 0; kx < kernel_w; kx++) {
                  sum += input_p[kx]*(*kernel_p--);
                }
                input_p += input_w;
              }
            } else {
              for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                for(kx = 0; kx < kernel_w; kx++) {
                  sum += input_p[kx]*(*kernel_p++);
                }
                input_p += input_w;
              }
            }
            *output_p += sum;
          }
        }
      }
    }
  }
}

/*
 * Description:
 *   base conv2D routine with reversed stride: 3D input, 4D output, 3D kernel
 *   this is useful for computing gradients with respect to kernels, where:
 *   input=input, kernel=gradOutput, output=gradWeight
 *
 *   - all chunks of data should be contiguous
 *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
 */
__global__ void conv2genericrev(float *input, float *kernel, float *output,
                                int input_n, int input_h, int input_w,
                                int kernel_n, int kernel_h, int kernel_w,
                                float alpha, int stride_h, int stride_w)
{
  // output dimensions
  int output_h = input_h - (kernel_h - 1) * stride_h;
  int output_w = input_w - (kernel_w - 1) * stride_w;

  // this thread only processes one output, defined by the block Ids
  int kk = blockIdx.x;
  int ii = blockIdx.y;

  // batch id
  int batch = threadIdx.z;

  // kernel id
  int kid = threadIdx.x;
  int nkids = blockDim.x;

  // thread ID
  int tid = kid + batch*blockDim.x;
  int nthreads = blockDim.x * blockDim.z;

  // one thread only sees one output
  output = output + (kk * input_n + ii) * output_h*output_w;

  // put the output in shared memory
  __shared__ float shared_output[CUDA_SHARED_MEM_SIZE];

  // generate tid outputs in shared memory
  float *output_s = shared_output + tid*output_w*output_h;

  // convolution loop
  int xx, yy, kx, ky;
  yy = threadIdx.y;
  float *output_p = output_s + yy * output_w;
  for(xx=0; xx<output_w; xx++) {
    // Dot product in two dimensions... (between input image and kernel)
    float *input_p = input + (ii + batch*input_n)*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
    float *kernel_p = kernel + (kk + batch*kernel_n)*kernel_w*kernel_h;
    float sum = 0;
    for(ky=0; ky<kernel_h; ky++) {
      for(kx=kid; kx<kernel_w; kx+=nkids) {
        sum += input_p[kx]*kernel_p[kx];
      }
      input_p += input_w;
      kernel_p += kernel_w;
    }
    *(output_p++) = sum;
  }
  __syncthreads();

  // reduce and write back
  if (yy == 0) {
    // reduce outputs
    for (int k=1; k<nthreads; k++) {
      for (int i=tid; i<output_w*output_h; i+=nthreads) {
        shared_output[i] += shared_output[k*output_h*output_w + i];
      }
    }
    __syncthreads();

    // add existing output, and write back
    for (int i=tid; i<output_w*output_h; i+=nthreads) {
      output[i] += alpha*shared_output[i];
    }
  }
}

// A helper macro for the common pattern of checking the input
// rows/columns for a small number of values, specializing the kernel
// template paremeters if rows/columns are equal and small, and
// otherwise just passing zero to the kernel.
#define FOR_KERNEL_SPECIALIZED_DIMENSION(ROWS, COLUMNS, KERNEL) \
  if ((ROWS) == (COLUMNS)) {                                    \
    switch ((ROWS)) {                                           \
      case 3: { KERNEL(3); break; }                             \
      case 4: { KERNEL(4); break; }                             \
      case 5: { KERNEL(5); break; }                             \
      case 6: { KERNEL(6); break; }                             \
      case 7: { KERNEL(7); break; }                             \
      case 8: { KERNEL(8); break; }                             \
      case 9: { KERNEL(9); break; }                             \
      case 10: { KERNEL(10); break; }                           \
      case 11: { KERNEL(11); break; }                           \
      case 12: { KERNEL(12); break; }                           \
      case 13: { KERNEL(13); break; }                           \
      default: { KERNEL(0); break; }                            \
    }                                                           \
  } else {                                                      \
    KERNEL(0);                                                  \
  }

/*
 * API-compatible with THRealTensor_conv2Dmv
 * 3D input, 4D kernel, 3D output
 * matrix vector product like: y <- Ax + beta*y
 */
THC_API void THCudaTensor_conv2Dmv(THCudaTensor *output, float beta, THCudaTensor *input,
                                  THCudaTensor *kernel, long srow, long scol, const char *type)
{
  long nInputPlane, nInputRows, nInputCols;
  long nKernelRows, nKernelCols;
  long nOutputPlane, nOutputRows, nOutputCols;

  THArgCheck(kernel->nDimension == 4 , 4, "kernel: 4D Tensor expected");
  THArgCheck(srow >= 1, 5, "Stride should be a positive integer");
  THArgCheck(scol >= 1, 6, "Stride should be a positive integer");
  THArgCheck(type[0] == 'v' || type[0] == 'f', 7, "type of convolution can 'v' or 'f'");
  THArgCheck(type[1] == 'c' || type[1] == 'x', 7, "type of convolution can 'x' or 'c'");

  input = THCudaTensor_newContiguous(input);
  kernel = THCudaTensor_newContiguous(kernel);

  nInputPlane = input->size[0];
  nInputRows  = input->size[1];
  nInputCols  = input->size[2];

  nKernelRows  = kernel->size[2];
  nKernelCols  = kernel->size[3];
  nOutputPlane = kernel->size[0];
  THArgCheck(kernel->size[1] == nInputPlane, 2, "invalid number of input planes");

  THArgCheck( (nInputRows >= nKernelRows && nInputCols >= nKernelCols) || *type == 'f', 2,
              "conv2Dmv : Input image is smaller than kernel");

  if (*type == 'f') {
    // output dims
    nOutputRows = (nInputRows - 1) * srow + nKernelRows;
    nOutputCols = (nInputCols - 1) * scol + nKernelCols;

    // use temp buffer
    static THCudaTensor *inputP;
    static int firstcall = 1;
    if (firstcall) {
      inputP = THCudaTensor_new();
      firstcall = 0;
    }

    // create a zero-padded input
    long nInputRowsPadded = (nOutputRows - 1) * srow + nKernelRows;
    long nInputColsPadded = (nOutputCols - 1) * scol + nKernelCols;
    THCudaTensor_resize3d(inputP, nInputPlane, nInputRowsPadded, nInputColsPadded);
    THCudaTensor_zero(inputP);

    THCudaTensor *centered = THCudaTensor_new();
    THCudaTensor_narrow(centered, inputP, 2, nKernelCols-1, nInputCols);
    THCudaTensor_narrow(centered, NULL, 1, nKernelRows-1, nInputRows);
    THCudaTensor_copy(centered, input);
    THCudaTensor_free(centered);

    // remap input to newly created tensor
    THCudaTensor_free(input);
    input = inputP;
    nInputRows = nInputRowsPadded;
    nInputCols = nInputColsPadded;

  } else { // 'v'
    // output dims
    nOutputRows = (nInputRows - nKernelRows) / srow + 1;
    nOutputCols = (nInputCols - nKernelCols) / scol + 1;
  }

  long nelem = THCudaTensor_nElement(output);
  THCudaTensor_resize3d(output, nOutputPlane, nOutputRows, nOutputCols);

  if (beta == 0 || nelem != THCudaTensor_nElement(output)) {
    THCudaTensor_zero(output);
  } else if (beta != 1) {
    THCudaTensor_mul(output, beta);
  }

  float *input_data = THCudaTensor_data(input);
  float *weight_data = THCudaTensor_data(kernel);
  float *output_data = THCudaTensor_data(output);

  // cuda blocks & threads:
  int yblocks = (int)(16L / nOutputPlane);
  yblocks = yblocks < 1 ? 1 : yblocks;
  dim3 blocks(nOutputPlane,yblocks);
  dim3 threads(32,8);

  // convolution: xcorr2 or conv2
  if (type[1] == 'x') {
#define X_CONV_KERNEL(dim)                                              \
    conv2generic <false, (dim), (dim)> <<<blocks, threads>>> (          \
        input_data, weight_data, output_data,                           \
        nInputPlane, nInputRows, nInputCols,                            \
        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,             \
        srow, scol);

    FOR_KERNEL_SPECIALIZED_DIMENSION(nKernelRows, nKernelCols, X_CONV_KERNEL);
#undef X_CONV_KERNEL
  } else { // 'c'
#define C_CONV_KERNEL(dim)                                              \
    conv2generic <true, (dim), (dim)> <<<blocks, threads>>> (           \
        input_data, weight_data, output_data,                           \
        nInputPlane, nInputRows, nInputCols,                            \
        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,             \
        srow, scol);

    FOR_KERNEL_SPECIALIZED_DIMENSION(nKernelRows, nKernelCols, C_CONV_KERNEL);
#undef C_CONV_KERNEL
  }

  // clean
  if (*type != 'f') THCudaTensor_free(input);
  THCudaTensor_free(kernel);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in conv2Dmv: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}

/*
 * API-compatible with THRealTensor_conv2Dmm
 * 4D input, 4D kernel, 4D output
 * matrix vector product like: y <- Ax + beta*y
 */
THC_API void THCudaTensor_conv2Dmm(THCudaTensor *output, float beta, THCudaTensor *input,
                                  THCudaTensor *kernel, long srow, long scol, const char *type)
{
  long nbatch, nInputPlane, nInputRows, nInputCols;
  long nKernelRows, nKernelCols;
  long nOutputPlane, nOutputRows, nOutputCols;

  THArgCheck(kernel->nDimension == 4 , 4, "kernel: 4D Tensor expected");
  THArgCheck(srow >= 1, 5, "Stride should be a positive integer");
  THArgCheck(scol >= 1, 6, "Stride should be a positive integer");
  THArgCheck(type[0] == 'v' || type[0] == 'f', 7, "type of convolution can 'v' or 'f'");
  THArgCheck(type[1] == 'c' || type[1] == 'x', 7, "type of convolution can 'x' or 'c'");

  input = THCudaTensor_newContiguous(input);
  kernel = THCudaTensor_newContiguous(kernel);

  nbatch      = input->size[0];
  nInputPlane = input->size[1];
  nInputRows  = input->size[2];
  nInputCols  = input->size[3];

  nKernelRows  = kernel->size[2];
  nKernelCols  = kernel->size[3];
  nOutputPlane = kernel->size[0];
  THArgCheck(kernel->size[1] == nInputPlane, 2, "invalid number of input planes");

  THArgCheck( (nInputRows >= nKernelRows && nInputCols >= nKernelCols) || *type == 'f', 2,
              "conv2Dmm : Input image is smaller than kernel");

  if (*type == 'f') {
    // output dims
    nOutputRows = (nInputRows - 1) * srow + nKernelRows;
    nOutputCols = (nInputCols - 1) * scol + nKernelCols;

    // use temp buffer
    static THCudaTensor *inputP;
    static int firstcall = 1;
    if (firstcall) {
      inputP = THCudaTensor_new();
      firstcall = 0;
    }

    // create a zero-padded input
    long nInputRowsPadded = (nOutputRows - 1) * srow + nKernelRows;
    long nInputColsPadded = (nOutputCols - 1) * scol + nKernelCols;
    THCudaTensor_resize4d(inputP, nbatch, nInputPlane, nInputRowsPadded, nInputColsPadded);
    THCudaTensor_zero(inputP);

    THCudaTensor *centered = THCudaTensor_new();
    THCudaTensor_narrow(centered, inputP, 3, nKernelCols-1, nInputCols);
    THCudaTensor_narrow(centered, NULL, 2, nKernelRows-1, nInputRows);
    THCudaTensor_copy(centered, input);
    THCudaTensor_free(centered);

    // remap input to newly created tensor
    THCudaTensor_free(input);
    input = inputP;
    nInputRows = nInputRowsPadded;
    nInputCols = nInputColsPadded;

  } else { // 'v'
    // output dims
    nOutputRows = (nInputRows - nKernelRows) / srow + 1;
    nOutputCols = (nInputCols - nKernelCols) / scol + 1;
  }

  long nelem = THCudaTensor_nElement(output);
  THCudaTensor_resize4d(output, nbatch, nOutputPlane, nOutputRows, nOutputCols);

  if (beta == 0 || nelem != THCudaTensor_nElement(output)) {
    THCudaTensor_zero(output);
  } else if (beta != 1) {
    THCudaTensor_mul(output, beta);
  }

  float *input_data = THCudaTensor_data(input);
  float *weight_data = THCudaTensor_data(kernel);
  float *output_data = THCudaTensor_data(output);

  // cuda blocks & threads:
  int yblocks = (int)(16L / nOutputPlane);
  yblocks = yblocks < 1 ? 1 : yblocks;
  dim3 blocks(nOutputPlane*nbatch,yblocks);
  dim3 threads(32,8);

  // convolution: xcorr2 or conv2
  if (type[1] == 'x') {
#define X_CONV_KERNEL(dim)                                              \
    conv2generic <false, (dim), (dim)> <<<blocks, threads>>> (          \
        input_data, weight_data, output_data,                           \
        nInputPlane, nInputRows, nInputCols,                            \
        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,             \
        srow, scol);

    FOR_KERNEL_SPECIALIZED_DIMENSION(nKernelCols, nKernelRows, X_CONV_KERNEL);
#undef X_CONV_KERNEL
  } else { // 'c'
#define C_CONV_KERNEL(dim)                                              \
    conv2generic <true, (dim), (dim)> <<<blocks, threads>>> (           \
        input_data, weight_data, output_data,                           \
        nInputPlane, nInputRows, nInputCols,                            \
        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,             \
        srow, scol);                                                    \

    FOR_KERNEL_SPECIALIZED_DIMENSION(nKernelCols, nKernelRows, C_CONV_KERNEL);
#undef C_CONV_KERNEL
  }

  // clean
  if (*type != 'f') THCudaTensor_free(input);
  THCudaTensor_free(kernel);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    printf("error in conv2Dmm: %s\n", hipGetErrorString(err));
    printf("requested grid size: %dx%dx%d, max allowed: %dx%dx%d\n",
           blocks.x, blocks.y, blocks.z,
           deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("requested block size: %dx%dx%d, max allowed: %dx%dx%d\n",
           threads.x, threads.y, threads.z,
           deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    THError("aborting");
  }
}

/*
 * API-compatible with THRealTensor_conv2DRevger
 * 3D input, 3D kernel, 4D output
 * like rank1 update
 * A <- xx' + beta*A
 * for sr,sc=1 this is equivalent to xcorr2Dger, but otherwise it is useful for
 * calculating derivatives wrt a kernel that is applied with stride sr,sc != 1
 */
THC_API void THCudaTensor_conv2DRevger(THCudaTensor *output, float beta, float alpha,
                                      THCudaTensor *input, THCudaTensor *kernel,
                                      long srow, long scol)
{
  long nInputPlane, nInputRows, nInputCols;
  long nKernelPlane, nKernelRows, nKernelCols;
  long nOutputRows, nOutputCols;

  THArgCheck(input->nDimension == 3 , 3, "input: 3D Tensor expected");
  THArgCheck(kernel->nDimension == 3 , 4, "kernel: 3D Tensor expected");
  THArgCheck(srow >= 1, 5, "Stride should be a positive integer");
  THArgCheck(scol >= 1, 6, "Stride should be a positive integer");

  input = THCudaTensor_newContiguous(input);
  kernel = THCudaTensor_newContiguous(kernel);

  nInputPlane = input->size[0];
  nInputRows  = input->size[1];
  nInputCols  = input->size[2];

  nKernelPlane = kernel->size[0];
  nKernelRows = kernel->size[1];
  nKernelCols = kernel->size[2];

  THArgCheck(nInputRows >= nKernelRows && nInputCols >= nKernelCols , 2,
             "conv2DRevger : Input image is smaller than kernel");

  nOutputRows = nInputRows - (nKernelRows - 1) * srow;
  nOutputCols = nInputCols - (nKernelCols - 1) * scol;

  long nelem = THCudaTensor_nElement(output);
  THCudaTensor_resize4d(output, nKernelPlane, nInputPlane, nOutputRows, nOutputCols);

  if (nelem == 0 || beta == 0 || nelem != THCudaTensor_nElement(output)) {
    THCudaTensor_zero(output);
  } else if (beta != 1) {
    THCudaTensor_mul(output, beta);
  }

  float *input_data = THCudaTensor_data(input);
  float *kernel_data = THCudaTensor_data(kernel);
  float *output_data = THCudaTensor_data(output);

  // auto compute nb of blocks and threads
  dim3 blocks(nKernelPlane, nInputPlane);
  dim3 threads(128/nOutputRows, nOutputRows);

  // compute rev conv
  conv2genericrev <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                         nInputPlane, nInputRows, nInputCols,
                                         nKernelPlane, nKernelRows, nKernelCols,
                                         alpha, srow, scol);

  // clean
  THCudaTensor_free(input);
  THCudaTensor_free(kernel);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in conv2DRevger: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}

/*
 * API-compatible with THRealTensor_conv2DRevgerm
 * 4D input, 4D kernel, 4D output
 * conv2DRevgerm is doing the same thing as conv2DRevger, but with batch inputs
 */
THC_API void THCudaTensor_conv2DRevgerm(THCudaTensor *output, float beta, float alpha,
                                       THCudaTensor *input, THCudaTensor *kernel,
                                       long srow, long scol)
{
  long nInputPlane, nInputRows, nInputCols;
  long nKernelPlane, nKernelRows, nKernelCols;
  long nOutputRows, nOutputCols;
  long nbatch;

  THArgCheck(input->nDimension == 4 , 3, "input: 3D Tensor expected");
  THArgCheck(kernel->nDimension == 4 , 4, "kernel: 3D Tensor expected");
  THArgCheck(srow >= 1, 5, "Stride should be a positive integer");
  THArgCheck(scol >= 1, 6, "Stride should be a positive integer");

  input = THCudaTensor_newContiguous(input);
  kernel = THCudaTensor_newContiguous(kernel);

  nbatch      = input->size[0];
  nInputPlane = input->size[1];
  nInputRows  = input->size[2];
  nInputCols  = input->size[3];

  nKernelPlane = kernel->size[1];
  nKernelRows = kernel->size[2];
  nKernelCols = kernel->size[3];

  THArgCheck(nInputRows >= nKernelRows && nInputCols >= nKernelCols , 2,
             "conv2DRevger : Input image is smaller than kernel");

  nOutputRows = nInputRows - (nKernelRows - 1) * srow;
  nOutputCols = nInputCols - (nKernelCols - 1) * scol;

  long nelem = THCudaTensor_nElement(output);
  THCudaTensor_resize4d(output, nKernelPlane, nInputPlane, nOutputRows, nOutputCols);

  if (nelem == 0 || beta == 0 || nelem != THCudaTensor_nElement(output)) {
    THCudaTensor_zero(output);
  } else if (beta != 1) {
    THCudaTensor_mul(output, beta);
  }

  float *input_data = THCudaTensor_data(input);
  float *kernel_data = THCudaTensor_data(kernel);
  float *output_data = THCudaTensor_data(output);

  // kernel is called multiple times
  // (the arbitrary split below is just here to make sure we dont go over 256 threads)
  for (int sl=0; sl<nbatch; sl+=6) {
    // auto compute nb of blocks and threads
    dim3 blocks(nKernelPlane, nInputPlane);
    int subbatch = 6;
    if (sl+subbatch > nbatch) subbatch = nbatch - sl;
    int cst = 256 / (subbatch * nOutputRows);
    dim3 threads(cst, nOutputRows, subbatch);

    // compute rev conv
    conv2genericrev <<<blocks, threads>>> (input_data + input->stride[0]*sl,
                                           kernel_data + kernel->stride[0]*sl, 
                                           output_data,
                                           nInputPlane, nInputRows, nInputCols,
                                           nKernelPlane, nKernelRows, nKernelCols,
                                           alpha, srow, scol);
  }

  // clean
  THCudaTensor_free(input);
  THCudaTensor_free(kernel);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in conv2DRevger: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}


///////////////////////////////////
///// ConvolutionMap
/*
 * Description:
 *   base conv2D routine: 3D input, 3D output, 4D kernel
 *
 *   - all chunks of data should be contiguous
 *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
 *   - the templated kernel size is useful to generate code that's 2x faster
 *     but can be set to 0 to allow arbitrary kernel sizes
 *   ---- the table should have the first dim with the outputs, each output 
 *   ---- should have a fanin set of inputs contiguously
 */
template <bool swapkernel, int T_kernel_h, int T_kernel_w>
  __global__ void conv2mapgeneric(float *input, float *kernel, float *output,
                                  int input_n, int input_h, int input_w,
                                  int kernel_n, int kernel_h, int kernel_w,
                                  int stride_w, int stride_h,
                                  float *table, int fanin)
{
  // output dimensions
  int output_h = (input_h - kernel_h) / stride_h + 1;
  int output_w = (input_w - kernel_w) / stride_w + 1;

  // xcorr or conv
  int koffset = swapkernel ? kernel_w*kernel_h-1 : 0;

  // nb outputs
  // int output_n = kernel_n / fanin;

  // generate offsets according to block/thread ids
  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  int oo_start = blockIdx.x;
  int oo_end = oo_start+1;

  int table_start = blockIdx.x * (fanin * 2);
  int table_end = table_start + (fanin * 2);

  // nb threads, unique thread id
  int tid = blockDim.x*blockDim.y*threadIdx.z 
    + blockDim.x * threadIdx.y + threadIdx.x;
  int nthreads = blockDim.x * blockDim.y * blockDim.z;

  // iterators
  int oo, ii, xx, yy, kx, ky, kk;

  // do the kernels fit in shared mem ?
  if (kernel_w*kernel_h*kernel_n <= CUDA_SHARED_MEM_SIZE) { 
    // put the kernel in shared memory
    __shared__ float shared_kernel[CUDA_SHARED_MEM_SIZE];

    // first thread of each block does the copy
    for (kk = tid; kk < kernel_w*kernel_h*kernel_n; kk += nthreads) {
      shared_kernel[kk] = kernel[kk];
    }
    __syncthreads();

    // templated kernel size
    if ((T_kernel_w > 0) && (T_kernel_h > 0)) {
      // unrolled convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for (ii = table_start; ii < table_end; ii = ii + 2) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + ((long)table[ii]-1)*input_h*input_w 
                + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              float *kernel_p = shared_kernel 
                + ((long)table[ii + 1]-1) *kernel_w*kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
#pragma unroll
                for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                  for(kx = 0; kx < T_kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
#pragma unroll
                for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                  for(kx = 0; kx < T_kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    } else {
      // default convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for (ii = table_start; ii < table_end; ii++) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dims (between input image and the mask)
              float *input_p = input + ((long)table[ii]-1)*input_h*input_w 
                + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w 
                + xx;
              float *kernel_p = shared_kernel 
                + ((long)table[ii + 1]-1) *kernel_w*kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    }

  } else { // not enough shared mem for kernels, simply stream them

    // convolution loop
    for(oo = oo_start; oo < oo_end; oo++) {
      for (ii = table_start; ii < table_end; ii = ii + 2) {
        for(yy = yy_start; yy < yy_end; yy+=yy_step) {
          for(xx = xx_start; xx < xx_end; xx+=xx_step) {
            // Dot product in two dimensions... (between input image and the mask)
            float *input_p = input + ((long)table[ii]-1)*input_h*input_w 
              + yy*stride_h*input_w + xx*stride_w;
            float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
            float *kernel_p = kernel + ((long)table[ii + 1]-1) *kernel_w*kernel_h + koffset;
            float sum = 0;
            if (swapkernel) {
              for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                for(kx = 0; kx < kernel_w; kx++) {
                  sum += input_p[kx]*(*kernel_p--);
                }
                input_p += input_w;
              }
            } else {
              for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                for(kx = 0; kx < kernel_w; kx++) {
                  sum += input_p[kx]*(*kernel_p++);
                }
                input_p += input_w;
              }
            }
            *output_p += sum;
          }
        }
      }
    }
  }
}


/*
 * API-compatible with THRealTensor_conv2Dmv
 * 3D input, 4D kernel, 3D output
 * matrix vector product like: y <- Ax + beta*y
 */
THC_API void THCudaTensor_conv2Dmap(THCudaTensor *output, THCudaTensor *input,
                                   THCudaTensor *kernel, long stride_x, long stride_y,
                                   THCudaTensor *table, long fanin)
{
  long nInputPlane, nInputRows, nInputCols;
  long nKernelRows, nKernelCols;
  long nOutputPlane, nOutputRows, nOutputCols;

  THArgCheck(kernel->nDimension == 3 , 4, "kernel: 3D Tensor expected");
  THArgCheck(stride_x >= 1, 5, "Stride should be a positive integer");
  THArgCheck(stride_y >= 1, 6, "Stride should be a positive integer");

  input = THCudaTensor_newContiguous(input);
  kernel = THCudaTensor_newContiguous(kernel);
  table = THCudaTensor_newContiguous(table);

  nInputPlane = input->size[0];
  nInputRows  = input->size[1];
  nInputCols  = input->size[2];

  nKernelRows  = kernel->size[1];
  nKernelCols  = kernel->size[2];
  nOutputPlane = kernel->size[0] / fanin;
  // THArgCheck(kernel->size[1] == nInputPlane, 2, "invalid number of input planes");

  THArgCheck( (nInputRows >= nKernelRows && nInputCols >= nKernelCols), 2,
              "conv2Dmap : Input image is smaller than kernel");

  // output dims
  nOutputRows = (nInputRows - nKernelRows) / stride_y + 1;
  nOutputCols = (nInputCols - nKernelCols) / stride_x + 1;

  // long nelem = THCudaTensor_nElement(output);
  THCudaTensor_resize3d(output, nOutputPlane, nOutputRows, nOutputCols);

  float *input_data = THCudaTensor_data(input);
  float *kernel_data = THCudaTensor_data(kernel);
  float *output_data = THCudaTensor_data(output);
  float *table_data = THCudaTensor_data(table);

  // set the number of blocks and threads
  int nthreads_x = 32;
  int nthreads_y = 8;
  int block_height = (int)(16L / nOutputPlane);
  if (block_height < 1)
    block_height = 1;
  dim3 blocks(nOutputPlane,block_height);
  dim3 threads(nthreads_x,nthreads_y);

#define GENERIC_MAP_KERNEL(dim)                                         \
  conv2mapgeneric <false, (dim), (dim)> <<<blocks, threads>>> (         \
      input_data, kernel_data, output_data, nInputPlane, nInputRows,    \
      nInputCols, nOutputPlane*fanin, nKernelRows, nKernelCols,         \
      stride_x, stride_y, table_data, fanin);

  FOR_KERNEL_SPECIALIZED_DIMENSION(nKernelCols, nKernelRows, GENERIC_MAP_KERNEL);
#undef GENERIC_MAP_KERNEL
  // clean
  THCudaTensor_free(input);
  THCudaTensor_free(kernel);
  THCudaTensor_free(table);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in conv2Dmap: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}

#undef FOR_KERNEL_SPECIALIZED_DIMENSION
