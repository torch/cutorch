#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCNumerics.cuh"
#include "THCReduce.cuh"
#include "THCReduceAll.cuh"
#include <thrust/functional.h>

// Reduction operators that support `half`, unlike Thrust
template <typename InT, typename AccT>
struct ReduceAdd {
  inline __device__ AccT operator()(AccT a, InT b) const {
    return a + (AccT) b;
  }
};

#ifdef CUDA_HALF_TENSOR
template <>
struct ReduceAdd<half, half> {
  inline __device__ half operator()(half a, half b) const {
#ifdef CUDA_HALF_INSTRUCTIONS
    return __hadd(a, b);
#else
    float fa = __half2float(a);
    float fb = __half2float(b);
    return __float2half(fa + fb);
#endif
  }
};

template <>
struct ReduceAdd<half, float> {
  inline __device__ float operator()(float a, half b) const {
    return a + __half2float(b);
  }
};
#endif // CUDA_HALF_TENSOR

template <typename InT, typename AccT>
struct ReduceMultiply {
  inline __device__ AccT operator()(AccT a, InT b) const {
    return a * (AccT) b;
  }
};

#ifdef CUDA_HALF_TENSOR
template <>
struct ReduceMultiply<half, half> {
  inline __device__ half operator()(half a, half b) const {
#ifdef CUDA_HALF_INSTRUCTIONS
    return __hmul(a, b);
#else
    float fa = __half2float(a);
    float fb = __half2float(b);
    return __float2half(fa * fb);
#endif
  }
};

template <>
struct ReduceMultiply<half, float> {
  inline __device__ float operator()(float a, half b) const {
    return a * __half2float(b);
  }
};
#endif // CUDA_HALF_TENSOR

template <typename T>
struct ReduceMin {
  inline __device__ T operator()(T a, T b) const {
    return THCNumerics<T>::lt(a, b) ? a : b;
  }
};

template <typename T>
struct ReduceMax {
  inline __device__ T operator()(T a, T b) const {
    return THCNumerics<T>::gt(a, b) ? a : b;
  }
};

struct LogicalAll {
  inline __device__ unsigned char operator()(unsigned char x,
                                             unsigned char y) const {
    return (x && y);
  }
};

struct LogicalAny {
  inline __device__ unsigned char operator()(unsigned char x,
                                             unsigned char y) const {
    return (x || y);
  }
};


THC_API int
THCudaByteTensor_logicalall(THCState *state, THCudaByteTensor *self) {
  THAssert(THCudaByteTensor_checkGPU(state, 1, self));
  unsigned char result;
  if (!THC_reduceAll(state, self,
                     thrust::identity<unsigned char>(),
                     LogicalAll(),
                     LogicalAll(),
                     (unsigned char) 1, &result, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  return (int) result;
}

THC_API int
THCudaByteTensor_logicalany(THCState *state, THCudaByteTensor *self) {
  THAssert(THCudaByteTensor_checkGPU(state, 1, self));
  unsigned char result;
  if (!THC_reduceAll(state, self,
                     thrust::identity<unsigned char>(),
                     LogicalAny(),
                     LogicalAny(),
                     (unsigned char) 0, &result, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  return (int) result;
}


#include <thrust/functional.h>

/* A set of reduction kernels that take in binary ops on thrust pairs (of value, index).
   These are useful when you not only have to do a reduction, but you might have
   to preserve the location of contention (for example min/max operations).
   The structure of the kernels follows the structure of the reduction kernels.
*/
template <typename K, typename Index, class BinaryFunction>
__global__ void
kernelTransformReduceOuterDimIndex(K *tgt1,
                                   Index *tgt2,
                                   K *src_,
                                   unsigned num_orows,
                                   unsigned num_irows,
                                   unsigned row_size,
                                   thrust::pair<K, Index> init,
                                   BinaryFunction binary_op) {
  for (unsigned orow = blockIdx.x; orow < num_orows; orow += gridDim.x) {
    for (unsigned irow = blockIdx.y * blockDim.x + threadIdx.x;
         irow < num_irows;
         irow += gridDim.y * blockDim.x) {
      K *src = src_ + orow * row_size * num_irows + irow;
      thrust::pair<K, Index> acc = init;

      for (unsigned col = 0; col < row_size; ++col) {
        // +1 for Lua index
        acc = binary_op(thrust::make_pair<K, Index>(*src, col+1),
                        acc);
        src += num_irows;
      }

      tgt1[orow * num_irows + irow] = acc.first;
      tgt2[orow * num_irows + irow] = acc.second;
    }
  }
}

template <typename TensorTypeK,
          typename TensorTypeIndex,
          typename BinaryFunction>
__host__ void
THC_transformReduceOuterDimIndex(THCState *state,
                                 TensorTypeK *tgt1,
                                 TensorTypeIndex *tgt2,
                                 TensorTypeK *src,
                                 long rdim,
                                 const thrust::pair<
                                 typename TensorUtils<TensorTypeK>::DataType,
                                 typename TensorUtils<TensorTypeIndex>::DataType>& init,
                                 BinaryFunction binary_op) {
  unsigned ndim = TensorUtils<TensorTypeK>::getDims(state, src);
  unsigned num_orows = 1;
  for (unsigned dim = 0; dim < rdim; dim++) {
    num_orows *= TensorUtils<TensorTypeK>::getSize(state, src, dim);
  }
  unsigned row_size = TensorUtils<TensorTypeK>::getSize(state, src, rdim);
  unsigned num_irows = 1;
  for (unsigned dim = rdim + 1; dim < ndim; dim++) {
    num_irows *= TensorUtils<TensorTypeK>::getSize(state, src, dim);
  }

  dim3 threads(min(512, num_irows));
  unsigned maxGridDim = 1024;
  dim3 grid(min(maxGridDim, num_orows),
            min(maxGridDim, THCCeilDiv(num_irows, threads.x)));

  kernelTransformReduceOuterDimIndex
    <<<grid, threads, 0, THCState_getCurrentStream(state)>>>(
      TensorUtils<TensorTypeK>::getData(state, tgt1),
      TensorUtils<TensorTypeIndex>::getData(state, tgt2),
      TensorUtils<TensorTypeK>::getData(state, src),
      num_orows, num_irows, row_size, init, binary_op);

  THCudaCheck(hipGetLastError());
}

/* Reduce the innermost dimension of a tensor (on thrust::pair functors which are (value, index))
 *
 * For an n-d tensor (n <= 4) where the reduction is along the innermost dimension:
 *
 * - block.x is the innermost dimension, i.e. dimension 0;
 * - block.y and grid.y make up dimension 1; and
 * - grid.x and grid z are the remaining two outer dimensions (if any)
 *
 * Reduction along other dimensions is handled in a separate kernel.
 */
template <typename K, typename Index, class BinaryFunction>
__global__ void
kernelTransformReduceInnermostDimIndex(K *tgt1,
                                       Index* tgt2,
                                       K *src_,
                                       unsigned num_rows,
                                       unsigned row_size,
                                       thrust::pair<K, Index> init,
                                       BinaryFunction binary_op) {
  __shared__ K sbuf[32][16 + 1]; // avoid bank conflict
  __shared__ Index ibuf[32][16 + 1]; // avoid bank conflict

  for (unsigned block_row = blockIdx.x * blockDim.y;
       block_row < num_rows;
       block_row += blockDim.y * gridDim.x) {
    unsigned row = block_row + threadIdx.y;
    thrust::pair<K, Index> acc = init;
    if (row < num_rows) {
      K *src = src_ + row * row_size;
      // Sequential reduction within a thread.
      for (unsigned col = threadIdx.x; col < row_size; col += blockDim.x) {
        acc = binary_op(thrust::make_pair<K, Index>(src[col], col + 1), acc);
      }
    }

    sbuf[threadIdx.y][threadIdx.x] = acc.first;
    ibuf[threadIdx.y][threadIdx.x] = acc.second;

    __syncthreads();

    // Reduce intermediate values to single value.
    K* sline = &sbuf[threadIdx.y][0];
    Index* iline = &ibuf[threadIdx.y][0];
    for (unsigned s = 8; s > 0; s >>= 1) {
      if (row < num_rows && threadIdx.x < s) {
        thrust::pair<K, Index> arg1 =
          thrust::make_pair<K, Index>(sline[threadIdx.x], iline[threadIdx.x]);
        thrust::pair<K, Index> arg2 =
          thrust::make_pair<K, Index>(sline[threadIdx.x + s], iline[threadIdx.x + s]);
        thrust::pair<K, Index> res = binary_op(arg1, arg2);

        sline[threadIdx.x] = res.first;
        iline[threadIdx.x] = res.second;
      }
      __syncthreads();
    }

    if (row < num_rows && threadIdx.x == 0) {
      tgt1[row] = sline[0];
      tgt2[row] = iline[0];
    }
    __syncthreads();
  }
}

template <typename TensorTypeK,
          typename TensorTypeIndex,
          typename BinaryFunction>
__host__ void
THC_transformReduceInnermostDimIndex(THCState *state,
                                     TensorTypeK *tgt1,
                                     TensorTypeIndex *tgt2,
                                     TensorTypeK *src,
                                     const thrust::pair<
                                     typename TensorUtils<TensorTypeK>::DataType,
                                     typename TensorUtils<TensorTypeIndex>::DataType>& init,
                                     BinaryFunction binary_op) {
  unsigned ndim = TensorUtils<TensorTypeK>::getDims(state, src);
  unsigned num_rows = 1;
  for (unsigned dim = 0; dim < ndim - 1; dim++) {
    num_rows *= TensorUtils<TensorTypeK>::getSize(state, src, dim);
  }
  unsigned row_size = TensorUtils<TensorTypeK>::getSize(state, src, ndim - 1);

  dim3 threads(16, 32);
  dim3 grid(min(1024, THCCeilDiv(num_rows, threads.y)));

  kernelTransformReduceInnermostDimIndex
    <<<grid, threads, 0, THCState_getCurrentStream(state)>>>(
      TensorUtils<TensorTypeK>::getData(state, tgt1),
      TensorUtils<TensorTypeIndex>::getData(state, tgt2),
      TensorUtils<TensorTypeK>::getData(state, src),
      num_rows, row_size, init, binary_op);

  THCudaCheck(hipGetLastError());
}

template <typename TensorTypeK,
          typename TensorTypeIndex,
          typename BinaryFunction>
void
THC_reduceDimIndex(THCState *state,
                   TensorTypeK *tgt1_,
                   TensorTypeIndex *tgt2_,
                   TensorTypeK *src,
                   long dimension,
                   const thrust::pair<
                   typename TensorUtils<TensorTypeK>::DataType,
                   typename TensorUtils<TensorTypeIndex>::DataType>& init,
                   BinaryFunction binary_op)
{
  THArgCheck(dimension >= 0 &&
             dimension < TensorUtils<TensorTypeK>::getDims(state, src),
             3, "dimension out of range");

  THLongStorage *dim = TensorUtils<TensorTypeK>::newSizeOf(state, src);
  THLongStorage_set(dim, dimension, 1);
  TensorUtils<TensorTypeK>::resize(state, tgt1_, dim, NULL);
  TensorUtils<TensorTypeIndex>::resize(state, tgt2_, dim, NULL);
  THLongStorage_free(dim);

  TensorTypeK *tgt1 = TensorUtils<TensorTypeK>::newContiguous(state, tgt1_);
  TensorTypeIndex *tgt2 = TensorUtils<TensorTypeIndex>::newContiguous(state, tgt2_);
  src = TensorUtils<TensorTypeK>::newContiguous(state, src);

  if (dimension == TensorUtils<TensorTypeK>::getDims(state, src) - 1) {
    THC_transformReduceInnermostDimIndex(state, tgt1, tgt2, src, init, binary_op);
  } else {
    THC_transformReduceOuterDimIndex(state, tgt1, tgt2, src, dimension, init, binary_op);
  }

  TensorUtils<TensorTypeK>::free(state, src);
  TensorUtils<TensorTypeK>::freeCopyTo(state, tgt1, tgt1_);
  TensorUtils<TensorTypeIndex>::freeCopyTo(state, tgt2, tgt2_);
}

template <typename T, typename Index>
struct MaxValuePair {
  __host__ __device__
  thrust::pair<T, Index> operator()(const thrust::pair<T, Index>& a,
                                    const thrust::pair<T, Index>& b) {
    return THCNumerics<T>::ge(a.first, b.first) ? a : b;
  }
};

template <typename T, typename Index>
struct MinValuePair {
  __host__ __device__
  thrust::pair<T, Index> operator()(const thrust::pair<T, Index>& a,
                                    const thrust::pair<T, Index>& b) {
    return THCNumerics<T>::le(a.first, b.first) ? a : b;
  }
};

#include "generic/THCTensorMathReduce.cu"
#include "THCGenerateAllTypes.h"
