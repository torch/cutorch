#include "hip/hip_runtime.h"
#include "THCGeneral.h"
#include "THCTensorMath.h"
#include "THCTensorCopy.h"
#include <algorithm>

#ifdef USE_MAGMA
#include <magma.h>
#else
#include "THCBlas.h"
#endif

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

#define NoMagma(name) "No CUDA implementation of '" #name "'. Install MAGMA and rebuild cutorch (http://icl.cs.utk.edu/magma/)"

void THCMagma_init(THCState *state)
{
#ifdef USE_MAGMA
  magma_init();
#endif
}

#ifdef USE_MAGMA
static inline float* th_magma_smalloc_pinned(size_t n)
{
  float* ptr;
  if (MAGMA_SUCCESS != magma_smalloc_pinned(&ptr, n))
    THError("$ Torch: not enough memory: you tried to allocate %dGB. Buy new RAM!", n/268435456);
  return ptr;
}

static inline int* th_magma_imalloc_pinned(size_t n)
{
  int* ptr;
  if (MAGMA_SUCCESS != magma_imalloc_pinned(&ptr, n))
    THError("$ Torch: not enough memory: you tried to allocate %dGB. Buy new RAM!", n/268435456);
  return ptr;
}

static void THCudaTensor_copyArray1d(THCState *state, THCudaTensor *self, float *src, int k)
{
  long size[1] = { k };
  long stride[1] = { 1 };
  THCudaTensor_rawResize(state, self, 1, size, stride);
  size_t len = k * sizeof(float);
  THCudaCheck(hipMemcpy(self->storage->data + self->storageOffset, src, len, hipMemcpyHostToDevice));
}

static void THCudaTensor_copyArray2d(THCState *state, THCudaTensor *self, float *src, int m, int n)
{
  long size[2] = { m, n };
  long stride[2] = { 1, m };
  THCudaTensor_rawResize(state, self, 2, size, stride);
  size_t len = m * n * sizeof(float);
  THCudaCheck(hipMemcpy(self->storage->data + self->storageOffset, src, len, hipMemcpyHostToDevice));
}

static void THCudaTensor_copyTensor2d(THCState *state, float *dst, THCudaTensor *self)
{
  THAssert(self->nDimension == 2);
  size_t len = THCudaTensor_nElement(state, self)*sizeof(float);
  THCudaTensor *temp = THCudaTensor_newTranspose(state, self, 0, 1);
  THCudaTensor *selfc = THCudaTensor_newContiguous(state, temp);
  THCudaCheck(hipMemcpy(dst, selfc->storage->data + selfc->storageOffset, len, hipMemcpyDeviceToHost));
  THCudaTensor_free(state, temp);
  THCudaTensor_free(state, selfc);
}

#endif

static THCudaTensor* THCudaTensor_newColumnMajor(THCState *state, THCudaTensor *self, THCudaTensor *src)
{
  THAssert(src->nDimension == 2);
  if (self == src && self->stride[0] == 1 && self->stride[1] == self->size[0])
  {
    THCudaTensor_retain(state, self);
    return self;
  }

  if (self == src)
    self = THCudaTensor_new(state);
  else
    THCudaTensor_retain(state, self);

  long size[2] = { src->size[0], src->size[1] };
  long stride[2] = { 1, src->size[0] };

  THCudaTensor_rawResize(state, self, 2, size, stride);
  THCudaTensor_copy(state, self, src);
  return self;
}


void THCudaTensor_gesv(THCState *state, THCudaTensor *rb_, THCudaTensor *ra_, THCudaTensor *b_, THCudaTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 1, "A should be 2 dimensional");
  THArgCheck(b_->nDimension == 2, 2, "b should be 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 1, "A should be square");
  THArgCheck(b_->size[0] == a_->size[0], 2, "A,b size incompatible");

  int n = a_->size[0];
  int nrhs = b_->size[1];

  THCudaTensor *a = THCudaTensor_newColumnMajor(state, ra_, a_);
  THCudaTensor *b = THCudaTensor_newColumnMajor(state, rb_, b_);
  float *a_data = THCudaTensor_data(state, a);
  float *b_data = THCudaTensor_data(state, b);

  int *ipiv = th_magma_imalloc_pinned(n);

  int info;
  magma_sgesv_gpu(n, nrhs, a_data, n, ipiv, b_data, n, &info);

  if (info < 0)
    THError("MAGMA gesv : Argument %d : illegal value", -info);
  else if (info > 0)
    THError("MAGMA gesv : U(%d,%d) is zero, singular U.", info, info);

  magma_free_pinned(ipiv);
  THCudaTensor_freeCopyTo(state, a, ra_);
  THCudaTensor_freeCopyTo(state, b, rb_);
#else
  THError(NoMagma(gesv));
#endif
}

void THCudaTensor_gels(THCState *state, THCudaTensor *rb_, THCudaTensor *ra_, THCudaTensor *b_, THCudaTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 1, "A should be 2 dimensional");
  THArgCheck(b_->nDimension == 2, 1, "b should be 2 dimensional");
  THArgCheck(a_->size[0] == b_->size[0], 2, "size incompatible A,b");
  THArgCheck(a_->size[0] >= a_->size[1], 2, "A should have m >= n");

  THCudaTensor *a = THCudaTensor_newColumnMajor(state, ra_, a_);
  THCudaTensor *b = THCudaTensor_newColumnMajor(state, rb_, b_);
  float *a_data = THCudaTensor_data(state, a);
  float *b_data = THCudaTensor_data(state, b);

  int m = a->size[0];
  int n = a->size[1];
  int nrhs = b->size[1];
  float wkopt;

  int info;
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);

  float *hwork = th_magma_smalloc_pinned((size_t)wkopt);
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
  magma_free_pinned(hwork);

  if (info != 0)
    THError("MAGMA gels : Argument %d : illegal value", -info);

  THCudaTensor_freeCopyTo(state, a, ra_);
  THCudaTensor_freeCopyTo(state, b, rb_);
#else
  THError(NoMagma(gels));
#endif
}

void THCudaTensor_syev(THCState *state, THCudaTensor *re_, THCudaTensor *rv_, THCudaTensor *a, const char *jobzs, const char *uplos)
{
#ifdef USE_MAGMA
  int n = a->size[0];
  int lda = n;

  magma_uplo_t uplo = uplos[0] == 'U' ?  MagmaUpper : MagmaLower;
  magma_vec_t jobz = jobzs[0] == 'N' ? MagmaNoVec : MagmaVec;

  THCudaTensor *input = THCudaTensor_newColumnMajor(state, rv_, a);
  float *input_data = THCudaTensor_data(state, input);

  // eigen values and workspace
  float *w = th_magma_smalloc_pinned(n);
  float *wA = th_magma_smalloc_pinned(lda);

  // compute optimal size of work array
  int info;
  float lwork;
  int liwork;
  magma_ssyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, &lwork, -1, &liwork, -1, &info);

  float *work = th_magma_smalloc_pinned((size_t)lwork);
  int *iwork = th_magma_imalloc_pinned(liwork);

  // compute eigenvalues and, optionally, eigenvectors
  magma_ssyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, work, (int) lwork, iwork, liwork, &info);

  // copy eigen values from w to re_
  if (info == 0)
    THCudaTensor_copyArray1d(state, re_, w, n);

  magma_free_pinned(iwork);
  magma_free_pinned(work);
  magma_free_pinned(wA);
  magma_free_pinned(w);

  // check error value
  if (info > 0)
    THError("MAGMA syev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
  else if (info < 0)
    THError("MAGMA syev : Argument %d : illegal value", -info);

  THCudaTensor_freeCopyTo(state, input, rv_);
#else
  THError(NoMagma(syev));
#endif
}

void THCudaTensor_geev(THCState *state, THCudaTensor *re_, THCudaTensor *rv_, THCudaTensor *a_, const char *jobvrs)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 3, "A should be 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 3, "A should be square");

  magma_vec_t jobvr = jobvrs[0] == 'N' ? MagmaNoVec : MagmaVec;
  int n = a_->size[0];

  float *a_data = th_magma_smalloc_pinned(n * n);
  THCudaTensor_copyTensor2d(state, a_data, a_);

  float *wr = th_magma_smalloc_pinned(n);
  float *wi = th_magma_smalloc_pinned(n);

  float *vr_data = NULL;
  int ldvr = 1;
  if (jobvr == MagmaVec)
  {
    vr_data = th_magma_smalloc_pinned(n * n);
    ldvr = n;
  }

  float wkopt;
  int info;

  magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, &wkopt, -1, &info);

  int lwork = (int) wkopt;
  float *work_data = th_magma_smalloc_pinned(lwork);

  magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, work_data, lwork, &info);

  if (info > 0)
    THError("MAGMA geev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
  else if (info < 0)
    THError("MAGMA geev : Argument %d : illegal value", -info);

  {
    THCudaTensor_resize2d(state, re_, 2, n);
    THCudaTensor *re = THCudaTensor_newContiguous(state, re_);
    THCudaCheck(hipMemcpy(re->storage->data + re->storageOffset, wr, n*sizeof(float), hipMemcpyHostToDevice));
    THCudaCheck(hipMemcpy(re->storage->data + re->storageOffset + n, wi, n*sizeof(float), hipMemcpyHostToDevice));
    THCudaTensor_freeCopyTo(state, re, re_);
    THCudaTensor_transpose(state, re_, NULL, 0, 1);
  }

  if (jobvr == MagmaVec)
    THCudaTensor_copyArray2d(state, rv_, vr_data, n, n);

  magma_free_pinned(work_data);
  magma_free_pinned(vr_data);
  magma_free_pinned(wi);
  magma_free_pinned(wr);
  magma_free_pinned(a_data);

#else
  THError(NoMagma(geev));
#endif
}

void THCudaTensor_gesvd(THCState *state, THCudaTensor *ru_, THCudaTensor *rs_, THCudaTensor *rv_, THCudaTensor *a, const char *jobu)
{
#ifdef USE_MAGMA
  THCudaTensor *ra_ = THCudaTensor_new(state);
  THCudaTensor_gesvd2(state, ru_, rs_, rv_,  ra_, a, jobu);
  THCudaTensor_free(state, ra_);
#else
  THError(NoMagma(gesvd));
#endif
}

void THCudaTensor_gesvd2(THCState *state, THCudaTensor *ru_, THCudaTensor *rs_, THCudaTensor *rv_, THCudaTensor *ra_, THCudaTensor *a, const char *jobus)
{
#ifdef USE_MAGMA
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");

  magma_vec_t jobu = jobus[0] == 'A' ? MagmaAllVec : jobus[0] == 'S' ? MagmaSomeVec : jobus[0] == 'O' ? MagmaOverwriteVec : MagmaNoVec;
  magma_vec_t jobvt = jobu;

  int m = a->size[0];
  int n = a->size[1];
  int k = m < n ? m : n;
  int j = (jobu == MagmaAllVec) ? m : k;

  float *a_data = th_magma_smalloc_pinned(m * n);
  THCudaTensor_copyTensor2d(state, a_data, a);

  float *rs_data = th_magma_smalloc_pinned(k);
  float *ru_data = th_magma_smalloc_pinned(m * j);
  float *rv_data = th_magma_smalloc_pinned(n * n);

  float wkopt;
  int info;
  magma_sgesvd(jobu, jobvt, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, &wkopt, -1, &info);

  int lwork = (int) wkopt;
  float *work_data = th_magma_smalloc_pinned(lwork);

  magma_sgesvd(jobu, jobvt, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, work_data, lwork, &info);

  if (info > 0)
    THError("MAGMA gesvd : %d superdiagonals failed to converge", info);
  else if (info < 0)
    THError("MAGMA gesvd : Argument %d : illegal value", -info);

  THCudaTensor_copyArray2d(state, rv_, rv_data, n, n);
  THCudaTensor_transpose(state, rv_, NULL, 0, 1);
  THCudaTensor_copyArray2d(state, ru_, ru_data, m, j);
  THCudaTensor_copyArray1d(state, rs_, rs_data, k);
  THCudaTensor_copyArray2d(state, ra_, a_data,  m, n);

  magma_free_pinned(work_data);
  magma_free_pinned(rv_data);
  magma_free_pinned(ru_data);
  magma_free_pinned(rs_data);
  magma_free_pinned(a_data);
#else
  THError(NoMagma(gesvd2));
#endif
}

void THCudaTensor_getri(THCState *state, THCudaTensor *ra_, THCudaTensor *a)
{
#ifdef USE_MAGMA
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int info;
  int n = a->size[0];
  int lwork = n * magma_get_sgetri_nb(n);

  THCudaTensor *input = THCudaTensor_newColumnMajor(state, ra_, a);
  float *input_data = THCudaTensor_data(state, input);

  int *ipiv = th_magma_imalloc_pinned(n);

  THCudaTensor *work = THCudaTensor_newWithSize1d(state, lwork);
  float *work_data = THCudaTensor_data(state, work);

  // Run LU
  magma_sgetrf_gpu(n, n, input_data, n, ipiv, &info);
  if (info > 0)
    THError("MAGMA getrf : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("MAGMA getrf : Argument %d : illegal value", -info);

  // Inverse
  magma_sgetri_gpu(n, input_data, n, ipiv, work_data, lwork, &info);
  if (info > 0)
    THError("MAGMA getri : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("MAGMA getri : Argument %d : illegal value", -info);

  THCudaTensor_free(state, work);
  magma_free_pinned(ipiv);
  THCudaTensor_freeCopyTo(state, input, ra_);
#else
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int n = a->size[0];

  // input
  THCudaTensor *input = THCudaTensor_newColumnMajor(state, ra_, a);
  // output
  THCudaTensor *output = THCudaTensor_newColumnMajor(state, ra_, a);

  size_t matrices_size = sizeof(float*);

  float **matrices1 = (float **)THAlloc(matrices_size);
  const float **matrices1_const = (const float **)THAlloc(matrices_size);
  float **matrices2 = (float **)THAlloc(matrices_size);
  matrices1[0] = THCudaTensor_data(state, input);
  matrices1_const[0] = THCudaTensor_data(state, input);
  matrices2[0] = THCudaTensor_data(state, output);

  // Copy pointers to device.
  float **d_matrices1, **d_matrices2;
  const float **d_matrices1_const;
  THCudaCheck(THCudaMalloc(state, (void**)&d_matrices1, matrices_size));
  THCudaCheck(THCudaMalloc(state, (void**)&d_matrices1_const, matrices_size));
  THCudaCheck(THCudaMalloc(state, (void**)&d_matrices2, matrices_size));

  THCudaCheck(hipMemcpyAsync(d_matrices1, matrices1, matrices_size,
                              hipMemcpyHostToDevice, THCState_getCurrentStream(state)));
  THCudaCheck(hipMemcpyAsync(d_matrices1_const, matrices1_const, matrices_size,
                              hipMemcpyHostToDevice, THCState_getCurrentStream(state)));
  THCudaCheck(hipMemcpyAsync(d_matrices2, matrices2, matrices_size,
                              hipMemcpyHostToDevice, THCState_getCurrentStream(state)));
  int info;
  int *info_gpu;
  THCudaCheck(THCudaMalloc(state, (void**)&info_gpu, sizeof(int)));

  int *ipiv_gpu;
  THCudaCheck(THCudaMalloc(state, (void**)&ipiv_gpu, n * sizeof(int)));

  // Run LU
  THCudaBlas_Sgetrf(state, n, d_matrices1, n, ipiv_gpu, info_gpu, 1);

  THCudaCheck(hipMemcpy(&info, info_gpu, sizeof(int), hipMemcpyDeviceToHost));

  if (info > 0)
    THError("CUBLAS getrf : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("CUBLAS getrf : Argument %d : illegal value", -info);

  // Inverse
  THCudaBlas_Sgetri(state, n, d_matrices1_const, n, ipiv_gpu, d_matrices2, n, info_gpu, 1);
  if (info > 0)
    THError("CUBLAS getri : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("CUBLAS getri : Argument %d : illegal value", -info);

  THCudaCheck(THCudaFree(state, ipiv_gpu));
  THCudaCheck(THCudaFree(state, info_gpu));
  THCudaTensor_freeCopyTo(state, output, input);
#endif

}


__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
    const int r = idx % n;
    const int c = idx / n;
    if (r > c) {
      input[idx] = input[r*n + c];
    }
  }
}

void THCudaTensor_potri(THCState *state, THCudaTensor *ra_, THCudaTensor *a)
{
#ifdef USE_MAGMA
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int n = a->size[0];

  THCudaTensor *input = THCudaTensor_newColumnMajor(state, ra_, a);
  float *input_data = THCudaTensor_data(state, input);

  int info;
  magma_spotrf_gpu(MagmaUpper, n, input_data, n, &info);
  if (info > 0)
    THError("MAGMA potrf : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potrf : Argument %d : illegal value", -info);

  magma_spotri_gpu(MagmaUpper, n, input_data, n, &info);
  if (info > 0)
    THError("MAGMA potri : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potri : Argument %d : illegal value", -info);

  hipStream_t stream = THCState_getCurrentStream(state);
  const int len = n*n;
  dim3 blocks(std::min(DIVUP(len, 128), 65535));
  dim3 threads(128);
  THCudaTensor_copyUpperSymmetric<<<blocks, threads, 0, stream>>>(input_data, n, len);

  THCudaTensor_freeCopyTo(state, input, ra_);
#else
  THError(NoMagma(potri));
#endif
}

void THCudaTensor_potrf(THCState *state, THCudaTensor *ra_, THCudaTensor *a)
{
#ifdef USE_MAGMA
  THArgCheck(a->nDimension == 2, 2, "A should be 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int n = a->size[0];

  THCudaTensor *input = THCudaTensor_newColumnMajor(state, ra_, a);
  float *input_data = THCudaTensor_data(state, input);

  int info;
  magma_spotrf_gpu(MagmaUpper, n, input_data, n, &info);

  // check error value
  if (info > 0)
    THError("MAGMA potrf : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potrf : Argument %d : illegal value", -info);

  THCudaTensor_triu(state, ra_, input, 0);
  THCudaTensor_free(state, input);
#else
  THError(NoMagma(potrf));
#endif
}

void THCudaTensor_potrs(THCState *state, THCudaTensor *rb_, THCudaTensor *b, THCudaTensor *a)
{
#ifdef USE_MAGMA
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int n = a->size[0];
  int nrhs = b->size[1];

  THCudaTensor *b_ = THCudaTensor_newColumnMajor(state, rb_, b);
  float *b_data = THCudaTensor_data(state, b_);
  THCudaTensor *a_ = THCudaTensor_newColumnMajor(state, a, a);
  float *a_data = THCudaTensor_data(state, a_);

  int info;
  magma_spotrs_gpu(MagmaUpper, n, nrhs, a_data, n, b_data, n, &info);

  // check error value
  if (info < 0)
    THError("MAGMA potrs : Argument %d : illegal value", -info);

  THCudaTensor_freeCopyTo(state, b_, rb_);
  THCudaTensor_free(state, a_);
#else
  THError(NoMagma(potrs));
#endif
}

void THCudaTensor_qr(THCState *state, THCudaTensor *rq_, THCudaTensor *rr_, THCudaTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 2, "A should be 2 dimensional");

  THCudaTensor *a = THCudaTensor_newColumnMajor(state, rr_, a_);
  int m = a->size[0];
  int n = a->size[1];
  int k = (m < n ? m : n);

#ifdef MAGMA_V2
  int nb = magma_get_sgeqrf_nb(m, n);
#else
  int nb = magma_get_sgeqrf_nb(m);
#endif

  float *a_data = THCudaTensor_data(state, a);
  float *tau_data = th_magma_smalloc_pinned(n*n);

  THCudaTensor *work = THCudaTensor_newWithSize1d(state, (2*k + ((n+31)/32)*32)*nb);
  float *work_data = THCudaTensor_data(state, work);

  int info;
  magma_sgeqrf_gpu(m, n, a_data, m, tau_data, work_data, &info);

  if (info != 0)
    THError("MAGMA geqrf : Argument %d : illegal value.", -info);

  THCudaTensor *q = THCudaTensor_newColumnMajor(state, rq_, a);
  float *q_data = THCudaTensor_data(state, q);

  THCudaTensor_narrow(state, a, a, 0, 0, k);
  THCudaTensor_triu(state, rr_, a, 0);
  THCudaTensor_free(state, a);

  magma_sorgqr_gpu(m, n, k, q_data, m, tau_data, work_data, nb, &info);

  if (info != 0)
    THError("MAGMA orgqr : Argument %d : illegal value.", -info);

  THCudaTensor_free(state, work);
  magma_free_pinned(tau_data);

  THCudaTensor_narrow(state, q, q, 1, 0, k);
  THCudaTensor_freeCopyTo(state, q, rq_);
#else
  THError(NoMagma(qr));
#endif
}
