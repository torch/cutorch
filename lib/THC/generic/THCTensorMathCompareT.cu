
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathCompareT.cu"
#else

THC_API void
THCTensor_(ltTensor)(THCState *state, THCudaByteTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorLTOp<typename TensorUtils<THCTensor>::DataType,
                    unsigned char>());
}

THC_API void
THCTensor_(gtTensor)(THCState *state, THCudaByteTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorGTOp<typename TensorUtils<THCTensor>::DataType,
                    unsigned char>());
}

THC_API void
THCTensor_(leTensor)(THCState *state, THCudaByteTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorLEOp<typename TensorUtils<THCTensor>::DataType,
                    unsigned char>());
}

THC_API void
THCTensor_(geTensor)(THCState *state, THCudaByteTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorGEOp<typename TensorUtils<THCTensor>::DataType,
                    unsigned char>());
}

THC_API void
THCTensor_(eqTensor)(THCState *state, THCudaByteTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorEQOp<typename TensorUtils<THCTensor>::DataType,
                    unsigned char>());
}

THC_API void
THCTensor_(neTensor)(THCState *state, THCudaByteTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorNEOp<typename TensorUtils<THCTensor>::DataType,
                    unsigned char>());
}

THC_API void
THCTensor_(ltTensorT)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorLTOp<typename TensorUtils<THCTensor>::DataType,
                    typename TensorUtils<THCTensor>::DataType>());
}

THC_API void
THCTensor_(gtTensorT)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorGTOp<typename TensorUtils<THCTensor>::DataType,
                    typename TensorUtils<THCTensor>::DataType>());
}

THC_API void
THCTensor_(leTensorT)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorLEOp<typename TensorUtils<THCTensor>::DataType,
                    typename TensorUtils<THCTensor>::DataType>());
}

THC_API void
THCTensor_(geTensorT)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorGEOp<typename TensorUtils<THCTensor>::DataType,
                    typename TensorUtils<THCTensor>::DataType>());
}

THC_API void
THCTensor_(eqTensorT)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorEQOp<typename TensorUtils<THCTensor>::DataType,
                    typename TensorUtils<THCTensor>::DataType>());
}

THC_API void
THCTensor_(neTensorT)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THC_logicalTensor(state, self_, src1, src2,
                    TensorNEOp<typename TensorUtils<THCTensor>::DataType,
                    typename TensorUtils<THCTensor>::DataType>());
}

#endif
