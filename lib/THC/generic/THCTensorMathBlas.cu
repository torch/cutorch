
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathBlas.cu"
#else

THC_API accreal
THCTensor_(dot)(THCState *state, THCTensor *self, THCTensor *src)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  THAssert(THCTensor_(checkGPU)(state, 2, self, src));
  THArgCheck(THCTensor_(nElement)(state, self) ==
             THCTensor_(nElement)(state, src), 2, "sizes do not match");

  self = THCTensor_(newContiguous)(state, self);
  src = THCTensor_(newContiguous)(state, src);

#ifdef THC_REAL_IS_FLOAT
  accreal result = THCudaBlas_Sdot(state,
                                THCTensor_(nElement)(state, self),
                                THCTensor_(data)(state, self), 1,
                                THCTensor_(data)(state, src), 1);
#elif defined(THC_REAL_IS_DOUBLE)
  accreal result = THCudaBlas_Ddot(state,
                                THCTensor_(nElement)(state, self),
                                THCTensor_(data)(state, self), 1,
                                THCTensor_(data)(state, src), 1);
#elif defined(THC_REAL_IS_HALF)
  accreal result = THCudaBlas_Hdot(state,
                                THCTensor_(nElement)(state, self),
                                THCTensor_(data)(state, self), 1,
                                THCTensor_(data)(state, src), 1);
#endif

  THCTensor_(free)(state, src);
  THCTensor_(free)(state, self);
  return result;

#else
  THError("unimplemented data type");
  return ScalarConvert<int, accreal>::to(0);
#endif
}

THC_API void
THCTensor_(addmv)(THCState *state, THCTensor *r_, real beta, THCTensor *t, real alpha, THCTensor *mat, THCTensor *vec)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  THAssert(THCTensor_(checkGPU)(state, 4, r_, t, mat, vec));
  if( (mat->nDimension != 2) || (vec->nDimension != 1) )
    THError("matrix and vector expected");

  if( mat->size[1] != vec->size[0] )
    THError("size mismatch");

  if(t->nDimension != 1)
    THError("size mismatch");

  if(t->size[0] != mat->size[0])
    THError("size mismatch");

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  if(r_ != t)
  {
    THCTensor_(resizeAs)(state, r_, t);
    THCTensor_(copy)(state, r_, t);
  }

  if(mat->stride[0] == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 'n', mat->size[0], mat->size[1],
                    alpha, THCTensor_(data)(state, mat), mat->stride[1],
                    THCTensor_(data)(state, vec), vec->stride[0],
                    beta, THCTensor_(data)(state, r_), r_->stride[0]);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 'n', mat->size[0], mat->size[1],
                    alpha, THCTensor_(data)(state, mat), mat->stride[1],
                    THCTensor_(data)(state, vec), vec->stride[0],
                    beta, THCTensor_(data)(state, r_), r_->stride[0]);
#endif
  }
  else if(mat->stride[1] == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 't',  mat->size[1], mat->size[0],
                    alpha, THCTensor_(data)(state, mat), mat->stride[0],
                    THCTensor_(data)(state, vec), vec->stride[0],
                    beta, THCTensor_(data)(state, r_), r_->stride[0]);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 't',  mat->size[1], mat->size[0],
                     alpha, THCTensor_(data)(state, mat), mat->stride[0],
                     THCTensor_(data)(state, vec), vec->stride[0],
                     beta, THCTensor_(data)(state, r_), r_->stride[0]);
#endif
  }
  else
  {
    THCTensor *cmat = THCTensor_(newContiguous)(state, mat);

#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemv(state, 't',  mat->size[1], mat->size[0],
                    alpha, THCTensor_(data)(state, cmat), cmat->stride[0],
                    THCTensor_(data)(state, vec), vec->stride[0],
                    beta, THCTensor_(data)(state, r_), r_->stride[0]);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemv(state, 't',  mat->size[1], mat->size[0],
                    alpha, THCTensor_(data)(state, cmat), cmat->stride[0],
                    THCTensor_(data)(state, vec), vec->stride[0],
                    beta, THCTensor_(data)(state, r_), r_->stride[0]);
#endif

    THCTensor_(free)(state, cmat);
  }

#elif defined(THC_REAL_IS_HALF)
    // Currently no Hgemv/SgemvEx in Cublas
    THCTensor *vecAsMatrix = THCTensor_(newWithTensor)(state, vec);
    THCTensor_(resize2d)(state, vecAsMatrix, vecAsMatrix->size[0], 1);

    THCTensor *tAsMatrix = THCTensor_(newWithTensor)(state, t);
    THCTensor_(resize2d)(state, tAsMatrix, tAsMatrix->size[0], 1);

    THCTensor_(addmm)(state, r_, beta, tAsMatrix, alpha, mat, vecAsMatrix);

    // r_ will have answer as matrix, need to return a vecotr
    THCTensor_(resize1d)(state, r_, r_->size[0]);
    THCTensor_(free)(state, vecAsMatrix);
    THCTensor_(free)(state, tAsMatrix);
#endif
#else
  THError("unimplemented data type");
#endif
}

THC_API void
THCTensor_(addr)(THCState *state, THCTensor *r_, real beta, THCTensor *t, real alpha, THCTensor *vec1, THCTensor *vec2)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)
  THAssert(THCTensor_(checkGPU)(state, 4, r_, t, vec1, vec2));
  if ( (vec1->nDimension != 1) || (vec2->nDimension != 1) ) {
    THError("vector and vector expected");
  }

  if (t->nDimension != 2) {
    THError("size mismatch");
  }

  if ( (t->size[0] != vec1->size[0]) || (t->size[1] != vec2->size[0]) ) {
    THError("size mismatch");
  }

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  if (r_ != t) {
    THCTensor_(resizeAs)(state, r_, t);
    THCTensor_(copy)(state, r_, t);
  }

  if(THCNumerics<real>::ne(beta, ScalarConvert<int, real>::to(1))) {
    THCTensor_(mul)(state, r_, r_, beta);
  }

  if(r_->stride[0] == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sger(state, vec1->size[0], vec2->size[0],
                   alpha, THCTensor_(data)(state, vec1), vec1->stride[0],
                   THCTensor_(data)(state, vec2), vec2->stride[0],
                   THCTensor_(data)(state, r_), r_->stride[1]);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dger(state, vec1->size[0], vec2->size[0],
                   alpha, THCTensor_(data)(state, vec1), vec1->stride[0],
                   THCTensor_(data)(state, vec2), vec2->stride[0],
                   THCTensor_(data)(state, r_), r_->stride[1]);
#endif
  }
  else if(r_->stride[1] == 1)
  {
#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sger(state, vec2->size[0], vec1->size[0],
                   alpha, THCTensor_(data)(state, vec2), vec2->stride[0],
                   THCTensor_(data)(state, vec1), vec1->stride[0],
                   THCTensor_(data)(state, r_), r_->stride[0]);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dger(state, vec2->size[0], vec1->size[0],
                   alpha, THCTensor_(data)(state, vec2), vec2->stride[0],
                   THCTensor_(data)(state, vec1), vec1->stride[0],
                   THCTensor_(data)(state, r_), r_->stride[0]);
#endif
  }
  else
  {
    THCTensor *cr = THCTensor_(newClone)(state, r_);

#ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sger(state, vec2->size[0], vec1->size[0],
                   alpha, THCTensor_(data)(state, vec2), vec2->stride[0],
                   THCTensor_(data)(state, vec1), vec1->stride[0],
                   THCTensor_(data)(state, cr), cr->stride[0]);
#elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dger(state, vec2->size[0], vec1->size[0],
                   alpha, THCTensor_(data)(state, vec2), vec2->stride[0],
                   THCTensor_(data)(state, vec1), vec1->stride[0],
                   THCTensor_(data)(state, cr), cr->stride[0]);
#endif

    THCTensor_(freeCopyTo)(state, cr, r_);
  }
#elif defined(THC_REAL_IS_HALF)
  // currently no Hger/SgerEx in Cublas.
  THCTensor *vec2T = THCTensor_(newWithTensor)(state, vec2);
  THCTensor_(resize2d)(state, vec2T, vec2T->size[0], 1);
  THCTensor_(transpose)(state, vec2T, NULL, 0, 1);

  THCTensor *vec1M = THCTensor_(newWithTensor)(state, vec1);
  THCTensor_(resize2d)(state, vec1M, vec1M->size[0], 1);

  THCTensor_(addmm)(state, r_, beta, t, alpha, vec1M, vec2T);
  THCTensor_(free)(state, vec2T);
  THCTensor_(free)(state, vec1M);
#endif
#else
  THError("unimplemented data type");
#endif
}

THC_API void
THCTensor_(addmm)(THCState *state, THCTensor *r_, real beta, THCTensor *t, real alpha, THCTensor *m1, THCTensor *m2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)

  THAssert(THCTensor_(checkGPU)(state, 4, r_, t, m1, m2));
  char transpose_r, transpose_m1, transpose_m2;
  THCTensor *r__, *m1_, *m2_;

  if( (m1->nDimension != 2) || (m2->nDimension != 2) )
    THError("matrix and matrix expected");

  if(t->nDimension != 2)
    THError("size mismatch");

  if( (t->size[0] != m1->size[0]) || (t->size[1] != m2->size[1]) || (m1->size[1] != m2->size[0]) )
    THError("size mismatch");

  if(t != r_)
  {
    THCTensor_(resizeAs)(state, r_, t);
    THCTensor_(copy)(state, r_, t);
  }

  /* r_ */
  if(r_->stride[0] == 1 &&
     r_->stride[1] != 0)
  {
    transpose_r = 'n';
    r__ = r_;
  }
  else if(r_->stride[1] == 1 &&
          r_->stride[0] != 0)
  {
    THCTensor *swap = m2;
    m2 = m1;
    m1 = swap;
    transpose_r = 't';
    r__ = r_;
  }
  else
  {
    transpose_r = 'n';

    THCTensor *transp_r_ = THCTensor_(newTranspose)(state, r_, 0, 1);
    r__ = THCTensor_(newClone)(state, transp_r_);
    THCTensor_(free)(state, transp_r_);
    THCTensor_(transpose)(state, r__, NULL, 0, 1);
  }

  /* m1 */
  if(m1->stride[(transpose_r == 'n' ? 0 : 1)] == 1 &&
     m1->stride[(transpose_r == 'n' ? 1 : 0)] != 0)
  {
    transpose_m1 = 'n';
    m1_ = m1;
  }
  else if(m1->stride[(transpose_r == 'n' ? 1 : 0)] == 1 &&
          m1->stride[(transpose_r == 'n' ? 0 : 1)] != 0)
  {
    transpose_m1 = 't';
    m1_ = m1;
  }
  else
  {
    transpose_m1 = (transpose_r == 'n' ? 't' : 'n');
    m1_ = THCTensor_(newContiguous)(state, m1);
  }

  /* m2 */
  if(m2->stride[(transpose_r == 'n' ? 0 : 1)] == 1 &&
     m2->stride[(transpose_r == 'n' ? 1 : 0)] != 0)
  {
    transpose_m2 = 'n';
    m2_ = m2;
  }
  else if(m2->stride[(transpose_r == 'n' ? 1 : 0)] == 1 &&
          m2->stride[(transpose_r == 'n' ? 0 : 1)] != 0)
  {
    transpose_m2 = 't';
    m2_ = m2;
  }
  else
  {
    transpose_m2 = (transpose_r == 'n' ? 't' : 'n');
    m2_ = THCTensor_(newContiguous)(state, m2);
  }

#ifdef THC_REAL_IS_HALF
  THCudaBlas_Hgemm(state,
                   transpose_m1,
                   transpose_m2,
                   r__->size[(transpose_r == 'n' ? 0 : 1)],
                   r__->size[(transpose_r == 'n' ? 1 : 0)],
                   m1_->size[(transpose_r == 'n' ? 1 : 0)],
                   alpha,
                   THCTensor_(data)(state, m1_),
                   (transpose_m1 == 'n' ? m1_->stride[(transpose_r == 'n' ? 1 : 0)] : m1_->stride[(transpose_r == 'n' ? 0 : 1)]),
                   THCTensor_(data)(state, m2_),
                   (transpose_m2 == 'n' ? m2_->stride[(transpose_r == 'n' ? 1 : 0)] : m2_->stride[(transpose_r == 'n' ? 0 : 1)]),
                   beta,
                   THCTensor_(data)(state, r__),
                   r__->stride[(transpose_r == 'n' ? 1 : 0)]);
#elif defined(THC_REAL_IS_FLOAT)
  THCudaBlas_Sgemm(state,
                   transpose_m1,
                   transpose_m2,
                   r__->size[(transpose_r == 'n' ? 0 : 1)],
                   r__->size[(transpose_r == 'n' ? 1 : 0)],
                   m1_->size[(transpose_r == 'n' ? 1 : 0)],
                   alpha,
                   THCTensor_(data)(state, m1_),
                   (transpose_m1 == 'n' ? m1_->stride[(transpose_r == 'n' ? 1 : 0)] : m1_->stride[(transpose_r == 'n' ? 0 : 1)]),
                   THCTensor_(data)(state, m2_),
                   (transpose_m2 == 'n' ? m2_->stride[(transpose_r == 'n' ? 1 : 0)] : m2_->stride[(transpose_r == 'n' ? 0 : 1)]),
                   beta,
                   THCTensor_(data)(state, r__),
                   r__->stride[(transpose_r == 'n' ? 1 : 0)]);
#elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_Dgemm(state,
                   transpose_m1,
                   transpose_m2,
                   r__->size[(transpose_r == 'n' ? 0 : 1)],
                   r__->size[(transpose_r == 'n' ? 1 : 0)],
                   m1_->size[(transpose_r == 'n' ? 1 : 0)],
                   alpha,
                   THCTensor_(data)(state, m1_),
                   (transpose_m1 == 'n' ? m1_->stride[(transpose_r == 'n' ? 1 : 0)] : m1_->stride[(transpose_r == 'n' ? 0 : 1)]),
                   THCTensor_(data)(state, m2_),
                   (transpose_m2 == 'n' ? m2_->stride[(transpose_r == 'n' ? 1 : 0)] : m2_->stride[(transpose_r == 'n' ? 0 : 1)]),
                   beta,
                   THCTensor_(data)(state, r__),
                   r__->stride[(transpose_r == 'n' ? 1 : 0)]);
#endif

  /* free intermediate variables */
  if(m1_ != m1) {
    THCTensor_(free)(state, m1_);
  }

  if(m2_ != m2) {
    THCTensor_(free)(state, m2_);
  }

  if(r__ != r_) {
    THCTensor_(freeCopyTo)(state, r__, r_);
  }
#else
  THError("unimplemented data type");
#endif
}

THC_API void
THCTensor_(addbmm)(THCState *state, THCTensor *result, real beta, THCTensor *t,
                   real alpha, THCTensor *batch1, THCTensor *batch2) {
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  THAssert(THCTensor_(checkGPU)(state, 4, result, t, batch1, batch2));
  THArgCheck(THCTensor_(nDimension)(state, t) == 2, 4, "expected 2D tensor");
  THArgCheck(THCTensor_(nDimension)(state, batch1) == 3, 6, "expected 3D tensor");
  THArgCheck(THCTensor_(nDimension)(state, batch2) == 3, 7, "expected 3D tensor");

  long batchnum = THCTensor_(size)(state, batch1, 0);
  long m1d1 = THCTensor_(size)(state, batch1, 1);
  long innerdim = THCTensor_(size)(state, batch1, 2);
  long m2d2 = THCTensor_(size)(state, batch2, 2);

  THArgCheck(batchnum == THCTensor_(size)(state, batch2, 0), 7,
      "equal number of batches expected");
  // M is t, as listed in the docs under addbmm
  THArgCheck(m1d1 == THCTensor_(size)(state, t, 0), 6,
      "first dimension must match first dimension of M");
  THArgCheck(m2d2 == THCTensor_(size)(state, t, 1), 7,
      "second dimension must match second dimension of M");
  THArgCheck(innerdim == THCTensor_(size)(state, batch2, 1), 6,
      "second dimension must match first dimension of batch2");

  if (t != result) {
    THCTensor_(resizeAs)(state, result, t);
    THCTensor_(copy)(state, result, t);
  }

  THCTensor *slice1 = THCTensor_(new)(state);
  THCTensor *slice2 = THCTensor_(new)(state);
  for (long i=0; i<batchnum; i++) {
    THCTensor_(select)(state, slice1, batch1, 0, i);
    THCTensor_(select)(state, slice2, batch2, 0, i);

    THCTensor_(addmm)(state, result, beta, result, alpha, slice1, slice2);
    beta = ScalarConvert<int, real>::to(1);
  }
  THCTensor_(free)(state, slice1);
  THCTensor_(free)(state, slice2);
#else
  THError("unimplemented data type");
#endif
}

__global__ void createBatchGemmBuffer(const real** buffer, real* data,
                                      long stride, long num_batches) {
  const long idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_batches) {
    buffer[idx] = data + idx * stride;
   }
}

THC_API void
THCTensor_(baddbmm)(THCState *state, THCTensor *result, real beta, THCTensor *t,
                    real alpha, THCTensor *batch1, THCTensor *batch2) {
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  THAssert(THCTensor_(checkGPU)(state, 4, result, t, batch1, batch2));
  THArgCheck(THCTensor_(nDimension)(state, t) == 3, 4, "expected 3D tensor");
  THArgCheck(THCTensor_(nDimension)(state, batch1) == 3, 6, "expected 3D tensor");
  THArgCheck(THCTensor_(nDimension)(state, batch2) == 3, 7, "expected 3D tensor");
  THArgCheck(THCTensor_(size)(state, t, 0) == THCTensor_(size)(state, batch1, 0), 6,
             "equal number of batches expected");
  THArgCheck(THCTensor_(size)(state, t, 0) == THCTensor_(size)(state, batch2, 0), 7,
             "equal number of batches expected");
  THArgCheck(THCTensor_(size)(state, t, 1) == THCTensor_(size)(state, batch1, 1), 6,
             "wrong matrix size");
  THArgCheck(THCTensor_(size)(state, t, 2) == THCTensor_(size)(state, batch2, 2), 7,
             "wrong matrix size");
  THArgCheck(THCTensor_(size)(state, batch1, 2) == THCTensor_(size)(state, batch2, 1), 6,
             "wrong matrix size");

  if (t != result) {
    THCTensor_(resizeAs)(state, result, t);
    THCTensor_(copy)(state, result, t);
  }

  bool transpose_result;
  char transpose_batch1, transpose_batch2;
  long lda, ldb, ldc;
  THCTensor *result_, *batch1_, *batch2_;
  if (result->stride[1] == 1)
  {
    transpose_result = false;
    result_ = result;
    ldc = result_->stride[2];
  }
  else if (result->stride[2] == 1)
  {
    transpose_result = true;

    THCTensor *swap = batch2;
    batch2 = batch1;
    batch1 = swap;

    result_ = result;
    ldc = result_->stride[1];
  }
  else
  {
    transpose_result = false;

    THCTensor *transp_r_ = THCTensor_(newTranspose)(state, result, 1, 2);
    result_ = THCTensor_(newClone)(state, transp_r_);
    THCTensor_(free)(state, transp_r_);
    THCTensor_(transpose)(state, result_, NULL, 1, 2);

    ldc = result_->stride[2];
  }

  if (batch1->stride[transpose_result ? 2 : 1] == 1)
  {
    transpose_batch1 = 'n';
    batch1_ = batch1;
    lda = batch1_->stride[transpose_result ? 1 : 2];
  }
  else if (batch1->stride[transpose_result ? 1 : 2] == 1)
  {
    transpose_batch1 = 't';
    batch1_ = batch1;
    lda = batch1_->stride[transpose_result ? 2 : 1];
  }
  else
  {
    transpose_batch1 = transpose_result ? 'n' : 't';
    batch1_ = THCTensor_(newContiguous)(state, batch1);
    lda = batch1_->stride[1];
  }

  if (batch2->stride[transpose_result ? 2 : 1] == 1)
  {
    transpose_batch2 = 'n';
    batch2_ = batch2;
    ldb = batch2_->stride[transpose_result ? 1 : 2];
  }
  else if (batch2->stride[transpose_result ? 1 : 2] == 1)
  {
    transpose_batch2 = 't';
    batch2_ = batch2;
    ldb = batch2_->stride[transpose_result ? 2 : 1];
  }
  else
  {
    transpose_batch2 = transpose_result ? 'n' : 't';
    batch2_ = THCTensor_(newContiguous)(state, batch2);
    ldb = batch2_->stride[1];
  }

  // Compute pointers to matrices in each batch.
  long num_batches = result_->size[0];
  size_t matrices_size = num_batches * sizeof(real*);

  // Copy pointers to device.
  const real **d_matrices1, **d_matrices2;
  real **d_result_matrices;
  THCudaCheck(THCudaMalloc(state, (void**)&d_matrices1, matrices_size));
  THCudaCheck(THCudaMalloc(state, (void**)&d_matrices2, matrices_size));
  THCudaCheck(THCudaMalloc(state, (void**)&d_result_matrices, matrices_size));

  const long block = 512;
  const long grid = (num_batches + block - 1) / block;

  createBatchGemmBuffer<<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    d_matrices1, THCTensor_(data)(state, batch1_), batch1_->stride[0],
    num_batches);
  createBatchGemmBuffer<<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    d_matrices2, THCTensor_(data)(state, batch2_), batch2_->stride[0],
    num_batches);
  createBatchGemmBuffer<<<grid, block, 0, THCState_getCurrentStream(state)>>>(
    (const real**)d_result_matrices, THCTensor_(data)(state,result_),
    result_->stride[0], num_batches);

#ifdef THC_REAL_IS_FLOAT
  THCudaBlas_SgemmBatched(
      state,
      transpose_batch1,
      transpose_batch2,
      result_->size[transpose_result ? 2 : 1],
      result_->size[transpose_result ? 1 : 2],
      batch1_->size[transpose_result ? 1 : 2],
      alpha,
      d_matrices1, lda,
      d_matrices2, ldb,
      beta,
      d_result_matrices, ldc,
      num_batches);
#elif defined(THC_REAL_IS_DOUBLE)
  THCudaBlas_DgemmBatched(
      state,
      transpose_batch1,
      transpose_batch2,
      result_->size[transpose_result ? 2 : 1],
      result_->size[transpose_result ? 1 : 2],
      batch1_->size[transpose_result ? 1 : 2],
      alpha,
      d_matrices1, lda,
      d_matrices2, ldb,
      beta,
      d_result_matrices, ldc,
      num_batches);
#endif

  THCudaFree(state, d_matrices1);
  THCudaFree(state, d_matrices2);
  THCudaFree(state, d_result_matrices);

  if (batch1_ != batch1) {
    THCTensor_(free)(state, batch1_);
  }

  if (batch2_ != batch2) {
    THCTensor_(free)(state, batch2_);
  }

  if (result_ != result) {
    THCTensor_(freeCopyTo)(state, result_, result);
  }

#else
  THError("unimplemented data type");
#endif
}

#endif
