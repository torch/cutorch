
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMath.cu"
#else

THC_API void
THCTensor_(fill)(THCState* state, THCTensor *self_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self_));

  if (!THC_pointwiseApply1(
        state, self_, TensorFillOp<real>(value))) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(zero)(THCState *state, THCTensor *self_)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self_));
  if (THCTensor_(isContiguous)(state, self_)) {
    THCudaCheck(cudaMemsetAsync(THCTensor_(data)(state, self_),
                                0,
                                sizeof(real) * THCTensor_(nElement)(state, self_),
                                THCState_getCurrentStream(state)));
  } else {
    if (!THC_pointwiseApply1(
          state, self_,
          TensorFillOp<real>(ScalarConvert<int, real>::to(0)))) {
      THArgCheck(false, 1, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(zeros)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THAssert(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(zero)(state, r_);
}

THC_API void
THCTensor_(ones)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THAssert(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(fill)(state, r_, ScalarConvert<int, real>::to(1));
}

THC_API void
THCTensor_(reshape)(THCState *state, THCTensor *r_, THCTensor *t, THLongStorage *size)
{
  THAssert(THCTensor_(checkGPU)(state, 2, r_, t));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(copy)(state, r_, t);
}

ptrdiff_t
THCTensor_(numel)(THCState *state, THCTensor *t)
{
  return THCTensor_(nElement)(state, t);
}

void THCTensor_(cat)(THCState *state, THCTensor *result,
		     THCTensor *ta, THCTensor *tb, int dimension)
{
  THCTensor* inputs[2];
  inputs[0] = ta;
  inputs[1] = tb;
  THCTensor_(catArray)(state, result, inputs, 2, dimension);
}

void THCTensor_(catArray)(THCState *state, THCTensor *result,
			  THCTensor **inputs, int numInputs, int dimension)
{
  THLongStorage *size;
  int i, j, cohortMax;
  long offset;
  bool hasEmptyInput = false;

  // Even in the case where dimension is negative (i.e. when we want
  // to cat along the last dimension), this logic still works, as the
  // loop below will overwrite the value
  int maxDim = dimension + 1;

  // ldimension is the actual dimension we cat along (minus 1, for 0-based indexing)
  int ldimension = dimension;

  for (i = 0; i < numInputs; i++)
  {
    int inputDim = THCTensor_(nDimension)(state, inputs[i]);
    hasEmptyInput |= !inputDim;
    maxDim = THMax(maxDim, inputDim);
  }

  // In the event that the user specified -1 as the concat dimension, then
  // we want to pick the maxDim  as dimension to cat along (and thus maxDim - 1 as the
  // value due to 0-based indexing). If the maxDim is // 0 (i.e. we are catting all
  // empty tensors), then we set ldimension to be 0
  if (dimension + TH_INDEX_BASE == -1) {
    ldimension = maxDim ? (maxDim - 1) : 0;
  }

  THArgCheck(numInputs > 0, 3, "invalid number of inputs %d", numInputs);
  THArgCheck(ldimension >= 0, 4, "invalid dimension %d", dimension + TH_INDEX_BASE);

  size = THLongStorage_newWithSize(maxDim);
  for(i = 0; i < maxDim; i++)
  {
    // dimSize is either the size of the dim if it exists, either 1 if #dim > 0, otherwise 0
    long dimSize = i < THCTensor_(nDimension)(state, inputs[0])
                       ? THCTensor_(size)(state, inputs[0], i)
                       : THMin(THCTensor_(nDimension)(state, inputs[0]), 1);
    if (i == ldimension)
    {
      for (j = 1; j < numInputs; j++)
      {
        // accumulate the size over the dimension we want to cat on.
        // Empty tensors are allowed
        dimSize += i < THCTensor_(nDimension)(state, inputs[j])
                       ? THCTensor_(size)(state, inputs[j], i)
                       : THMin(THCTensor_(nDimension)(state, inputs[j]), 1);
      }
    }
    else
    {
      for (j = 1; j < numInputs; j++)
      {
        long sz = i < THCTensor_(nDimension)(state, inputs[j])
                      ? THCTensor_(size)(state, inputs[j], i)
                      : THMin(THCTensor_(nDimension)(state, inputs[j]), 1);

        // If it's a dimension we're not catting on
        // Then fail if sizes are different AND > 0
        if (dimSize != sz && dimSize && sz) {
          THLongStorage_free(size);
          THError("inconsistent tensor sizes");
        }
        else if(!dimSize)
        {
          dimSize = sz;
        }
      }
    }
    size->data[i] = dimSize;
  }

  THCTensor_(resize)(state, result, size, NULL);
  THLongStorage_free(size);

  // We parallelize the copy if all 6 conditions pass:
  //
  // 1. There is more than one input tensor
  // 2. No empty inputs
  // 3. The result tensor is 32-bit indexable
  // 4. The number of dimensions is <= 4
  // 5. All input tensors are contiguous (output tensor may be non-contig)
  // 6. All input tensors can use 32-bit indexing
  // 7. All input tensors are on the same device

  if (numInputs > 1 &&
      !hasEmptyInput &&
      THCTensor_(nDimension)(state, result) <= CAT_ARRAY_MAX_INPUT_DIMS &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, result) &&
      TensorUtils<THCTensor>::allContiguous(state, inputs, numInputs) &&
      TensorUtils<THCTensor>::all32BitIndexable(state, inputs, numInputs) &&
      TensorUtils<THCTensor>::allSameDevice(state, inputs, numInputs)) {

    // First, let's set up our kernel parameters. We start with a raw pointer to the storage
    // for the output Tensor.
    real *data = THCTensor_(data)(state, result);

    // Kernel Parameter
    CatArrInputTensor<real, unsigned int> stackInputs[CAT_ARRAY_BATCH_SIZE];
    CatArrInputTensor<real, unsigned int> *d_inputs;

    // Attempt to re-use stream's scratch space for the input metadata
    bool usedScratch = false;
    size_t tensorMetadataSize = sizeof(CatArrInputTensor<real, unsigned int>) * CAT_ARRAY_BATCH_SIZE;
    if (THCState_getCurrentDeviceScratchSpaceSize(state) > tensorMetadataSize) {
      void* space = THCState_getCurrentDeviceScratchSpace(state);
      if (space) {
        d_inputs = (CatArrInputTensor<real, unsigned int> *) space;
        usedScratch = true;
      }
    }
    if (!usedScratch) {
      // Fallback to allocating GPU memory
      THCudaCheck(THCudaMalloc(state, (void**) &d_inputs, tensorMetadataSize));
    }

    OutputTensorSizeStride<unsigned int, CAT_ARRAY_MAX_INPUT_DIMS> param;

    // Next, let's initialize the size, stride arrays for the output Tensor.
    for (i = 0; i < maxDim; ++i) {
      param.outputSize[i] = THCTensor_(size)(state, result, i);
      param.outputStride[i] = THCTensor_(stride)(state, result, i);
    }

    // Template Declarations for dim = 1, 2, 3, 4
#define HANDLE_CASE(DIMS) \
  CatArrayBatchedCopy<real, unsigned int, DIMS><<<applyGrid, applyBlock>>>(data, d_inputs, param, ldimension, param.outputStride[dimension]);

    // Now we loop
    offset = 0;
    for (i = 0; i < numInputs; i += CAT_ARRAY_BATCH_SIZE) {
      cohortMax = 0;
      for (j = 0; j < CAT_ARRAY_BATCH_SIZE && (i+j) < numInputs; ++j) {
        long dimSize = ldimension < THCTensor_(nDimension)(state, inputs[i+j])
          ? THCTensor_(size)(state, inputs[i+j], ldimension)
          : 1;

        stackInputs[j].input = THCTensor_(data)(state, inputs[i+j]);
        stackInputs[j].offset = offset;
        stackInputs[j].dimSize = dimSize;
        stackInputs[j].nElements = THCTensor_(nElement)(state, inputs[i+j]);
        cohortMax = cohortMax > stackInputs[j].nElements ? cohortMax : stackInputs[j].nElements;

        // update offset
        offset += dimSize;
      }
      cudaMemcpy(d_inputs, stackInputs, j * sizeof(CatArrInputTensor<real, unsigned int>), cudaMemcpyHostToDevice);

      // Next, let's consider how we set our kernel launch parameters.
      // We borrow from THCApply, which the kernel's internal indexing
      // is based on.
      dim3 applyBlock = getApplyBlock();

      // We also re-use the applyGrid - but note that we use the maximum number of
      // elements for a given tensor in this grouping to determine the count
      dim3 applyGrid;
      getApplyGrid(state, cohortMax, applyGrid);

      // Next, we set our grid's y component to be the number of tensors in
      // the batch. This will allow the kernel to determine which input
      // tensor it is responsible for copying
      applyGrid.y = j;

      switch (maxDim) {
        case 1:
          HANDLE_CASE(1);
          break;
        case 2:
          HANDLE_CASE(2);
          break;
        case 3:
          HANDLE_CASE(3);
          break;
        case 4:
          HANDLE_CASE(4);
          break;
      }
      THCudaCheck(cudaGetLastError());
    }
    if (!usedScratch) {
      THCudaCheck(THCudaFree(state, (void *)d_inputs));
    }
#undef HANDLE_CASE
  } else {
    offset = 0;
    for (j = 0; j < numInputs; j++)
    {
      // No reason to copy when input is empty
      if (!THCTensor_(nDimension)(state, inputs[j])) continue;

      long dimSize = ldimension < THCTensor_(nDimension)(state, inputs[j])
               ? THCTensor_(size)(state, inputs[j], ldimension)
               : 1;

      THCTensor *nt = THCTensor_(newWithTensor)(state, result);
      THCTensor_(narrow)(state, nt, NULL, ldimension, offset, dimSize);
      THCTensor_(copy)(state, nt, inputs[j]);
      THCTensor_(free)(state, nt);
      offset += dimSize;
    }
  }
}

void THCTensor_(nonzero)(THCState* state, THCudaLongTensor *tensor,
                          THCTensor *self)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self  ));
  THAssert(THCudaLongTensor_checkGPU(state, 1, tensor));


  using namespace thrust::placeholders;
  THCThrustAllocator thrustAlloc(state);
  self = THCTensor_(newContiguous)(state, self);
  thrust::device_ptr<real> self_data(THCTensor_(data)(state, self));

  int num_dim = THCTensor_(nDimension)(state, self);
  long N = THCTensor_(nElement)(state, self);

  THCudaLongTensor_resize2d(state, tensor, N, num_dim);
  tensor = THCudaLongTensor_newContiguous(state, tensor);
  thrust::device_ptr<long> tensor_data(THCudaLongTensor_data(state, tensor));

  thrust::counting_iterator<long> idxfirst(0);
  thrust::counting_iterator<long> idxlast = idxfirst + N;

  typedef thrust::device_ptr<long> Iter;
  strided_range<Iter> strided_tensor(tensor_data,
                                     tensor_data+N*num_dim, num_dim);

#if CUDA_VERSION >= 7000
  cudaStream_t stream = THCState_getCurrentStream(state);
#endif

  strided_range<Iter>::iterator dend = thrust::copy_if(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(stream),
#endif
    idxfirst,
    idxlast,
    self_data,
    strided_tensor.begin(),
    NonZeroOp<real>()
  );

  long num_nonzeros = thrust::distance(strided_tensor.begin(), dend);

  long div = 1;
  for (int dim = num_dim-1; dim >= 0; dim--) {
    strided_range<Iter> stride_dim(tensor_data+dim,
                                   tensor_data+N*num_dim, num_dim);
    thrust::transform(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(stream),
#endif
      strided_tensor.begin(),
      strided_tensor.end(),
      stride_dim.begin(),
      idx_functor(div, self->size[dim])
    );
    div *= self->size[dim];
  }

  THCudaLongTensor_resize2d(state, tensor, num_nonzeros, num_dim);

  THCTensor_(free)(state, self);
  THCudaLongTensor_free(state, tensor);

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(diag)(THCState *state, THCTensor *self_, THCTensor *src_, long k){
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  int nDimension = THCTensor_(nDimension)(state, src_);
  THArgCheck((nDimension == 2) || (nDimension == 1), 1, "expected a matrix or a vector");
  if (nDimension == 2) {
    long stride0 = THCTensor_(stride)(state, src_, 0);
    long stride1 = THCTensor_(stride)(state, src_, 1);
    long size0 = THCTensor_(size)(state, src_, 0);
    long size1 = THCTensor_(size)(state, src_, 1);
    long size = (k > 0) ? min((long long)size0, (long long)size1 - k) : min((long long)size0 + k, (long long)size1);
    THCTensor_(resize1d)(state, self_, size);
    long strideSelf = THCTensor_(stride)(state, self_, 0);
    const dim3 threads(min((long long)THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock, (long long)size));
    dim3 grid(min((long long)1024, (long long)THCCeilDiv(size, (long)threads.x)));
    long start = (k >= 0 ? k * stride1 : -k * stride0);
    THCTensor_copyFromDiagonal<real><<<grid, threads, 0, THCState_getCurrentStream(state)>>>
    (THCTensor_(data)(state, self_), THCTensor_(data)(state, src_), start, size, stride0 + stride1, strideSelf);
  } else {
    ptrdiff_t totalElements = THCTensor_(nElement)(state, src_);
    ptrdiff_t size = (k > 0) ? totalElements + k : totalElements - k;
    long strideSrc = THCTensor_(stride)(state, src_, 0);
    THCTensor_(resize2d)(state, self_, size, size);
    THCTensor_(zero)(state, self_);
    long stride0 = THCTensor_(stride)(state, self_, 0);
    long stride1 = THCTensor_(stride)(state, self_, 1);
    const dim3 threads(min((long long)THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock, (long long)size));
    dim3 grid(min((long long)1024, (long long)THCCeilDiv(size, (ptrdiff_t)threads.x)));
    ptrdiff_t start = (k >= 0 ? k * stride1 : -k * stride0);
    THCTensor_copyToDiagonal<real><<<grid, threads, 0, THCState_getCurrentStream(state)>>>
    (THCTensor_(data)(state, self_), THCTensor_(data)(state, src_), start, totalElements, stride0 + stride1, strideSrc);
  }
  THCudaCheck(cudaGetLastError());
}

accreal THCTensor_(trace)(THCState *state, THCTensor *src_) {
  THAssert(THCTensor_(checkGPU)(state, 1, src_));
  THArgCheck((src_->nDimension == 2), 1, "expected a matrix");
  THCTensor *diag = THCTensor_(new)(state);
  THCTensor_(diag)(state, diag, src_, 0);
  accreal trace = THCTensor_(sumall)(state, diag);
  THCTensor_(free)(state, diag);
  return trace;
}
#endif
