
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorRandom.cu"
#else

#define NUM_BLOCKS min((int)THCCeilDiv(size, (ptrdiff_t) BLOCK_SIZE), MAX_NUM_BLOCKS)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

THC_API void THCTensor_(uniform)(THCState* state, THCTensor *self_, double a, double b)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  real *data = THCTensor_(data)(state, self);

  generate_uniform<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, a, b);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(normal)(THCState* state, THCTensor *self_, double mean, double stdv)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  real *data = THCTensor_(data)(state, self);

  generate_normal<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, mean, stdv);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(normal_means)(THCState *state, THCTensor *self, THCTensor *means, double stddev) {
  THCTensor_(resizeAs)(state, self, means);
  THCTensor_(normal)(state, self, 0, stddev);
  THCTensor_(cadd)(state, self, self, ScalarConvert<int, real>::to(1), means);
}

THC_API void THCTensor_(normal_stddevs)(THCState *state, THCTensor *self, double mean, THCTensor *stddevs)
{
  THCTensor_(resizeAs)(state, self, stddevs);
  THCTensor_(normal)(state, self, 0, 1);
  THCTensor_(cmul)(state, self, self, stddevs);
  THCTensor_(add)(state, self, self, ScalarConvert<double, real>::to(mean));
}

THC_API void THCTensor_(normal_means_stddevs)(THCState *state, THCTensor *self, THCTensor *means, THCTensor *stddevs)
{
  THCTensor_(resizeAs)(state, self, means);
  THCTensor_(normal)(state, self, 0, 1);
  THCTensor_(cmul)(state, self, self, stddevs);
  THCTensor_(cadd)(state, self, self, ScalarConvert<int, real>::to(1), means);
}

THC_API void THCTensor_(logNormal)(THCState* state, THCTensor *self_, double mean, double stdv)
{

  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  real *data = THCTensor_(data)(state, self);

  generateLogNormal<real><<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, mean, stdv);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(exponential)(THCState* state, THCTensor *self_, double lambda)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  real *data = THCTensor_(data)(state, self);

  generate_exponential<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, lambda);

  THCTensor_(freeCopyTo)(state, self, self_);
};

THC_API void THCTensor_(cauchy)(THCState* state, THCTensor *self_, double median, double sigma)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  real *data = THCTensor_(data)(state, self);

  generate_cauchy<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, median, sigma);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(renormRows)(struct THCState* state,
                             THCTensor* t) {
  THAssert(THCTensor_(nDimension)(state, t) == 2);
  long rows = THCTensor_(size)(state, t, 0);
  long cols = THCTensor_(size)(state, t, 1);

  cudaDeviceProp* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);

  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;

  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(cols < maxThreads ? cols : maxThreads);

  renormRowsL1<real>
    <<<grid, block, block.x * sizeof(real),
    THCState_getCurrentStream(state)>>>(THCTensor_(data)(state, t),
                                        rows, cols);
}

THC_API void THCTensor_(multinomial)(struct THCState *state,
                                      THCudaLongTensor *self,
                                      THCTensor *prob_dist,
                                      int n_sample,
                                      int with_replacement)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, prob_dist));
  Generator* gen = THCRandom_getGenerator(state);

  int inputSize = THCTensor_(nDimension)(state, prob_dist);
  THArgCheck(inputSize > 0 && inputSize <= 2, 2,
             "prob_dist must be 1 or 2 dim");

  // Categories are in the innermost dimension
  long numDist =
    inputSize == 1 ? 1 : THCTensor_(size)(state, prob_dist, 0);
  long numCategoriesLong =
    inputSize == 1 ? THCTensor_(size)(state, prob_dist, 0) :
    THCTensor_(size)(state, prob_dist, 1);

  // Since the index tensor is float, numCategories cannot exceed max
  // float integer precision
  THArgCheck(numCategoriesLong <= FLOAT32_MAX_CONSECUTIVE_INT, 2,
             "number of categories cannot exceed 2^24");
  int numCategories = (int) numCategoriesLong;

  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");

  if (!with_replacement) {
    THArgCheck(n_sample <= numCategories, 2,
               "cannot sample n_sample > prob_dist:size(1) samples without "
               "replacement");
  }

  // It is possible that prob_dist is non-contiguous
  THCTensor* probDistContig =
    THCTensor_(newContiguous)(state, prob_dist);

  // Restructure data for 2d
  if (inputSize == 1) {
    THCTensor_(resize2d)(state, probDistContig, 1, numCategories);
  }

  THCudaLongTensor_resize2d(state, self, numDist, n_sample);

  // get current device properties
  cudaDeviceProp* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;
  int maxShared = props->sharedMemPerBlock;
  int requiredShared = (numCategories < maxThreads ? numCategories : maxThreads)
                                * (sizeof(real) * sizeof(accreal));
                                
  if (n_sample == 1 && maxShared >= requiredShared) {
    // Optimized allocation-free implementation
    // To exploit greater parallelism for the sampling, generate the
    // Uniform random samples in a separate kernel launch, into
    // temporarily allocated memory. The device RNG is thread-limited
    THCTensor *sampled = THCTensor_(newWithSize2d)(state, numDist, n_sample);
    THCTensor_(uniform)(state, sampled, 0.0, 1.0);

    dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
    dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);

    sampleMultinomialOnce<real, accreal>
      <<<grid, block,
         requiredShared,
         THCState_getCurrentStream(state)>>>(
      THCudaLongTensor_data(state, self),
      numDist,
      numCategories,
      THCTensor_(data)(state, sampled),
      THCTensor_(data)(state, probDistContig));
    THCTensor_(free)(state, sampled);
  } else {
    // Generic, slow implementation with memory allocations

    // For sampling without replacement, we modify the distribution
    // for subsequent samples in this space
    THCTensor* origDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, origDist, probDistContig);
    THCTensor_(copy)(state, origDist, probDistContig);

    THCTensor* normDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, normDist, probDistContig);

    THCTensor* prefixSum = THCTensor_(new)(state);

    // Renorm along rows
    THCTensor_(copy)(state, normDist, origDist);
    THCTensor_(renormRows)(state, normDist);

    // Prefix sum along rows
    THCTensor_(cumsum)(state, prefixSum, normDist, 1);

    if (with_replacement) {
      // Sample with replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from one
      // distribution concurrently.
      dim3 grid(numDist < MAX_NUM_BLOCKS ? numDist : MAX_NUM_BLOCKS);

      
      //Create the matrix of uniformly sampled numbers
      THCTensor *uniform_idx = THCTensor_(newWithSize1d)(state, n_sample);
      THCTensor_(uniform)(state, uniform_idx, 0, 1);
      
      
      sampleMultinomialWithReplacement
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
	   THCTensor_(data)(state, uniform_idx),
	   n_sample,
	   THCudaLongTensor_data(state, self),
	   numDist, numCategories,
	   THCTensor_(data)(state, prefixSum));
      
      THCTensor_(free)(state, uniform_idx);
    } else {
      // Sample without replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from a different
      // distribution concurrently.
      ptrdiff_t numBlocks = THCCeilDiv(numDist, 4L);
      dim3 grid(numBlocks < MAX_NUM_BLOCKS ? numBlocks : MAX_NUM_BLOCKS);

      for (int sample = 0; sample < n_sample; ++sample) {
        if (sample > 0) {
          // Update probabilities
          // Renorm along rows
          THCTensor_(copy)(state, normDist, origDist);
          THCTensor_(renormRows)(state, normDist);

          // Prefix sum along rows
          THCTensor_(cumsum)(state, prefixSum, normDist, 1);
        }

        // The kernel can only draw one sample before we have to
        // recalculate our distribution
        sampleMultinomialWithoutReplacement
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
	     gen->gen_states,
	     n_sample,	     
	     sample,	     
	     THCudaLongTensor_data(state, self),
	     numDist, numCategories,
	     THCTensor_(data)(state, origDist),
	     THCTensor_(data)(state, prefixSum));
      }
    }

    THCTensor_(free)(state, prefixSum);
    THCTensor_(free)(state, normDist);
    THCTensor_(free)(state, origDist);
  }

  // Revert data restructuring based on input sizes
  if (inputSize == 1) {
    THCudaLongTensor_resize1d(state, self, n_sample);

    // Unfortunately, if prob_dist is contiguous already,
    // newContiguous is not a private copy, so we have to restructure
    // this too, so as to not affect prob_dist
    THCTensor_(resize1d)(state, probDistContig, numCategories);
  }

  THCTensor_(free)(state, probDistContig);
}

THC_API void THCTensor_(multinomialAliasSetup)(THCState *state, THCTensor *_probs, THCudaLongTensor *_J, THCTensor *_q){
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  THAssert(THCTensor_(isContiguous)(state, _probs));
  long inputsize = THCTensor_(nElement)(state, _probs);
  THCudaLongTensor *smaller = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *smaller_short = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger_short = THCudaLongTensor_newWithSize1d(state, inputsize);
  
  THCudaLongTensor_resize1d(state, _J, inputsize);
  THCTensor_(resize1d)(state, _q, inputsize);
  
  real one = ScalarConvert<long, real>::to(1);
  int inputBlockDim = THCCeilDiv((int)inputsize + BLOCK_SIZE - 1, BLOCK_SIZE);
  aliasMultinomialFilter
    <<<inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state) >>>(
								     THCTensor_(data)(state, _q),
								     THCTensor_(data)(state, _probs),
								     THCudaLongTensor_data(state, smaller),
								     THCudaLongTensor_data(state, larger),
								     THCudaLongTensor_data(state, _J),
								     THCudaLongTensor_data(state, smaller_short),
								     THCudaLongTensor_data(state, larger_short),
								     one, inputsize
								     );
  
  THCudaLongTensor_nonzero(state, smaller_short, smaller);
  THCudaLongTensor_nonzero(state, larger_short, larger);
  int h_large_c = THCudaLongTensor_nElement(state, larger_short);
  THCudaLongTensor_resize1d(state, smaller_short, inputsize);
  THCudaLongTensor_resize1d(state, larger_short, inputsize);
  aliasMultinomialSetup
    <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
						    THCudaLongTensor_data(state, _J),
						    THCTensor_(data)(state, _q),
						    inputsize,
						    THCudaLongTensor_data(state, smaller_short),
						    THCudaLongTensor_data(state, larger_short),
						    inputsize - h_large_c, h_large_c
						    );
  real q_max = THCTensor_(maxall)(state, _q);
  condDiv<<<
    inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
								      THCTensor_(data)(state, _q),
								      THCudaLongTensor_data(state, _J),
								      inputsize, q_max
								      );
  
  THCudaLongTensor_free(state, smaller);
  THCudaLongTensor_free(state, larger);
  THCudaLongTensor_free(state, smaller_short);
  THCudaLongTensor_free(state, larger_short);
}

THC_API void THCTensor_(multinomialAliasDraw)(THCState *state, THCudaLongTensor *self, THCudaLongTensor *_J, THCTensor *_q){
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  Generator* gen = THCRandom_getGenerator(state);
  long K = THCudaLongTensor_nElement(state, _J);
  long output_nelem = THCudaLongTensor_nElement(state, self);
  ptrdiff_t size = THCudaLongTensor_nElement(state, self);

  THCTensor *uniform = THCTensor_(newWithSize1d)(state, output_nelem);
  THCTensor *bernoulli = THCTensor_(newWithSize1d)(state, output_nelem);

  THCTensor_(uniform)(state, uniform, 0, K);
  THCTensor_(uniform)(state, bernoulli, 0, 1);

  multinomialAliasDrawKernel
    <<<THCCeilDiv((int)output_nelem+BLOCK_SIZE-1, BLOCK_SIZE), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
				  size,
				  THCudaLongTensor_data(state, self),
				  THCudaLongTensor_data(state, _J),
				  THCTensor_(data)(state, _q),
				  K,
				  THCTensor_(data)(state, uniform),
				  THCTensor_(data)(state, bernoulli)
				  );
}

THC_API void THCTensor_(rand)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(uniform)(state, r_, 0, 1);
}

void THCTensor_(randn)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(normal)(state, r_, 0, 1);
}

#endif

#if defined(THC_REAL_IS_DOUBLE)
GENERATE_KERNEL1(generate_bernoulli, double, double p, double, curand_uniform_double, x <= p)
#else
GENERATE_KERNEL1(generate_bernoulli, real, double p, float, curand_uniform, (ScalarConvert<bool, real>::to(x <= p)))
#endif

THC_API void THCTensor_(bernoulli)(THCState* state, THCTensor *self_, double p)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);
  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  real *data = THCTensor_(data)(state, self);

  generate_bernoulli<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, p);

  THCTensor_(freeCopyTo)(state, self, self_);
};

#define DEFINE_BERNOULLI_TENSOR(NAME, PROB_TYPE, PROB_DATA_TYPE)               \
THC_API void THCTensor_(NAME)(THCState* state,                                 \
        THCTensor *self_, PROB_TYPE *probs_)                                   \
{                                                                              \
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, probs_));                     \
  Generator* gen = THCRandom_getGenerator(state);                              \
  THCTensor *self = THCTensor_(newContiguous)(state, self_);                   \
  PROB_TYPE *probs = PROB_TYPE##_newContiguous(state, probs_);                 \
  ptrdiff_t size = THCTensor_(nElement)(state, self);                          \
  ptrdiff_t prob_size = PROB_TYPE##_nElement(state, probs);                    \
  real *result_data = THCTensor_(data)(state, self);                           \
  PROB_DATA_TYPE *probs_data = PROB_TYPE##_data(state, probs);                 \
                                                                               \
  THArgCheck(size == prob_size, 3, "inconsistent tensor size");                \
                                                                               \
  generate_bernoulli_tensor<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>( \
      gen->gen_states, size, result_data, probs_data);                         \
                                                                               \
  PROB_TYPE##_free(state, probs);                                              \
  THCTensor_(freeCopyTo)(state, self, self_);                                  \
}

DEFINE_BERNOULLI_TENSOR(bernoulli_FloatTensor, THCudaTensor, float)
DEFINE_BERNOULLI_TENSOR(bernoulli_DoubleTensor, THCudaDoubleTensor, double)

#if defined(THC_REAL_IS_DOUBLE)

GENERATE_KERNEL1(generate_geometric, double, double p, double, curand_uniform_double, ceil(log(x) / log(1-p)))
#else
GENERATE_KERNEL1(generate_geometric, real, double p, float, curand_uniform, (ScalarConvert<float, real>::to(ceilf(logf(x) / log(1-p)))))
#endif

THC_API void THCTensor_(geometric)(THCState* state, THCTensor *self_, double p)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  real *data = THCTensor_(data)(state, self);

  generate_geometric<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, p);

  THCTensor_(freeCopyTo)(state, self, self_);
};
#undef NUM_BLOCKS

#endif
