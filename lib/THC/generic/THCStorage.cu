
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCStorage.cu"
#else

void THCStorage_(fill)(THCState *state, THCStorage *self, hostreal _value)
{
  thrust::device_ptr<real> self_data(self->data);
  real value = hostrealToReal(_value);
  thrust::fill(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+self->size, value);
}

void THCStorage_(resize)(THCState *state, THCStorage *self, long size)
{
  THArgCheck(size >= 0, 2, "invalid size");

  if(!(self->flag & TH_STORAGE_RESIZABLE))
    THError("Trying to resize storage that is not resizable");

  if(size == 0)
  {
    if(self->flag & TH_STORAGE_FREEMEM) {
      if(state->showAllocations) printf("Free CUDA storage %li bytes\n",
        self->size * sizeof(real));
      THCudaCheck(THCudaFree(state, self->data));
      THCHeapUpdate(state, -self->size * sizeof(real));
    }
    self->data = NULL;
    self->size = 0;
  }
  else
  {
    real *data = NULL;
    // update heap *before* attempting malloc, to free space for the malloc
    THCHeapUpdate(state, size * sizeof(real));
    if(state->showAllocations) printf("Alloc CUDA storage %li bytes\n",
      size * sizeof(real));
    cudaError_t err = THCudaMalloc(state, (void**)(&data), size * sizeof(real));
    if(err != cudaSuccess) {
      THCHeapUpdate(state, -size * sizeof(real));
    }
    THCudaCheck(err);

    if (self->data) {
      THCudaCheck(cudaMemcpyAsync(data,
                                  self->data,
                                  THMin(self->size, size) * sizeof(real),
                                  cudaMemcpyDeviceToDevice,
                                  THCState_getCurrentStream(state)));
      if(state->showAllocations) printf("Free CUDA storage %li bytes\n",
        self->size * sizeof(real));
      THCudaCheck(THCudaFree(state, self->data));
      THCHeapUpdate(state, -self->size * sizeof(real));
    }

    self->data = data;
    self->size = size;
  }
}
#endif
