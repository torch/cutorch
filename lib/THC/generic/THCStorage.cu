
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCStorage.cu"
#else

void THCStorage_(fill)(THCState *state, THCStorage *self, real value)
{
  thrust::device_ptr<real> self_data(self->data);
  thrust::fill(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+self->size, value);
}

void THCStorage_(resize)(THCState *state, THCStorage *self, long size)
{
  THArgCheck(size >= 0, 2, "invalid size");

  if(!(self->flag & TH_STORAGE_RESIZABLE))
    THError("Trying to resize storage that is not resizable");

  if(size == 0)
  {
    if(self->flag & TH_STORAGE_FREEMEM) {
      THMemoryCheck(THCudaFree(state, self->data));
      THCHeapUpdate(state, -self->size * sizeof(real));
    }
    self->data = NULL;
    self->size = 0;
  }
  else
  {
    real *data = NULL;
    // update heap *before* attempting malloc, to free space for the malloc
    THCHeapUpdate(state, size * sizeof(real));
    memoryStatus_t err = THCudaMalloc(state, (void**)(&data), size * sizeof(real));
    if(err != 0) {
      THCHeapUpdate(state, -size * sizeof(real));
    }
    THMemoryCheck(err);

    if (self->data) {
      THCudaCheck(cudaMemcpyAsync(data,
                                  self->data,
                                  THMin(self->size, size) * sizeof(real),
                                  cudaMemcpyDeviceToDevice,
                                  THCState_getCurrentStream(state)));
      THMemoryCheck(THCudaFree(state, self->data));
      THCHeapUpdate(state, -self->size * sizeof(real));
    }

    self->data = data;
    self->size = size;
  }
}
#endif
