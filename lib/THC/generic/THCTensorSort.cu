
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorSort.cu"
#else
# ifndef THC_GENERIC_NO_MATH
// In alignment with default sort on a c++ map, this function
// will permute key and value tensors identically, and
// in such a way that the 'key' tensor is ordered numerically
THC_API void THCTensor_(sortKeyValueInplace)(THCState* state,
                                           THCTensor* key,
                                           THCudaLongTensor* value,
                                           int dim, bool dir) {
  THLongStorage *valueSize = THCudaLongTensor_newSizeOf(state, value);
  THArgCheck(THCTensor_(isSize)(state, key, valueSize), 2,
             "Key tensor must have same size as value tensor");
  THLongStorage_free(valueSize);
  long dims = THCudaLongTensor_nDimension(state, value);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 3, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, key);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  ptrdiff_t inElements = THCTensor_(nElement)(state, key);
  long keySliceSize = THCTensor_(size)(state, key, dim);
  ptrdiff_t keySlices = inElements / keySliceSize;

  if (THCTensor_(nDimension)(state, key) == 0) {
    // Zero-dim tensor; do nothing
    return;
  }

  // The amount of shared memory and block size is based on
  // 2^ceil(lg(n)); we choose that sorting implementation for a given
  // size.
  long ceilPowerOf2 = nextHighestPowerOf2(keySliceSize);

  // FIXME: We'd have to find some other trick with Thrust to perform a
  // vectorized (key, value) sort by slice segment
  if (ceilPowerOf2 > 2048) {
    THError("sortKeyValueInplace only works for sizes <= 2048 at present");
  }

  // The grid is based on the number of independent slices that we
  // have to sort; one block per slice
  dim3 grid;
  if (!THC_getGridFromTiles(keySlices, grid)) {
    THError("Slice to sort is too large");
  }

#define HANDLE_CASE(TYPE, A, SIZE)                                      \
  do {                                                                  \
    int blockSize = SIZE / 2;                                           \
    if (blockSize < 1) {                                                \
      blockSize = 1;                                                    \
    }                                                                   \
                                                                        \
    dim3 block(blockSize);                                              \
                                                                        \
    if (dir) {                                                          \
      bitonicSortKVInPlace<real, long, A, -1, GTComp<real>, TYPE, SIZE> \
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(         \
          keyInfo,                                                      \
          keySlices,                                                    \
          (TYPE) keySliceSize,                                          \
          (TYPE) keyInfo.strides[collapseKeyDim],                       \
          valueInfo,                                                    \
          (TYPE) valueInfo.strides[collapseValueDim],                   \
          GTComp<real>());                                              \
    } else {                                                            \
      bitonicSortKVInPlace<real, long, A, -1, LTComp<real>, TYPE, SIZE> \
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(         \
          keyInfo,                                                      \
          keySlices,                                                    \
          (TYPE) keySliceSize,                                          \
          (TYPE) keyInfo.strides[collapseKeyDim],                       \
          valueInfo,                                                    \
          (TYPE) valueInfo.strides[collapseValueDim],                   \
          LTComp<real>());                                              \
    }                                                                   \
  } while (0)

#define HANDLE_SORT_CASE(TYPE, A)                       \
  {                                                     \
    switch (ceilPowerOf2) {                             \
      case 2048:                                        \
      HANDLE_CASE(TYPE, A, 2048);                       \
      break;                                            \
      case 1024:                                        \
      case 512:                                         \
      case 256:                                         \
      HANDLE_CASE(TYPE, A, 1024);                       \
      break;                                            \
      case 128:                                         \
      case 64:                                          \
      HANDLE_CASE(TYPE, A, 128);                        \
      break;                                            \
      case 32:                                          \
      case 16:                                          \
      case 8:                                           \
      case 4:                                           \
      case 2:                                           \
      HANDLE_CASE(TYPE, A, 32);                         \
      break;                                            \
      case 1:                                           \
      /* Nothing to do, data already sorted */          \
      break;                                            \
      default:                                          \
      assert(false);                                    \
    }                                                   \
  }

  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, key)) {
    TensorInfo<real, unsigned int> keyInfo =
      getTensorInfo<THCTensor, unsigned int>(state, key);
    keyInfo.reduceDim(dim);
    int collapseKeyDim = keyInfo.collapseDims(dim);

    TensorInfo<long, unsigned int> valueInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, value);
    valueInfo.reduceDim(dim);
    int collapseValueDim = valueInfo.collapseDims(dim);

    if (keyInfo.isContiguous()) {
      HANDLE_SORT_CASE(unsigned int, -2);
    } else {
      switch (keyInfo.dims) {
        case 2:
          HANDLE_SORT_CASE(unsigned int, 2);
          break;
        default:
          HANDLE_SORT_CASE(unsigned int, -1);
          break;
      }
    }
  } else {
    TensorInfo<real, unsigned long> keyInfo =
      getTensorInfo<THCTensor, unsigned long>(state, key);
    keyInfo.reduceDim(dim);
    int collapseKeyDim = keyInfo.collapseDims(dim);

    TensorInfo<long, unsigned long> valueInfo =
      getTensorInfo<THCudaLongTensor, unsigned long>(state, value);
    valueInfo.reduceDim(dim);
    int collapseValueDim = valueInfo.collapseDims(dim);

    // long case is rare, just instantiate the generic version
    HANDLE_SORT_CASE(unsigned long, -1);
  }
#undef HANDLE_CASE
#undef HANDLE_SORT_CASE
#undef HANDLE_A_CASE

  THCudaCheck(cudaGetLastError());
}

void sortViaThrust(THCState* state,
                   THCTensor* sorted,
                   THCudaLongTensor* indices,
                   THCTensor* input,
                   int dim, bool dir) {
  long nDims = THCTensor_(nDimension)(state, input);

  ptrdiff_t totalElements = THCTensor_(nElement)(state, input);
  long sliceSize = THCTensor_(size)(state, input, dim);
  long sliceStride = THCTensor_(stride)(state, input, dim);

  // We perform a vectorized segmented sort in Thrust.
  // Say we are sorting a (2, 3) tensor. We have in flattened form:
  // values 0.4 1.2 5.3 6.2 1.3 2.3
  // indices  0   1   2   3   4   5
  // where indices is a global index (across all slices)

  // First we sort by values, globally:
  // values 6.2 5.3 2.3 1.2 1.3 0.4
  // indices  3   2   5   1   4   0

  // Then we stable sort by segment, which is index / 3:
  // values 5.3 1.2 0.4 6.2 2.3 1.3
  // indices  2   1   0   3   5   4

  // Then we translate the global index to a per-slice Lua index
  // (index % 3) + 1:
  // values 5.3 1.2 0.4 6.2 2.3 1.3
  // indices  3   2   1   1   3   2

  // This method can only work if the slice we are sorting (`dim`) is
  // innermost, and both values and indices are contiguous. We do this
  // by re-arranging the input into this form as needed, which will
  // unfortunately allocate memory if the request is not in this form.
  // Vectorized sort is slower than iterated sort if the number of
  // slices is small (since we're sorting twice, instead of invoking a
  // smaller sort `numSlices` times), but the Thrust sort
  // implementation here is a catch-all, so we're not looking for
  // efficiency, but instead correctness.
  THCTensor_(copy)(state, sorted, input);
  THCTensor* trKeys = THCTensor_(newWithTensor)(state, sorted);
  THCudaLongTensor* trIndices = THCudaLongTensor_newWithTensor(state, indices);

  // Transpose dim to innermost
  if (dim != nDims - 1) {
    THCTensor_(transpose)(state, trKeys, NULL, dim, nDims - 1);
    THCudaLongTensor_transpose(state, trIndices, NULL, dim, nDims - 1);
  }

  // Thrust must operate on a contiguous layout
  THCTensor* trContigKey = THCTensor_(newContiguous)(state, trKeys);
  THCudaLongTensor* trContigIndices = THCudaLongTensor_newContiguous(state, trIndices);

  THCTensor_(free)(state, trKeys);
  THCudaLongTensor_free(state, trIndices);

  THCThrustAllocator thrustAlloc(state);

  thrust::device_ptr<real> keyIter(THCTensor_(data)(state, trContigKey));

  // Since we are composing a global index across all segments rather
  // than a per-segment index, we treat the memory as int so we don't
  // have problems sorting slices < 2^24 but where the entire tensor
  // has more than 2^24 elements
  thrust::device_ptr<long>
    indexIter((long*) THCudaLongTensor_data(state, trContigIndices));

  // Fill the indices with a global index across all slices
  thrust::counting_iterator<long> countIter(0);

  thrust::copy(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    countIter, countIter + totalElements, indexIter);

  // First, we sort globally (across all slices) according to key
  // (the values we're sorting)
  if (dir) {
    thrust::stable_sort_by_key(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      keyIter, keyIter + totalElements, indexIter, ThrustGTOp<real>());
  } else {
    thrust::stable_sort_by_key(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      keyIter, keyIter + totalElements, indexIter, ThrustLTOp<real>());
  }

  // Then, re-sort according to slice that each index is
  // in. This completes the segment sort in Thrust, since we're
  // stably sorting here, preserving the relative order of values
  // per each slice
  thrust::stable_sort_by_key(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    indexIter, indexIter + totalElements, keyIter,
    SliceComp(sliceSize));

  // Translate the global integer 0-based index to a per-slice real
  // Lua index
  thrust::for_each(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    indexIter, indexIter + totalElements,
    GlobalIndexToPerSliceIndex(sliceSize));

  // Reverse the transposition as needed
  if (dim != nDims - 1) {
    THCTensor_(transpose)(state, trContigKey, NULL, dim, nDims - 1);
    THCudaLongTensor_transpose(state, trContigIndices, NULL, dim, nDims - 1);
  }

  // Then copy back to the expected output
  THCTensor_(freeCopyTo)(state, trContigKey, sorted);
  THCudaLongTensor_freeCopyTo(state, trContigIndices, indices);
}

THC_API void THCTensor_(sort)(THCState* state,
                               THCTensor *sorted,
                               THCudaLongTensor *indices,
                               THCTensor *input,
                               int dim, int order) {
  THAssert(THCTensor_(checkGPU)(state, 2, sorted, input));
  THAssert(THCudaLongTensor_checkGPU(state, 1, indices));
  long dims = THCTensor_(nDimension)(state, sorted);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, input);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 3, CUTORCH_DIM_WARNING);

  // Make sure sufficient output space is allocated
  THCTensor_(resizeAs)(state, sorted, input);
  THLongStorage *inputSize = THCTensor_(newSizeOf)(state, input);
  THCudaLongTensor_resize(state, indices, inputSize, NULL);
  THLongStorage_free(inputSize);

  // How large are the slices that we are sorting?
  long sliceSize = THCTensor_(size)(state, input, dim);

  // Workaround:
  // CUDA 8 uses more shared memory than 7.5 for bitonicSortKVInPlace,
  // and so for the double word types,
  // we get "too many resources requested for launch" in the 2048 case
#if CUDA_VERSION >= 8000
#if defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_LONG)
  int maxSliceSize = 1024;
#else
  int maxSliceSize = 2048;
#endif
#else
  int maxSliceSize = 2048;
#endif

  if (sliceSize <= maxSliceSize) {
    // Fill `indices` (the values) with the
    // slice-relative index.
    THCudaLongTensor_fillSliceWithIndex(state, indices, dim);

    // We sort k/v pairs in-place; copy unsorted input to output
    THCTensor_(copy)(state, sorted, input);

    // Sort using our in-place k/v kernel that supports arbitrary
    // layout
    THCTensor_(sortKeyValueInplace)(state, sorted, indices, dim, order);
  } else {
    // Otherwise, fall back upon Thrust, which handles all other cases
    // (potentially slowly, with extra copies/memory allocations)
    sortViaThrust(state, sorted, indices, input, dim, (bool) order);
  }

  THCudaCheck(cudaGetLastError());
}
# endif /* THC_GENERIC_NO_MATH */
#endif
