
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMode.cu"
#else

THC_API void THCTensor_(calculateMode)(THCState *state,
                                        THCTensor *values,
                                        THCudaLongTensor *indices,
                                        THCTensor *input,
                                        THCudaLongStorage *sortBuffer,
                                        int dimension,
                                        THLongStorage *position) {
  THAssert(THCTensor_(isContiguous)(state, input));

  // Because the input is contiguous, we want to get a reference to the
  // location of the buffer at the innermost dimension that we are going
  // to calculate the mode for --> we do this by manually doing the stride
  // calculations to get an offset
  real *data = THCTensor_(data)(state, input);
  for (int i = 0; i < THLongStorage_size(position); ++i) {
    data += THLongStorage_data(position)[i] * THCTensor_(stride)(state, input, i);
  }

  long nElement = THCTensor_(size)(state, input, THCTensor_(nDimension)(state, input) - 1);
  THCThrustAllocator thrustAlloc(state);

  // Wrap input data, sortBuffer, in Thrust device vectors
  thrust::device_ptr<real> vecPtr = thrust::device_pointer_cast(data);
  thrust::device_vector<real> iter(vecPtr, vecPtr + nElement);
  thrust::device_ptr<long> sbPtr = thrust::device_pointer_cast(THCudaLongStorage_data(state, sortBuffer));
  thrust::device_vector<long> seq(sbPtr, sbPtr + nElement);

  // Fill sortBuffer with [0, 1, 2, ... nElement - 1]
  thrust::sequence(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    seq.begin(), seq.end());

  // Sort the input data. The original indices of the data are stored in seq
  thrust::sort_by_key(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    iter.begin(), iter.end(), seq.begin()
#if defined(THC_REAL_IS_HALF)
    , ThrustHalfLess()
#endif
  );

  // Count # of unique elements via an inner product between adjacent elements.
  // Add 1 if two neighboring element are not equal.
  int unique = 1 + thrust::inner_product(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    iter.begin(), iter.end() - 1, iter.begin() + 1, 0, thrust::plus<int>(),
#if defined(THC_REAL_IS_HALF)
    ThrustHalfNotEqualTo()
#else
    thrust::not_equal_to<real>()
#endif
  );

  // Count frequency of each element
  thrust::device_vector<real> keys(unique);
  thrust::device_vector<int> counts(unique);
  thrust::reduce_by_key(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    iter.begin(), iter.end(),
    thrust::constant_iterator<int>(1), keys.begin(), counts.begin()
#if defined(THC_REAL_IS_HALF)
    , ThrustHalfEqualTo()
#endif
  );

  // Find index of maximum count
  thrust::device_vector<int>::iterator it = thrust::max_element(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    counts.begin(), counts.end());
  real mode = keys[it - counts.begin()];

  // Find first index within which it occurs
#if defined(THC_REAL_IS_HALF)
  thrust::device_vector<real>::iterator positionIter = thrust::find_if(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    iter.begin(), iter.end(), ThrustHalfEqualToPredicate(mode));
#else
  thrust::device_vector<real>::iterator positionIter = thrust::find(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    iter.begin(), iter.end(), mode);
#endif

  THAssert(positionIter != iter.end());
  long index = TH_INDEX_BASE + seq[positionIter - iter.begin()];

  // Place mode, index in output
  ptrdiff_t valuesOffset = THCTensor_(storageOffset)(state, values);
  long indicesOffset = THCudaLongTensor_storageOffset(state, indices);

  for (int i = 0; i < THLongStorage_size(position); ++i) {
    long pos = THLongStorage_data(position)[i];
    valuesOffset += THCTensor_(stride)(state, values, i) * pos;
    indicesOffset += THCudaLongTensor_stride(state, indices, i) * pos;
  }
  THCStorage_(set)(state, THCTensor_(storage)(state, values), valuesOffset, mode);
  THCudaLongStorage_set(state, THCudaLongTensor_storage(state, indices), indicesOffset, index);
}

// this probably could be a loop, not a recursive algorithm
THC_API void THCTensor_(dimApplyMode)(THCState *state,
                               THCTensor *values,
                               THCudaLongTensor *indices,
                               THCTensor *input,
                               THCudaLongStorage *sortBuffer,
                               int dimension,
                               THLongStorage *position,
                               int curDim) {
  long ndim = THCTensor_(nDimension)(state, input);

  // Because we have transposed the Tensor, the data for the dimension we are mode'ing along
  // is always in the innermost dimension
  if (curDim == ndim - 1) {
    THCTensor_(calculateMode)(state, values, indices, input, sortBuffer, dimension, position);
  } else {
    // Loop through the values and recurse
    for (int i = 0; i < THCTensor_(size)(state, input, curDim); ++i) {
      position->data[curDim] = i;
      THCTensor_(dimApplyMode)(state, values, indices, input, sortBuffer, dimension, position, curDim + 1);
    }
  }
}

#define MAX_GRID_SIZE  65535
#define MAX_BLOCK_SIZE 1024

THC_API void THCTensor_(mode)(THCState *state,
                              THCTensor *values,
                              THCudaLongTensor *indices,
                              THCTensor *input,
                              int dimension,
                              int keepdim) {
  THLongStorage *dim;
  THCTensor *transposed, *contiguous, *valuesTransposed;
  THLongStorage *position;
  THCudaLongStorage *sortBuffer;
  THCudaLongTensor *indicesTransposed;
  long ndim, sliceSize, slices;


  THAssert(THCTensor_(checkGPU)(state, 1, values));

  // Verify they are asking for a valid dimension
  ndim = THCTensor_(nDimension)(state, input);
  THArgCheck(dimension >= 0 && dimension < ndim, 4, "Dimension of out bounds");

  sliceSize = THCTensor_(size)(state, input, dimension);
  slices = THCTensor_(nElement)(state, input) / sliceSize;

  // Resize output value, index Tensors to appropriate sizes (i.e. the same as
  // the input Tensor, except at dim=dimension, the size is 1)
  dim = THCTensor_(newSizeOf)(state, input);
  THLongStorage_set(dim, dimension, 1);
  THCTensor_(resize)(state, values, dim, NULL);
  THCudaLongTensor_resize(state, indices, dim, NULL);
  THLongStorage_free(dim);

  // If sliceSize is 1, copy input to values and set indices
  if (sliceSize == 1) {
    THCTensor_(copy)(state, values, input);
    THCudaLongTensor_fill(state, indices, TH_INDEX_BASE);
    return;
  }

  // Requirements for fused kernel implementation:
  //
  // 1. sliceSize <= 2 * max threads per block
  // 2. uses one block per slice, so number of slices must be less than the maximum number of blocks for
  // a kernel launch
  // 3. Can use 32-bit index math for indexing (mainly just for implementation conciseness, could be changed)
  if (sliceSize <= MAX_BLOCK_SIZE * 2 &&
      slices <= MAX_GRID_SIZE &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, input)) {
    // Beginning our optimized implementation. First thing we want to do is to transpose
    // the input Tensor along the sort dimension, and then make it contiguous
    transposed = THCTensor_(newTranspose)(state, input, dimension, ndim - 1);
    contiguous = THCTensor_(newContiguous)(state, transposed);

    // We also need to view the values and indices Tensors as transposed in order to
    // properly determine the offset into the underlying storage in which to place the
    // mode and index for a particular set of dimension values
    valuesTransposed = THCTensor_(newTranspose)(state, values, dimension, ndim-1);
    indicesTransposed = THCudaLongTensor_newTranspose(state, indices, dimension, ndim-1);

    // Set-up TensorInfo structs for passing to kernel
    TensorInfo<real, unsigned int> tiValues = getTensorInfo<THCTensor, unsigned int>(state, valuesTransposed);
    TensorInfo<long, unsigned int> tiIndices = getTensorInfo<THCudaLongTensor, unsigned int>(state, indicesTransposed);

    // The number of blocks is the number of slices that we need to calculate the mode for. Each block
    // is responsible for computing a single mode
    dim3 grid;
    THC_getGridFromTiles(slices, grid);

    // The blocksize is two elements per thread, rounded up to the nearest power of 2
    long ceilPowerOf2 = nextHighestPowerOf2(sliceSize);

    // Macro that calls kernel --> note that we set the block dimensions here, and
    // the amount of shared memory
  #define HANDLE_MODE(SIZE) \
  { \
    dim3 blockSize(SIZE / 2); \
\
    int memsize = modeSmemSize<real, SIZE>(state); \
    computeMode<real, SIZE> \
      <<<grid, blockSize, memsize, THCState_getCurrentStream(state)>>>( \
        THCTensor_(data)(state, contiguous), tiValues, tiIndices, sliceSize); \
  }

    // Tradeoff between compilation time and the number of specializations. Ideally we would have
    // one HANDLE_MODE for each power of 2
    switch(ceilPowerOf2) {
      case 2048:
        HANDLE_MODE(2048)
        break;
      case 1024:
      case 512:
      case 256:
        HANDLE_MODE(1024)
        break;
      case 128:
        HANDLE_MODE(128)
        break;
      case 64:
      case 32:
      case 16:
      case 8:
      case 4:
      case 2:
        HANDLE_MODE(64) // block size should be at least 1 full warp
        break;
      case 1:
      default:
        assert(false);
    }
    THCudaCheck(cudaGetLastError());

    THCTensor_(free)(state, transposed);
    THCTensor_(free)(state, contiguous);
    THCTensor_(free)(state, valuesTransposed);
    THCudaLongTensor_free(state, indicesTransposed);
  } else {
    // Beginning our naive implementation: We don't want to mutate the input Tensor, but
    // we need to be able to sort the inputs along the dimension in order to calculate the
    // mode. Additionally, its ideal if the data along the dimension is contiguous. So
    // we transpose the dimension with the innermost dimension and make a new contiguous
    // version that we can use.
    transposed = THCTensor_(newClone)(state, input);
    THCTensor_(transpose)(state, transposed, NULL, dimension, ndim - 1);
    contiguous = THCTensor_(newContiguous)(state, transposed);
    THCTensor_(free)(state, transposed);

    // We also need to view the values and indices Tensors as transposed in order to
    // properly determine the offset into the underlying storage in which to place the
    // mode and index for a particular set of dimension values
    valuesTransposed = THCTensor_(newTranspose)(state, values, dimension, ndim - 1);
    indicesTransposed = THCudaLongTensor_newTranspose(state, indices, dimension, ndim - 1);

    // Position is a Storage that will store the dimension values we are processing
    position = THLongStorage_newWithSize(ndim - 1);

    // Sort Buffer is a Storage that will be used in the internal sort required to calculate
    // the mode efficiently
    sortBuffer = THCudaLongStorage_newWithSize(state, sliceSize);

    // Call mode
    THCTensor_(dimApplyMode)(state, valuesTransposed, indicesTransposed, contiguous, sortBuffer, dimension, position, 0);

    THCTensor_(free)(state, contiguous);
    THLongStorage_free(position);
    THCTensor_(free)(state, valuesTransposed);
    THCudaLongTensor_free(state, indicesTransposed);
    THCudaLongStorage_free(state, sortBuffer);
  }

  if (!keepdim) {
    THCTensor_(squeeze1d)(state, values, values, dimension);
    THCudaLongTensor_squeeze1d(state, indices, indices, dimension);
  }
}

#undef MAX_GRID_SIZE
#undef MAX_BLOCK_SIZE

#endif
