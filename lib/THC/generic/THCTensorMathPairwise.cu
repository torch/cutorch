
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathPairwise.cu"
#else

THC_API void
THCTensor_(add)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorAddConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorAddConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(sub)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorSubConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorSubConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(mul)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorMulConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorMulConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(div)(THCState* state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(value != ScalarConvert<int, real>::to(0), 3, "divide by zero");

  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorDivConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorDivConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(fmod)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorFmodOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorFmodOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(remainder)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorRemainderOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorRemainderOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(tril)(THCState *state, THCTensor *self_, THCTensor *src_, long k)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCTensor *src = src_;
  if (self_ == src_)
    src = THCTensor_(newContiguous)(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  real *start = THCTensor_(data)(state, src) + src->storageOffset;

  TensorTriOp<real, 0> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCTensor_(freeCopyTo)(state, src, src_);

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(triu)(THCState *state, THCTensor *self_, THCTensor *src_, long k)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCTensor *src = src_;
  if (self_ == src_)
    src = THCTensor_(newContiguous)(state, src_);

  long stride0 = src->stride[0];
  long stride1 = src->stride[1];
  real *start = THCTensor_(data)(state, src) + src->storageOffset;

  TensorTriOp<real, 1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCTensor_(freeCopyTo)(state, src, src_);

  THCudaCheck(cudaGetLastError());
}

THC_API int THCTensor_(equal)(THCState *state, THCTensor *self_, THCTensor *src_)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (!THCTensor_(isSameSizeAs(state, self_, src_))) {
    return 0;
  }

  // This is not as efficient as TH, but the basic idea: create a buffer that stores
  // 1 if the two tensors are equal at a position, otherwise 0. If the minimum value
  // in this buffer is 1, the two tensors are equal, otherwise they are not

  THLongStorage *size = THCTensor_(newSizeOf)(state, self_);
  THCudaByteTensor *buf = THCudaByteTensor_newWithSize(state, size, NULL);

  if (!THC_pointwiseApply3(state, buf, self_, src_, TensorEQOp<real, unsigned char>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  unsigned char min = THCudaByteTensor_minall(state, buf);

  THLongStorage_free(size);
  THCudaByteTensor_free(state, buf);

  return min != 0;
}

#endif
