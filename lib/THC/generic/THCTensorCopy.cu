
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorCopy.cu"
#else

THC_API void
THCTensor_(copy)(THCState* state, THCTensor* dst, THCTensor* src) {
  THC_copyTensor<THCTensor, THCTensor>(state, dst, src);
}

THC_API void
THCTensor_(copyIgnoringOverlaps)(THCState* state, THCTensor* dst, THCTensor* src) {
  // Called when we are copying into an overlapping index `dst`, but
  // we don't care which writer wins. Hacky but it works.
  // This is itself invoked by pointwiseApply2 / THCTensor_copy in
  // case that there are write overlaps.
  // FIXME: really, overlapping writes should be illegal/an error in Torch
  THC_pointwiseApply2(
    state, dst, src,
    CopyOp<typename TensorUtils<THCTensor>::DataType,
           typename TensorUtils<THCTensor>::DataType>(),
    ReadOnly, /* ignore overwrites */
    ReadOnly);
}

#define IMPLEMENT_THC_CUDA_TENSOR_COPY(TYPEC, TYPECUDA)                 \
  THC_API void                                                          \
  THCTensor_(copyCuda##TYPEC)(THCState *state,                          \
                              THCTensor *self,                          \
                              THCuda##TYPECUDA##Tensor *src) {          \
    THC_copyTensor<THCTensor, THCuda##TYPECUDA##Tensor>(state, self, src); \
  }

IMPLEMENT_THC_CUDA_TENSOR_COPY(Byte, Byte)

#ifndef THC_GENERIC_NO_CHAR
IMPLEMENT_THC_CUDA_TENSOR_COPY(Char, Char)
#endif
#ifndef THC_GENERIC_NO_SHORT
IMPLEMENT_THC_CUDA_TENSOR_COPY(Short, Short)
#endif
#ifndef THC_GENERIC_NO_INT
IMPLEMENT_THC_CUDA_TENSOR_COPY(Int, Int)
#endif
IMPLEMENT_THC_CUDA_TENSOR_COPY(Long, Long)
// THCudaTensor aka the non-existent THCudaFloatTensor
IMPLEMENT_THC_CUDA_TENSOR_COPY(Float, )
#ifndef THC_GENERIC_NO_DOUBLE
IMPLEMENT_THC_CUDA_TENSOR_COPY(Double, Double)
#endif
#ifndef THC_GENERIC_NO_HALF 
#ifdef CUDA_HALF_TENSOR
IMPLEMENT_THC_CUDA_TENSOR_COPY(Half, Half)
#endif
#endif

#undef IMPLEMENT_THC_CUDA_TENSOR_COPY

#endif
