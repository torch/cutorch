
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathPointwise.cu"
#else
# ifndef THC_GENERIC_NO_MATH
#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)             \
  struct Tensor_##NAME##_##REAL##_Op {                                  \
    __device__ __forceinline__ void operator()(real* out, real* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(real* v) const {         \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCTensor_(NAME)(THCState* state, THCTensor* self_, THCTensor* src) { \
    THAssert(THCTensor_(checkGPU)(state, 2, self_, src));               \
    if (self_ == src) {                                                 \
      if (!THC_pointwiseApply1(state, self_, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    } else {                                                            \
      THCTensor_(resizeAs)(state, self_, src);                          \
                                                                        \
      if (!THC_pointwiseApply2(state, self_, src, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(cudaGetLastError());                                    \
  }

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC, REAL) \
  IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  log, THCNumerics<real>::log,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, THCNumerics<real>::log1p, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  exp, THCNumerics<real>::exp,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cos, THCNumerics<real>::cos,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sin, THCNumerics<real>::sin,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sqrt, THCNumerics<real>::sqrt,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(rsqrt, THCNumerics<real>::rsqrt, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( ceil, THCNumerics<real>::ceil,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(floor, THCNumerics<real>::floor, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(trunc, THCNumerics<real>::trunc, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  neg, THCNumerics<real>::neg,   Real)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( acos, THCNumerics<real>::acos,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( cosh, THCNumerics<real>::cosh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( asin, THCNumerics<real>::asin,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sinh, THCNumerics<real>::sinh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  tan, THCNumerics<real>::tan,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( atan, THCNumerics<real>::atan,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( tanh, THCNumerics<real>::tanh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(round, THCNumerics<real>::round, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( frac, THCNumerics<real>::frac,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( cinv, THCNumerics<real>::cinv,  Real)

#endif

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  abs, THCNumerics<real>::abs,   Real)

#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

void THCTensor_(sign)(THCState* state, THCTensor* self_, THCTensor* src) {
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorSignOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorSignOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(clamp)(THCState *state, THCTensor *self_, THCTensor *src, real min_value,
  real max_value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorClampOp<real>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorClampOp<real>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cross)(THCState *state, THCTensor *self, THCTensor *x, THCTensor *y, int dimension)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self, x, y));

  int i;
  long nd = THCTensor_(nDimension)(state, x);
  ptrdiff_t nelem = THCTensor_(nElement)(state, x);
  THArgCheck(nd == THCTensor_(nDimension)(state, y), 1, "tensors must have same number of dimensions");
  for (i = 0; i < nd; i++) {
    THArgCheck(THCTensor_(size)(state, x, i) == THCTensor_(size)(state, y, i), 1, "dimension %i of x and y does not match", i);
    if (dimension < 0 && THCTensor_(size)(state, x, i) == 3) {
      dimension = i;
    }
  }

  THArgCheck(dimension >= 0 && dimension < nd, 3, "dimension %d out of range", dimension+1);
  THArgCheck(THCTensor_(size)(state, x, dimension) == 3, 3,
      "dimension %d does not have size 3", dimension+1);
  THCTensor_(resizeAs)(state, self, x);

  long sx = THCTensor_(stride)(state, x, dimension);
  long sy = THCTensor_(stride)(state, y, dimension);
  long so = THCTensor_(stride)(state, self, dimension);
  THCTensor *nx = THCTensor_(newNarrow)(state, x, dimension, 0, 1);
  THCTensor *ny = THCTensor_(newNarrow)(state, y, dimension, 0, 1);
  THCTensor *nself = THCTensor_(newNarrow)(state, self, dimension, 0, 1);
  if (!THC_pointwiseApply3(state, nself, nx, ny, TensorCrossOp<real>(sx, sy, so))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }
  THCTensor_(free)(state, nx);
  THCTensor_(free)(state, ny);
  THCTensor_(free)(state, nself);
}


#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(sigmoid)(THCState* state, THCTensor* self_, THCTensor* src) {
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorSigmoidOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorSigmoidOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(pow)(THCState *state, THCTensor *self_, THCTensor *src, real value) {
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorPowOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorPowOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(tpow)(THCState *state, THCTensor *self_, real value, THCTensor *src)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorTPowOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorTPowOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(lerp)(THCState *state, THCTensor *result, THCTensor *a, THCTensor *b, real w)
{
  THAssert(THCTensor_(checkGPU)(state, 3, result, a, b));
  THArgCheck(THCTensor_(nElement)(state, a) ==
             THCTensor_(nElement)(state, b), 3, "sizes do not match");
  THCTensor_(resizeAs)(state, result, a);

  if (!THC_pointwiseApply3(state, result, a, b, TensorLerpOp<real>(w))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#endif

THC_API void
THCTensor_(cadd)(THCState *state, THCTensor *self_, THCTensor* src1, real value, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == ScalarConvert<int, real>::to(1)) {
      // self += src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorAddOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += value * src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorCAddOp<real>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    if (value == ScalarConvert<int, real>::to(1)) {
      // self = src1 + src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 + value * src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorCAddOp<real>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(csub)(THCState *state, THCTensor *self_, THCTensor* src1, real value, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == ScalarConvert<int, real>::to(1)) {
      // self -= src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorSubOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += -value * src2
      if (!THC_pointwiseApply2(state, self_, src2,
                                   TensorCAddOp<real>(
                                     ScalarNegate<real>::to(value)))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    if (value == ScalarConvert<int, real>::to(1)) {
      // self = src1 - src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorSubOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 - value * src2
      if (!THC_pointwiseApply3(state, self_, src1, src2,
                                   TensorCAddOp<real>(
                                     ScalarNegate<real>::to(value)))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cmul)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self *= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorMulOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 * src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorMulOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cpow)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self = pow(self, src2)
    if (!THC_pointwiseApply2(state, self_, src2, TensorCPowOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = pow(src1, src2)
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorCPowOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cdiv)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorDivOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorDivOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cmax)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorMaxOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorMaxOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cmin)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorMinOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorMinOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cremainder)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorCRemainderOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorCRemainderOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cfmod)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2(state, self, src2, TensorCFmodOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3(state, self, src1, src2, TensorCFmodOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cmaxValue)(THCState *state, THCTensor *self, THCTensor *src, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1(state, self, TensorMaxValueOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2(state, self, src, TensorMaxValueOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(cminValue)(THCState *state, THCTensor *self, THCTensor *src, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1(state, self, TensorMinValueOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2(state, self, src, TensorMinValueOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

THC_API void
THCTensor_(addcmul)(THCState *state, THCTensor *self_, THCTensor *t, real value, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCTensor_(resizeAs)(state, self_, t);
    THCTensor_(copy)(state, self_, t);
  }
  else
  {
    THArgCheck(THCTensor_(nElement)(state, self_) == THCTensor_(nElement)(state, src1),
               1, "sizes do not match");
  }

  THArgCheck(THCTensor_(nElement)(state, src1) == THCTensor_(nElement)(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddCMulOp<real>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(addcdiv)(THCState *state, THCTensor *self_, THCTensor *t, real value, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCTensor_(resizeAs)(state, self_, t);
    THCTensor_(copy)(state, self_, t);
  }
  else
  {
    THArgCheck(THCTensor_(nElement)(state, self_) == THCTensor_(nElement)(state, src1),
               1, "sizes do not match");
  }
  THArgCheck(THCTensor_(nElement)(state, src1) == THCTensor_(nElement)(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddCDivOp<real>(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}
# endif /* THC_GENERIC_NO_MATH */
#endif
