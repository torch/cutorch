
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathScan.cu"
#else
# ifndef THC_GENERIC_NO_MATH
template<class BinaryOp>
__host__ void THCTensor_(scanOuterDim)(THCState *state, THCTensor *tgt,
                                       THCTensor *src, long dimension,
                                       real init, BinaryOp binary_op)
{
  unsigned ndim = THCTensor_(nDimension)(state, src);
  // Treat all outer dimensions (i.e. dim < dimension) as one.
  unsigned num_orows = 1;
  for (long dim = 0; dim < dimension; dim++) {
    num_orows *= THCTensor_(size)(state, src, dim);
  }
  unsigned row_size = THCTensor_(size)(state, src, dimension);
  // Treat all inner dimensions (i.e. dim > dimension) as one.
  unsigned num_irows = 1;
  for (unsigned dim = dimension + 1; dim < ndim; dim++) {
    num_irows *= THCTensor_(size)(state, src, dim);
  }

  dim3 threads(min(512, num_irows));
  unsigned maxGridDim = 1024;
  dim3 grid(min(maxGridDim, num_orows), min(maxGridDim, THCCeilDiv(num_irows, threads.x)));

  THCTensor_kernel_scanOuterDim<real><<<grid, threads, 0, THCState_getCurrentStream(state)>>>(
    THCTensor_(data)(state, tgt), THCTensor_(data)(state, src),
    num_orows, num_irows, row_size, init, binary_op);

  THCudaCheck(cudaGetLastError());
}

template<class BinaryFunction>
__host__ void THCTensor_(scanInnermostDim)(THCState *state, THCTensor *tgt,
                                           THCTensor *src, real init,
                                           BinaryFunction binary_op)
{
  unsigned ndim = THCTensor_(nDimension)(state, src);
  // Treat all outer dimensions as a single dimension.
  unsigned num_rows = 1;
  for (unsigned dim = 0; dim < ndim - 1; dim++) {
    num_rows *= THCTensor_(size)(state, src, dim);
  }
  unsigned row_size = THCTensor_(size)(state, src, ndim - 1);

  dim3 threads(16, 32);
  dim3 grid(min(1024, THCCeilDiv(num_rows, threads.y)));

  THCTensor_kernel_scanInnermostDim<real, 16, 32><<<grid, threads, 0, THCState_getCurrentStream(state)>>>(
    THCTensor_(data)(state, tgt), THCTensor_(data)(state, src), num_rows, row_size, init, binary_op);

  THCudaCheck(cudaGetLastError());
}

template<class BinaryFunction>
void THCTensor_(scanDim)(THCState *state, THCTensor *self_, THCTensor *src,
                         long dimension, real init, BinaryFunction binary_op)
{
  THCTensor_(resizeAs)(state, self_, src);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  src = THCTensor_(newContiguous)(state, src);

  if (dimension == THCTensor_(nDimension)(state, src) - 1) {
    THCTensor_(scanInnermostDim)(state, self, src, init, binary_op);
  } else {
    THCTensor_(scanOuterDim)(state, self, src, dimension, init, binary_op);
  }

  THCTensor_(free)(state, src);
  THCTensor_(freeCopyTo)(state, self, self_);
}

void THCTensor_(cumsum)(THCState *state, THCTensor *self, THCTensor *src, long dimension)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self, src));
  return THCTensor_(scanDim)(state, self, src, dimension,
                             ScalarConvert<float, real>::to(0.0), AddOp<real>());
}

void THCTensor_(cumprod)(THCState *state, THCTensor *self, THCTensor *src, long dimension)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self, src));
  return THCTensor_(scanDim)(state, self, src, dimension,
                             ScalarConvert<float, real>::to(1.0), MulOp<real>());
}
# endif /* THC_GENERIC_NO_MATH */
#endif
