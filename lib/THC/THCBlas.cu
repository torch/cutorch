#include "THCBlas.h"
#include "THCGeneral.h"
#include "THCHalf.h"

float THCudaBlas_Sdot(THCState *state, long n, float *x, long incx, float *y, long incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Sdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

double THCudaBlas_Ddot(THCState *state, long n, double *x, long incx, double *y, long incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    double result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Ddot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

#ifdef CUDA_HALF_TENSOR
float THCudaBlas_Hdot(THCState *state, long n, half *x, long incx, half *y, long incy)
{
#if CUDA_VERSION >= 8000
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDotEx(handle, i_n, x, HIP_R_16F, i_incx, y, HIP_R_16F, i_incy, &result, HIP_R_32F, HIP_R_32F));
    return result;
}

  THError("Cublas_Hdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
#else
  THError("Cublas_Hdot requires CUDA 8.0+");
  return 0;
#endif
}
#endif

/* Level 2 */
void THCudaBlas_Sgemv(THCState *state, char trans, long m, long n, float alpha, float *a, long lda, float *x, long incx, float beta, float *y, long incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Sgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Dgemv(THCState *state, char trans, long m, long n, double alpha, double *a, long lda, double *x, long incx, double beta, double *y, long incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Dgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Sger(THCState *state, long m, long n, float alpha, float *x, long incx, float *y, long incy, float *a, long lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasSger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Sger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dger(THCState *state, long m, long n, double alpha, double *x, long incx, double *y, long incy, double *a, long lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasDger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Dger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}


hipblasOperation_t convertTransToCublasOperation(char trans) {
  if (trans == 't') return HIPBLAS_OP_T;
  else if (trans == 'n') return HIPBLAS_OP_N;
  else if (trans == 'c') return HIPBLAS_OP_C;
  else {
    THError("trans must be one of: t, n, c");
    return HIPBLAS_OP_T;
  }
}

void adjustLd(char transa, char transb, long m, long n, long k, long *lda, long *ldb, long *ldc)
{
  int transa_ = ((transa == 't') || (transa == 'T'));
  int transb_ = ((transb == 't') || (transb == 'T'));

  if(n == 1)
    *ldc = m;

  if(transa_)
  {
    if(m == 1)
      *lda = k;
  }
  else
  {
    if(k == 1)
      *lda = m;
  }

  if(transb_)
  {
    if(k == 1)
      *ldb = n;
  }
  else
  {
    if(n == 1)
      *ldb = k;
  }
}

/* Level 3 */
void THCudaBlas_Sgemm(THCState *state, char transa, char transb, long m, long n, long k, float alpha, float *a, long lda, float *b, long ldb, float beta, float *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasSgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Sgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

#ifdef CUDA_HALF_TENSOR
// In CUDA 8.0, definition of data types for sgemmex changed
#if CUDA_VERSION < 8000
#  define HIP_R_16F CUBLAS_DATA_HALF
#endif

void THCudaBlas_Hgemm(THCState *state, char transa, char transb, long m, long n, long k, half alpha, half *a, long lda, half *b, long ldb, half beta, half *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));

    // Check for native Hgemm support
    if (THC_fastHalfInstructions(state)) {
      THCublasCheck(hipblasHgemm(handle, opa, opb,
				i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb,
				&beta, c, i_ldc));
    } else {
      // Simulated Hgemm
      float fAlpha = THC_half2float(alpha);
      float fBeta = THC_half2float(beta);

      THCublasCheck(cublasSgemmEx(handle, opa, opb,
				  i_m, i_n, i_k, &fAlpha,
                                  a, HIP_R_16F, i_lda, b, HIP_R_16F,
				  i_ldb, &fBeta, c, HIP_R_16F, i_ldc));
    }

    return;
  }
  THError("Cublas_Hgemm only supports m, n, k, lda, ldb, ldc"
          "with th bound [val] <= %d", INT_MAX);
}
#endif

void THCudaBlas_Dgemm(THCState *state, char transa, char transb, long m, long n, long k, double alpha, double *a, long lda, double *b, long ldb, double beta, double *c, long ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasDgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Dgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}


void THCudaBlas_SgemmBatched(THCState *state, char transa, char transb, long m, long n, long k,
                             float alpha, const float *a[], long lda, const float *b[], long ldb,
                             float beta, float *c[], long ldc, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_SgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}

#if CUDA_VERSION >= 8000
void THCudaBlas_SgemmStridedBatched(THCState *state, char transa, char transb, long m, long n, long k,
                             float alpha, const float *a, long lda, long strideA, const float *b, long ldb, long strideB,
                             float beta, float *c, long ldc, long strideC, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
        
  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

void THCudaBlas_DgemmBatched(THCState *state, char transa, char transb, long m, long n, long k,
                             double alpha, const double *a[], long lda, const double *b[], long ldb,
                             double beta, double *c[], long ldc, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}

#if CUDA_VERSION >= 8000
void THCudaBlas_DgemmStridedBatched(THCState *state, char transa, char transb, long m, long n, long k,
                             double alpha, const double *a, long lda, long strideA, const double *b, long ldb, long strideB,
                             double beta, double *c, long ldc, long strideC, long batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC, 
                                   (int)batchCount));
}
#endif

/* Inverse */
void THCudaBlas_Sgetrf(THCState *state, int n, float **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
}

void THCudaBlas_Dgetrf(THCState *state, int n, double **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
}

THC_API void THCudaBlas_Sgetrs(THCState *state, char transa, int n, int nrhs, const float **a, int lda, int *pivot, float **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
}


THC_API void THCudaBlas_Dgetrs(THCState *state, char transa, int n, int nrhs, const double **a, int lda, int *pivot, double **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
}

void THCudaBlas_Sgetri(THCState *state, int n, const float **a, int lda, int *pivot, float **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Sgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasSgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
}

void THCudaBlas_Dgetri(THCState *state, int n, const double **a, int lda, int *pivot, double **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Dgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasDgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
}
