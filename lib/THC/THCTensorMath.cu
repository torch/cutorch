#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCTensorRandom.h"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

#define NB_THREADS_PER_BLOCK 256

void THCudaTensor_fill(THCudaTensor *self_, float value)
{
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  thrust::fill(self_data, self_data+THCudaTensor_nElement(self), value);

  THCudaTensor_freeCopyTo(self, self_);
}

void THCudaTensor_zero(THCudaTensor *self_)
{
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  hipMemset(THCudaTensor_data(self), 0, sizeof(float)*THCudaTensor_nElement(self));
  THCudaTensor_freeCopyTo(self, self_);
}

struct addvalue_functor
{
  const float value;

  addvalue_functor(float value_) : value(value_) {}

    __host__ __device__ float operator()(const float& x) const
  {
    return (x+value);
  }
};

void THCudaTensor_add(THCudaTensor *self_, float value)
{
  {
    THCudaTensor *self = THCudaTensor_newContiguous(self_);
    long size = THCudaTensor_nElement(self);
    thrust::device_ptr<float> self_data(THCudaTensor_data(self));

    thrust::transform(self_data, self_data+size, self_data, addvalue_functor(value));

    THCudaTensor_freeCopyTo(self, self_);
  }
}

void THCudaTensor_mul(THCudaTensor *self_, float value)
{
  THCudaTensor *self = THCudaTensor_newContiguous(self_);

  hipblasSscal(THCudaTensor_nElement(self), value, THCudaTensor_data(self), 1);
  THCublasCheck();

  THCudaTensor_freeCopyTo(self, self_);
}

void THCudaTensor_div(THCudaTensor *self_, float value)
{
  THCudaTensor *self = THCudaTensor_newContiguous(self_);

  hipblasSscal(THCudaTensor_nElement(self), 1/value, THCudaTensor_data(self), 1);
  THCublasCheck();

  THCudaTensor_freeCopyTo(self, self_);
}

void THCudaTensor_cadd(THCudaTensor *self_, float value, THCudaTensor *src)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src), 3, "size do not match");

  {
    THCudaTensor *self = THCudaTensor_newContiguous(self_);
    src = THCudaTensor_newContiguous(src);

    hipblasSaxpy(THCudaTensor_nElement(self), value, THCudaTensor_data(src), 1, THCudaTensor_data(self), 1);
    THCublasCheck();

    THCudaTensor_free(src);
    THCudaTensor_freeCopyTo(self, self_);
  }
}

void THCudaTensor_cadd_tst(THCudaTensor *self_, THCudaTensor* src1, float value, THCudaTensor *src2)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src1), 3, "size do not match");
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src2), 3, "size do not match");

  {
    THCudaTensor *self = THCudaTensor_newContiguous(self_);

    src1 = THCudaTensor_newContiguous(src1);
    src2 = THCudaTensor_newContiguous(src2);

    THCudaTensor_copy(self, src1);
    hipblasSaxpy(THCudaTensor_nElement(self), value, THCudaTensor_data(src2), 1, THCudaTensor_data(self), 1);
    THCublasCheck();

    THCudaTensor_free(src1);
    THCudaTensor_free(src2);
    THCudaTensor_freeCopyTo(self, self_);
  }
}

void THCudaTensor_cmul(THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src1), 2, "size do not match");
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src2), 3, "size do not match");

  {
    THCudaTensor *self = THCudaTensor_newContiguous(self_);
    long size = THCudaTensor_nElement(self);
    src1 = THCudaTensor_newContiguous(src1);
    src2 = THCudaTensor_newContiguous(src2);
    thrust::device_ptr<float> self_data(THCudaTensor_data(self));
    thrust::device_ptr<float> src1_data(THCudaTensor_data(src1));
    thrust::device_ptr<float> src2_data(THCudaTensor_data(src2));

    thrust::transform(src2_data, src2_data+size, src1_data, self_data, thrust::multiplies<float>());

    THCudaTensor_free(src1);
    THCudaTensor_free(src2);
    THCudaTensor_freeCopyTo(self, self_);
  }
}

void THCudaTensor_cdiv(THCudaTensor *self_, THCudaTensor *src)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src), 2, "size do not match");

  {
    THCudaTensor *self = THCudaTensor_newContiguous(self_);
    long size = THCudaTensor_nElement(self);
    src = THCudaTensor_newContiguous(src);
    thrust::device_ptr<float> self_data(THCudaTensor_data(self));
    thrust::device_ptr<float> src_data(THCudaTensor_data(src));

    thrust::transform(self_data, self_data+size, src_data, self_data, thrust::divides<float>());

    THCudaTensor_free(src);
    THCudaTensor_freeCopyTo(self, self_);
  }
}

__global__ void THCudaTensor_kernel_addcmul(float *data, float value, float *src1, float *src2, long size)
{
  long k = (((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x) + threadIdx.x;
  
  if(k < size)
    data[k] += value*src1[k]*src2[k];
}


void THCudaTensor_addcmul(THCudaTensor *self_, float value, THCudaTensor *src1, THCudaTensor *src2)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src1), 3, "size do not match");
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src2), 4, "size do not match");

  {
    THCudaTensor *self = THCudaTensor_newContiguous(self_);
    long size = THCudaTensor_nElement(self);
    src1 = THCudaTensor_newContiguous(src1);
    src2 = THCudaTensor_newContiguous(src2);

    int nBlockPerRow, nBlockPerColumn, nThreadPerBlock;
    THCudaGetGridSize(&nBlockPerRow, &nBlockPerColumn, &nThreadPerBlock, size);
    dim3 threads(nThreadPerBlock);
    dim3 grid(nBlockPerRow, nBlockPerColumn);

    THCudaTensor_kernel_addcmul<<<grid, threads>>>(THCudaTensor_data(self), value, THCudaTensor_data(src1), THCudaTensor_data(src2), size);

    hipError_t errcode = hipGetLastError();
    if(errcode != hipSuccess)
      THError(hipGetErrorString(errcode));

    THCudaTensor_free(src1);
    THCudaTensor_free(src2);
    THCudaTensor_freeCopyTo(self, self_);
  }
}

__global__ void THCudaTensor_kernel_addcdiv(float *data, float value, float *src1, float *src2, long size)
{
  long k = (((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x) + threadIdx.x;

  if(k < size)
    data[k] += value*src1[k]/src2[k];
}


void THCudaTensor_addcdiv(THCudaTensor *self_, float value, THCudaTensor *src1, THCudaTensor *src2)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src1), 3, "size do not match");
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src2), 4, "size do not match");

  {
    THCudaTensor *self = THCudaTensor_newContiguous(self_);
    long size = THCudaTensor_nElement(self);
    src1 = THCudaTensor_newContiguous(src1);
    src2 = THCudaTensor_newContiguous(src2);

    int nBlockPerRow, nBlockPerColumn, nThreadPerBlock;
    THCudaGetGridSize(&nBlockPerRow, &nBlockPerColumn, &nThreadPerBlock, size);
    dim3 threads(nThreadPerBlock);
    dim3 grid(nBlockPerRow, nBlockPerColumn);

    THCudaTensor_kernel_addcdiv<<<grid, threads>>>(THCudaTensor_data(self), value, THCudaTensor_data(src1), THCudaTensor_data(src2), size);

    hipError_t errcode = hipGetLastError();
    if(errcode != hipSuccess)
      THError(hipGetErrorString(errcode));

    THCudaTensor_free(src1);
    THCudaTensor_free(src2);
    THCudaTensor_freeCopyTo(self, self_);
  }
}

float THCudaTensor_dot(THCudaTensor *self, THCudaTensor *src)
{
  THArgCheck(THCudaTensor_nElement(self) == THCudaTensor_nElement(src), 2, "size do not match");

  {
    self = THCudaTensor_newContiguous(self);
    src = THCudaTensor_newContiguous(src);

    float result = hipblasSdot(THCudaTensor_nElement(self),
                              THCudaTensor_data(self), 1,
                              THCudaTensor_data(src), 1);

    THCublasCheck();

    THCudaTensor_free(src);
    THCudaTensor_free(self);

    return result;
  }
}

float THCudaTensor_minall(THCudaTensor *self)
{
  self = THCudaTensor_newContiguous(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  float result = thrust::reduce(self_data, self_data+THCudaTensor_nElement(self), (float)(THInf), thrust::minimum<float>());

  THCudaTensor_free(self);
  return result;
}

float THCudaTensor_maxall(THCudaTensor *self)
{
  self = THCudaTensor_newContiguous(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  float result = thrust::reduce(self_data, self_data+THCudaTensor_nElement(self), (float)(-THInf), thrust::maximum<float>());

  THCudaTensor_free(self);
  return result;
}

float THCudaTensor_sumall(THCudaTensor *self)
{
  self = THCudaTensor_newContiguous(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  float result = thrust::reduce(self_data, self_data+THCudaTensor_nElement(self), (float)(0), thrust::plus<float>());

  THCudaTensor_free(self);
  return result;
}



struct dim4 {
    unsigned arr[4];

    __host__ dim4(unsigned init=0) {
        for(unsigned i=0; i<4; i++) { arr[i] = init; }
    }

    __host__ __device__ unsigned& operator[](const unsigned& idx) { return arr[idx]; }
};



/* Reduce one of the outer dimensions of a tensor
 *
 * For an n-d tensor (n <= 4) where the reduction is *not* along the innermost
 * dimension:
 *
 * - block.x and grid.x make up the innermost dimension;
 * - The reduced dimension is looped over inside a block; and
 * - grid.y and grid.z are the remaining two dimensions (if any).
 * - block.y and block.z are not used as we're limited to 512 or 1024 threads
 *   in the block.
 *
 * For sizes/strides, index 3 is the reduced dimension, while the remaining
 * indices are for the remaining dimensions with index 0 the innermost dimension.
 *
 * Reduction along the innermost dimension is handled in a separate kernel.
 */
template<class UnaryFunction, class BinaryFunction>
__global__ void THCudaTensor_kernel_transformReduceOuterDim(float *tgt, float *src_,
        dim4 src_stride, dim4 tgt_stride, dim4 size,
        UnaryFunction unary_op, float init, BinaryFunction binary_op)
{
  const size_t reduce = 3;

  for(unsigned z = blockIdx.z; z < size[2] ; z += gridDim.z)
  for(unsigned y = blockIdx.y; y < size[1] ; y += gridDim.y)
  for(unsigned col = blockIdx.x * blockDim.x + threadIdx.x; col < size[0]; col += blockDim.x * gridDim.x) {
    float *src = src_ + z * src_stride[2] + y * src_stride[1] + col;
    float acc = init;
    for(unsigned i=0; i < size[reduce]; i++) {
      acc = binary_op(acc, unary_op(*src));
      src += src_stride[reduce];
    }
    tgt[z * tgt_stride[2] + y * tgt_stride[1] + col] = float(acc);
  }
}



template<class UnaryFunction, class BinaryFunction>
__host__ void THCudaTensor_transformReduceOuterDim(THCudaTensor *tgt, THCudaTensor *src,
        long rdim, UnaryFunction unary_op, float init, BinaryFunction binary_op)
{
  const size_t reduce = 3;
  dim4 src_stride(0);
  dim4 tgt_stride(0);
  dim4 size(1);

  unsigned ndim = THCudaTensor_nDimension(src);
  for(unsigned idim=0, o=ndim-2; idim < ndim; idim++) {
    unsigned odim = idim == rdim ? reduce : o--;
    src_stride[odim] = THCudaTensor_stride(src, idim);
    tgt_stride[odim] = THCudaTensor_stride(tgt, idim);
    size[odim]       = THCudaTensor_size(src, idim);
  }

  const unsigned nThreadPerBlock = 256;
  unsigned nBlockPerColumn = (size[0] + nThreadPerBlock - 1) / nThreadPerBlock;
  dim3 threads(nThreadPerBlock);
  unsigned maxGridDim = 1024; // anything < 64k is fine. The choice has no impact on performance.
  dim3 grid(min(maxGridDim, nBlockPerColumn), min(maxGridDim, size[1]), min(maxGridDim, size[2]));

  THCudaTensor_kernel_transformReduceOuterDim<<<grid, threads>>>(THCudaTensor_data(tgt),
          THCudaTensor_data(src), src_stride, tgt_stride, size, unary_op, init, binary_op);
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
}



/* Reduce the innermost dimension of a tensor
 *
 * For an n-d tensor (n <= 4) where the reduction is along the innermost dimension:
 *
 * - block.x is the innermost dimension, i.e. dimension 0;
 * - block.y and grid.y make up dimension 1; and
 * - grid.x and grid z are the remaining two outer dimensions (if any)
 *
 * Reduction along other dimensions is handled in a separate kernel.
 */
template<class UnaryFunction, class BinaryFunction>
__global__ void THCudaTensor_kernel_transformReduceInnermostDim(float *tgt, float *src_,
        dim4 src_stride, dim4 tgt_stride, dim4 size, UnaryFunction unary_op, float init, BinaryFunction binary_op)
{
  __shared__ float sbuf[16][32]; // 8kB

  for(unsigned z = blockIdx.z; z < size[3] ; z += gridDim.z)
  for(unsigned x = blockIdx.x; x < size[2] ; x += gridDim.x)
  for(unsigned bRow = blockIdx.y * blockDim.y; bRow < size[1]; bRow += blockDim.y * gridDim.y) {

    float acc = init;
    unsigned row = bRow + threadIdx.y;
    float *src = src_ + z * src_stride[3] + x * src_stride[2] + row * src_stride[1];
    bool reducing = threadIdx.x < blockDim.y && bRow + threadIdx.x < size[1] && threadIdx.y == 0;

    for(unsigned bCol=0; bCol < size[0]; bCol += blockDim.x) {

      sbuf[threadIdx.y][threadIdx.x] = init;
      unsigned col = bCol + threadIdx.x;
      if(row < size[1] && col < size[0]) {
        sbuf[threadIdx.y][threadIdx.x] = unary_op(src[col]);
      }
      __syncthreads();

      float* line = &sbuf[threadIdx.y][0];
      for(unsigned s = 16; s > 1; s >>= 1) {
        if(row < size[1] && threadIdx.x < s) {
          line[threadIdx.x] = binary_op(line[threadIdx.x], line[threadIdx.x + s]);
        }
        __syncthreads();
      }
      if(reducing) {
        sbuf[threadIdx.x][0] = binary_op(sbuf[threadIdx.x][0], sbuf[threadIdx.x][1]);
        acc = binary_op(acc, sbuf[threadIdx.x][0]);
      }
      __syncthreads();
    }

    if(reducing) {
      unsigned row = bRow + threadIdx.x;
      unsigned tgt_offset = z * tgt_stride[3] + x * tgt_stride[2];
      tgt[tgt_offset + row] = acc;
    }
  }
}



template<class UnaryFunction, class BinaryFunction>
__host__ void THCudaTensor_transformReduceInnermostDim(THCudaTensor *tgt, THCudaTensor *src,
        UnaryFunction unary_op, float init, BinaryFunction binary_op)
{
  dim4 src_stride(0);
  dim4 tgt_stride(0);
  dim4 size(1);

  unsigned ndim = THCudaTensor_nDimension(src);
  for(unsigned dim=0; dim < ndim; dim++) {
    unsigned odim = ndim - 1 - dim;
    src_stride[odim] = THCudaTensor_stride(src, dim);
    tgt_stride[odim] = THCudaTensor_stride(tgt, dim);
    size[odim]       = THCudaTensor_size(src, dim);
  }

  dim3 threads(32, 16);
  unsigned nBlockPerRow = (size[1] + threads.y - 1) / threads.y;
  unsigned maxGridDim = 1024; // anything < 64k is fine. The choice has no impact on performance.
  dim3 grid(min(maxGridDim, size[2]), min(maxGridDim, nBlockPerRow), min(maxGridDim, size[3]));

  THCudaTensor_kernel_transformReduceInnermostDim<<<grid, threads>>>(THCudaTensor_data(tgt),
          THCudaTensor_data(src), src_stride, tgt_stride, size, unary_op, init, binary_op);
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
}


template<class UnaryFunction, class BinaryFunction>
void THCudaTensor_transformReduceDim(THCudaTensor *self_, THCudaTensor *src,
        long dimension, UnaryFunction unary_op, float init, BinaryFunction binary_op)
{
  THArgCheck(dimension >= 0 && dimension < THCudaTensor_nDimension(src), 3, "dimension out of range");
  THArgCheck(THCudaTensor_nDimension(src) <= 4, 2, "too many dimensions (>4)");

  THLongStorage *dim = THCudaTensor_newSizeOf(src);
  THLongStorage_set(dim, dimension, 1);
  THCudaTensor_resize(self_, dim, NULL);
  THLongStorage_free(dim);

  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  src = THCudaTensor_newContiguous(src);

  if(dimension == THCudaTensor_nDimension(src)-1) {
    THCudaTensor_transformReduceInnermostDim(self, src, unary_op, init, binary_op);
  } else {
    THCudaTensor_transformReduceOuterDim(self, src, dimension, unary_op, init, binary_op);
  }

  THCudaTensor_free(src);
  THCudaTensor_freeCopyTo(self, self_);
}


template<class BinaryFunction>
void THCudaTensor_reduceDim(THCudaTensor *self_, THCudaTensor *src, long dimension, float init, BinaryFunction binary_op)
{
  THCudaTensor_transformReduceDim(self_, src, dimension, thrust::identity<float>(), init, binary_op);
}


void THCudaTensor_sum(THCudaTensor *self, THCudaTensor *src, long dimension)
{
  return THCudaTensor_reduceDim(self, src, dimension, 0.0f, thrust::plus<float>());
}


void THCudaTensor_max(THCudaTensor *self, THCudaTensor *src, long dimension)
{
  const float minfloat32 = -3.402823466e+38f;
  return THCudaTensor_reduceDim(self, src, dimension, minfloat32, thrust::maximum<float>());
}


void THCudaTensor_min(THCudaTensor *self, THCudaTensor *src, long dimension)
{
  const float maxfloat32 = 3.402823466e+38f;
  return THCudaTensor_reduceDim(self, src, dimension, maxfloat32, thrust::minimum<float>());
}


void THCudaTensor_addmv(THCudaTensor *self, float beta, float alpha, THCudaTensor *mat, THCudaTensor *vec)
{
  if( (mat->nDimension != 2) || (vec->nDimension != 1) )
    THError("matrix and vector expected");

  if( mat->size[1] != vec->size[0] )
    THError("size mismatch");

  if(self->nDimension != 1)
    THError("size mismatch");

  if( self->size[0] != mat->size[0] )
    THError("size mismatch");

  if(mat->stride[0] == 1)
  {
    hipblasSgemv('n', mat->size[0], mat->size[1],
                alpha, THCudaTensor_data(mat), mat->stride[1],
                THCudaTensor_data(vec), vec->stride[0],
                beta, THCudaTensor_data(self), self->stride[0]);
  }
  else if(mat->stride[1] == 1)
  {
    hipblasSgemv('t',  mat->size[1], mat->size[0],
                alpha, THCudaTensor_data(mat), mat->stride[0],
                THCudaTensor_data(vec), vec->stride[0],
                beta, THCudaTensor_data(self), self->stride[0]);
  }
  else
  {
    mat = THCudaTensor_newContiguous(mat);
    
    hipblasSgemv('t',  mat->size[1], mat->size[0],
                alpha, THCudaTensor_data(mat), mat->stride[0],
                THCudaTensor_data(vec), vec->stride[0],
                beta, THCudaTensor_data(self), self->stride[0]);
    
    THCudaTensor_free(mat);
  }

  THCublasCheck();  
}

void THCudaTensor_addmm(THCudaTensor *self, float beta, float alpha, THCudaTensor *m1, THCudaTensor *m2)
{
  char transpose, transpose_m1, transpose_m2;
  THCudaTensor *self_, *m1_, *m2_;

  if( (m1->nDimension != 2) || (m2->nDimension != 2) ) 
    THError("matrix and matrix expected"); 

  if(self->nDimension != 2)
    THError("size mismatch"); 

  if( (self->size[0] != m1->size[0]) || (self->size[1] != m2->size[1]) || (m1->size[1] != m2->size[0]) ) 
    THError("size mismatch"); 

  /* self */
  if(self->stride[0] == 1)
  {
    transpose = 'n';
    self_ = self;
  }
  else if(self->stride[1] == 1)
  {
    THCudaTensor *swap = m2;
    m2 = m1;
    m1 = swap;
    THCudaTensor_transpose(self, NULL, 0, 1);
    THCudaTensor_transpose(m1, NULL, 0, 1);
    THCudaTensor_transpose(m2, NULL, 0, 1);
    transpose = 't';
    self_ = self;
  }
  else
  {
    transpose = 'n';
    THCudaTensor_transpose(self, NULL, 0, 1);
    self_ = THCudaTensor_newClone(self);
    THCudaTensor_transpose(self, NULL, 0, 1);
    THCudaTensor_transpose(self_, NULL, 0, 1);
  }

  /* m1 */
  if(m1->stride[0] == 1)
  {
    transpose_m1 = 'n';
    m1_ = m1;
  }
  else if(m1->stride[1] == 1)
  {
    transpose_m1 = 't';
    m1_ = m1;
  }
  else
  {
    transpose_m1 = 't';
    m1_ = THCudaTensor_newContiguous(m1);
  }

  /* m2 */
  if(m2->stride[0] == 1)
  {
    transpose_m2 = 'n';
    m2_ = m2;
  }
  else if(m2->stride[1] == 1)
  {
    transpose_m2 = 't';
    m2_ = m2;
  }
  else
  {
    transpose_m2 = 't';
    m2_ = THCudaTensor_newContiguous(m2);
  }

  /* do the operation */
  hipblasSgemm(transpose_m1,
              transpose_m2,
              self_->size[0],
              self_->size[1],
              m1_->size[1],
              alpha,
              THCudaTensor_data(m1_),
              (transpose_m1 == 'n' ? m1_->stride[1] : m1_->stride[0]),
              THCudaTensor_data(m2_),
              (transpose_m2 == 'n' ? m2_->stride[1] : m2_->stride[0]),
              beta,
              THCudaTensor_data(self_),
              self_->stride[1]);

  THCublasCheck();

  /* free intermediate variables */
  if(m1_ != m1)
    THCudaTensor_free(m1_);

  if(m2_ != m2)
    THCudaTensor_free(m2_);

  if(self_ != self)
    THCudaTensor_freeCopyTo(self_, self);

  if(transpose == 't')
  {
    THCudaTensor_transpose(self, NULL, 0, 1);
    THCudaTensor_transpose(m1, NULL, 0, 1);
    THCudaTensor_transpose(m2, NULL, 0, 1);
  }
}

void THCudaTensor_addr(THCudaTensor *self, float alpha, THCudaTensor *vec1, THCudaTensor *vec2)
{
  if( (vec1->nDimension != 1) || (vec2->nDimension != 1) )
    THError("vector and vector expected");

  if(self->nDimension != 2)
    THError("size mismatch");

  if( (self->size[0] != vec1->size[0]) || (self->size[1] != vec2->size[0]) )
    THError("size mismatch");

  if(self->stride[0] == 1)
  {
    hipblasSger(vec1->size[0], vec2->size[0],
               alpha, THCudaTensor_data(vec1), vec1->stride[0],
               THCudaTensor_data(vec2), vec2->stride[0],
               THCudaTensor_data(self), self->stride[1]);
  }
  else if(self->stride[1] == 1)
  {
    hipblasSger(vec2->size[0], vec1->size[0],
               alpha, THCudaTensor_data(vec2), vec2->stride[0],
               THCudaTensor_data(vec1), vec1->stride[0],
               THCudaTensor_data(self), self->stride[0]);
  }
  else
  {
    THCudaTensor *cself = THCudaTensor_newClone(self);

    hipblasSger(vec2->size[0], vec1->size[0],
               alpha, THCudaTensor_data(vec2), vec2->stride[0],
               THCudaTensor_data(vec1), vec1->stride[0],
               THCudaTensor_data(cself), cself->stride[0]);

    THCudaTensor_freeCopyTo(cself, self);
  }

  THCublasCheck();
}

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC)                   \
  struct NAME##_functor                                                \
  {                                                                     \
    __host__ __device__ float operator()(const float& x) const          \
    {                                                                   \
      return CFUNC(x);                                                  \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCudaTensor_##NAME(THCudaTensor *self_)                         \
  {                                                                     \
    THCudaTensor *self = THCudaTensor_newContiguous(self_);             \
    long size = THCudaTensor_nElement(self);                            \
    thrust::device_ptr<float> self_data(THCudaTensor_data(self));       \
                                                                        \
    thrust::transform(self_data, self_data+size, self_data, NAME##_functor()); \
                                                                        \
    THCudaTensor_freeCopyTo(self, self_);                               \
  }

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log, log)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, log1p)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(exp, exp)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(cos, cos)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(acos, acos)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(cosh, cosh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sin, sin)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(asin, asin)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sinh, sinh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(tan, tan)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(atan, atan)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(tanh, tanh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sqrt, sqrt)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(ceil, ceil)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(floor, floor)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(abs, fabs)

struct pow_functor
{
  const float value;

  pow_functor(float value_) : value(value_) {}

    __host__ __device__ float operator()(const float& x) const
  {
    return pow(x, value);
  }
};

void THCudaTensor_pow(THCudaTensor *self_, THCudaTensor *src, float value)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src), 2, "sizes do not match");
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  src = THCudaTensor_newContiguous(src);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  thrust::device_ptr<float> src_data(THCudaTensor_data(src));
  
  thrust::transform(src_data, src_data+size, self_data, pow_functor(value));

  THCudaTensor_free(src);
  THCudaTensor_freeCopyTo(self, self_);
}


struct sign_functor
{
  __device__ float operator()(const float &v) const {
    return (v > 0) - (v < 0);
  }
};


void THCudaTensor_sign(THCudaTensor *self_, THCudaTensor *src)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src), 2, "size do not match");

  {
    THCudaTensor *self = THCudaTensor_newContiguous(self_);
    long size = THCudaTensor_nElement(self);
    src = THCudaTensor_newContiguous(src);
    thrust::device_ptr<float> self_data(THCudaTensor_data(self));
    thrust::device_ptr<float> src_data(THCudaTensor_data(src));

    thrust::transform(src_data, src_data+size, self_data, sign_functor());

    THCudaTensor_free(src);
    THCudaTensor_freeCopyTo(self, self_);
  }
}

float THCudaTensor_meanall(THCudaTensor *self)
{
  THArgCheck(self->nDimension > 0, 1, "empty Tensor");
  return THCudaTensor_sumall(self)/THCudaTensor_nElement(self);
}

void
THCudaTensor_mean(THCudaTensor *self, THCudaTensor *src, long dim)
{
  THCudaTensor_sum(self, src, dim);
  THCudaTensor_div(self, THCudaTensor_size(src, dim));
}

struct square_functor
{
  const float mean;

  square_functor(float mean_) : mean(mean_) {}

    __host__ __device__ float operator()(const float& x) const
  {
    return (x-mean)*(x-mean);
  }
};

float THCudaTensor_varall(THCudaTensor *self)
{
  self = THCudaTensor_newContiguous(self);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  float mean = THCudaTensor_meanall(self);
  float result = thrust::transform_reduce(self_data, self_data+size, square_functor(mean), (float)0, thrust::plus<float>());

  result = result/(THCudaTensor_nElement(self)-1);

  THCudaTensor_free(self);
  return result;
}

float THCudaTensor_stdall(THCudaTensor *self)
{
  return sqrt(THCudaTensor_varall(self));
}



template<class Op>
void THCudaTensor_logicalValue(THCudaTensor *self_, THCudaTensor *src, Op op)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src), 2, "size do not match");

  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  src = THCudaTensor_newContiguous(src);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  thrust::device_ptr<float> src_data(THCudaTensor_data(src));

  thrust::transform(src_data, src_data+size, self_data, op);

  THCudaTensor_free(src);
  THCudaTensor_freeCopyTo(self, self_);
}


struct partial_less_functor
{
  const float rhs;
  partial_less_functor(float rhs) : rhs(rhs) {}
  __host__ __device__ bool operator()(const float &lhs) const {return lhs < rhs;}
};


void THCudaTensor_ltValue(THCudaTensor *self_, THCudaTensor *src, float value)
{
  THCudaTensor_logicalValue(self_, src, partial_less_functor(value));
}


struct partial_greater_functor
{
  const float rhs;
  partial_greater_functor(float rhs) : rhs(rhs) {}
  __host__ __device__ bool operator()(const float &lhs) const {return lhs > rhs;}
};


void THCudaTensor_gtValue(THCudaTensor *self_, THCudaTensor *src, float value)
{
  THCudaTensor_logicalValue(self_, src, partial_greater_functor(value));
}


struct partial_less_equal_functor
{
  const float rhs;
  partial_less_equal_functor(float rhs) : rhs(rhs) {}
  __host__ __device__ bool operator()(const float &lhs) const {return lhs <= rhs;}
};


void THCudaTensor_leValue(THCudaTensor *self_, THCudaTensor *src, float value)
{
  THCudaTensor_logicalValue(self_, src, partial_less_equal_functor(value));
}


struct partial_greater_equal_functor
{
  const float rhs;
  partial_greater_equal_functor(float rhs) : rhs(rhs) {}
  __host__ __device__ bool operator()(const float &lhs) const {return lhs >= rhs;}
};


void THCudaTensor_geValue(THCudaTensor *self_, THCudaTensor *src, float value)
{
  THCudaTensor_logicalValue(self_, src, partial_greater_equal_functor(value));
}


struct partial_equal_functor
{
  const float rhs;
  partial_equal_functor(float rhs) : rhs(rhs) {}
  __host__ __device__ bool operator()(const float &lhs) const {return lhs == rhs;}
};


void THCudaTensor_eqValue(THCudaTensor *self_, THCudaTensor *src, float value)
{
  THCudaTensor_logicalValue(self_, src, partial_equal_functor(value));
}


struct partial_not_equal_functor
{
  const float rhs;
  partial_not_equal_functor(float rhs) : rhs(rhs) {}
  __host__ __device__ bool operator()(const float &lhs) const {return lhs != rhs;}
};


void THCudaTensor_neValue(THCudaTensor *self_, THCudaTensor *src, float value)
{
  THCudaTensor_logicalValue(self_, src, partial_not_equal_functor(value));
}


template<class Op>
void THCudaTensor_logicalTensor(THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2, Op op)
{
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src1), 2, "size does not match");
  THArgCheck(THCudaTensor_nElement(self_) == THCudaTensor_nElement(src2), 3, "size does not match");

  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  src1 = THCudaTensor_newContiguous(src1);
  src2 = THCudaTensor_newContiguous(src2);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  thrust::device_ptr<float> src1_data(THCudaTensor_data(src1));
  thrust::device_ptr<float> src2_data(THCudaTensor_data(src2));

  thrust::transform(src1_data, src1_data+size, src2_data, self_data, op);

  THCudaTensor_free(src1);
  THCudaTensor_free(src2);
  THCudaTensor_freeCopyTo(self, self_);
}


void THCudaTensor_ltTensor(THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THCudaTensor_logicalTensor(self_, src1, src2, thrust::less<float>());
}


void THCudaTensor_gtTensor(THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THCudaTensor_logicalTensor(self_, src1, src2, thrust::greater<float>());
}


void THCudaTensor_leTensor(THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THCudaTensor_logicalTensor(self_, src1, src2, thrust::less_equal<float>());
}


void THCudaTensor_geTensor(THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THCudaTensor_logicalTensor(self_, src1, src2, thrust::greater_equal<float>());
}


void THCudaTensor_eqTensor(THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THCudaTensor_logicalTensor(self_, src1, src2, thrust::equal_to<float>());
}


void THCudaTensor_neTensor(THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THCudaTensor_logicalTensor(self_, src1, src2, thrust::not_equal_to<float>());
}


struct norm_functor
{
  const float exponent;

  norm_functor(float exponent_) : exponent(exponent_) {}

    __host__ __device__ float operator()(const float& x) const
  {
    return pow(fabs(x), exponent);
  }
};


float THCudaTensor_normall(THCudaTensor *self, float value)
{
  self = THCudaTensor_newContiguous(self);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  float result;
  if(value == 0.0f) {
    result = thrust::transform_reduce(self_data, self_data+size, partial_not_equal_functor(0.0f), (float)0, thrust::plus<float>());
  } else {
    result = thrust::transform_reduce(self_data, self_data+size, norm_functor(value), (float)0, thrust::plus<float>());
    result = pow(result, (float)1.0/value);
  }

  THCudaTensor_free(self);
  return result;
}

void THCudaTensor_norm(THCudaTensor* self, THCudaTensor* src, float value, long dimension)
{
  if(value == 0.0f) {
    THCudaTensor_transformReduceDim(self, src, dimension, partial_not_equal_functor(0.0f), (float)0, thrust::plus<float>());
  } else {
    THCudaTensor_transformReduceDim(self, src, dimension, norm_functor(value), (float)0, thrust::plus<float>());
    THCudaTensor_pow(self, self, 1/value);
  }
}


struct dist_functor
{
  const float exponent;

  dist_functor(float exponent_) : exponent(exponent_) {}

  __host__ __device__ float operator()(const float& x, const float& y) const
  {
    return pow(fabs(x-y), exponent);
  }
};

float THCudaTensor_dist(THCudaTensor *self, THCudaTensor *src, float value)
{
  self = THCudaTensor_newContiguous(self);
  long size = THCudaTensor_nElement(self);
  src = THCudaTensor_newContiguous(src);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  thrust::device_ptr<float> src_data(THCudaTensor_data(src));

  float result = thrust::inner_product(self_data, self_data+size, src_data, (float) 0,thrust::plus<float>(), dist_functor(value));

  THCudaTensor_free(src);
  THCudaTensor_free(self);
  
  return pow(result, (float)1.0/value);
}

void THCudaTensor_rand(THCudaTensor *r_, THLongStorage *size)
{
  THCudaTensor_resize(r_, size, NULL);
  THCudaTensor_uniform(r_, 0, 1);
}

void THCudaTensor_randn(THCudaTensor *r_, THLongStorage *size)
{
  THCudaTensor_resize(r_, size, NULL);
  THCudaTensor_normal(r_, 0, 1);
}
