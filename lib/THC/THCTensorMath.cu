#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCApply.cuh"
#include "THCNumerics.cuh"

#include <cfloat>

void THCudaTensor_cat(THCState *state, THCudaTensor *result, THCudaTensor *ta, THCudaTensor *tb, int dimension)
{
  THCudaTensor* inputs[2];
  inputs[0] = ta;
  inputs[1] = tb;
  THCudaTensor_catArray(state, result, inputs, 2, dimension);
}

void THCudaTensor_catArray(THCState *state, THCudaTensor *result, THCudaTensor **inputs, int numInputs, int dimension)
{
  THLongStorage *size;
  int i, j;
  long offset;
  int ndim = dimension + 1;
  for (i = 0; i < numInputs; i++)
  {
    ndim = THMax(ndim, THCudaTensor_nDimension(state, inputs[i]));
  }

  THArgCheck(numInputs > 0, 3, "invalid number of inputs %d", numInputs);
  THArgCheck(dimension >= 0, 4, "invalid dimension %d", dimension+1);

  size = THLongStorage_newWithSize(ndim);
  for(i = 0; i < ndim; i++)
  {
    long dimSize = i < THCudaTensor_nDimension(state, inputs[0])
                       ? THCudaTensor_size(state, inputs[0], i)
                       : 1;
    if (i == dimension)
    {
      for (j = 1; j < numInputs; j++)
      {
        dimSize += i < THCudaTensor_nDimension(state, inputs[j])
                       ? THCudaTensor_size(state, inputs[j], i)
                       : 1;
      }
    }
    else
    {
      for (j = 1; j < numInputs; j++)
      {
        if (dimSize != (i < THCudaTensor_nDimension(state, inputs[j])
                            ? THCudaTensor_size(state, inputs[j], i)
                            : 1)) {
          THLongStorage_free(size);
          THError("inconsistent tensor sizes");
        }
      }
    }
    size->data[i] = dimSize;
  }

  THCudaTensor_resize(state, result, size, NULL);
  THLongStorage_free(size);

  offset = 0;
  for (j = 0; j < numInputs; j++)
  {
    long dimSize = dimension < THCudaTensor_nDimension(state, inputs[j])
                       ? THCudaTensor_size(state, inputs[j], dimension)
                       : 1;
    THCudaTensor *nt = THCudaTensor_newWithTensor(state, result);
    THCudaTensor_narrow(state, nt, NULL, dimension, offset, dimSize);
    THCudaTensor_copy(state, nt, inputs[j]);
    THCudaTensor_free(state, nt);
    offset += dimSize;
  }
}

struct TensorAddCMulOp {
  TensorAddCMulOp(float v) : val(v) {}

  __device__ __forceinline__ void
  operator()(float* out, float* in1, float* in2) {
    *out += val * *in1 * *in2;
  }

  float val;
};

void THCudaTensor_addcmul(THCState *state, THCudaTensor *self_, THCudaTensor *t, float value, THCudaTensor *src1, THCudaTensor *src2)
{
  THAssert(THCudaTensor_checkGPU(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCudaTensor_resizeAs(state, self_, t);
    THCudaTensor_copy(state, self_, t);
  }
  else
  {
    THArgCheck(THCudaTensor_nElement(state, self_) == THCudaTensor_nElement(state, src1),
               1, "sizes do not match");
  }

  THArgCheck(THCudaTensor_nElement(state, src1) == THCudaTensor_nElement(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddCMulOp(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

struct TensorAddCDivOp {
  TensorAddCDivOp(float v) : val(v) {}

  __device__ __forceinline__ void
  operator()(float* out, float* in1, float* in2) {
    *out += val * *in1 / *in2;
  }

  float val;
};

void THCudaTensor_addcdiv(THCState *state, THCudaTensor *self_, THCudaTensor *t, float value, THCudaTensor *src1, THCudaTensor *src2)
{
  THAssert(THCudaTensor_checkGPU(state, 4, self_, t, src1, src2));
  if(self_ != t)
  {
    THCudaTensor_resizeAs(state, self_, t);
    THCudaTensor_copy(state, self_, t);
  }
  else
  {
    THArgCheck(THCudaTensor_nElement(state, self_) == THCudaTensor_nElement(state, src1),
               1, "sizes do not match");
  }
  THArgCheck(THCudaTensor_nElement(state, src1) == THCudaTensor_nElement(state, src2),
             3, "sizes do not match");

  if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddCDivOp(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

template <typename T>
struct TensorFillOp {
  TensorFillOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* v) { *v = val; }

  const T val;
};

#include "generic/THCTensorMath.cu"
#include "THCGenerateAllTypes.h"
