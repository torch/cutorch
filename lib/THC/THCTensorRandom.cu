#include "hip/hip_runtime.h"
#include "THCTensorRandom.h"
#include "THCDeviceUtils.cuh"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCTensorMath.h"
#include "THCReduceApplyUtils.cuh"
#include "THCTensorRandom.cuh"

#include <thrust/functional.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256

/* Sets up generator. Allocates but does not create the generator states. */
__host__ void initializeGenerator(THCState *state, Generator* gen)
{
  THCudaCheck(THCudaMalloc(state, (void**)&gen->gen_states, MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32)));
  THCudaCheck(THCudaMalloc(state, (void**)&gen->kernel_params, sizeof(mtgp32_kernel_params)));
}

/* Frees memory allocated during setup. */
__host__ void destroyGenerator(THCState *state, Generator* gen)
{
  if (gen->gen_states)
  {
    THCudaCheck(THCudaFree(state, gen->gen_states));
    gen->gen_states = NULL;
  }
  if (gen->kernel_params)
  {
    THCudaCheck(THCudaFree(state, gen->kernel_params));
    gen->kernel_params = NULL;
  }
}

/* Creates a new generator state given the seed. */
__host__ void createGeneratorState(Generator* gen, unsigned long seed)
{
  if (hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, gen->kernel_params) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP constants failed.");
  }
  if (hiprandMakeMTGP32KernelState(gen->gen_states, mtgp32dc_params_fast_11213,
                                  gen->kernel_params, MAX_NUM_BLOCKS, seed) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP kernel state failed.");
  }
}

/* Initialize generator array (must be called before any other function) */
__host__ void THCRandom_init(THCState* state, int devices, int current_device)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  rng_state->num_devices = devices;
  rng_state->gen = (Generator*)malloc(rng_state->num_devices * sizeof(Generator));
  for (int i = 0; i < rng_state->num_devices; ++i)
  {
    rng_state->gen[i].initf = 0;
    rng_state->gen[i].initial_seed = 0;
    rng_state->gen[i].gen_states = NULL;
    rng_state->gen[i].kernel_params = NULL;
  }
}

/* Destroy generators and free memory */
__host__ void THCRandom_shutdown(THCState* state)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  if (rng_state->gen == NULL) return;
  for (int i = 0; i < rng_state->num_devices; ++i)
  {
    destroyGenerator(state, &rng_state->gen[i]);
  }
  free(rng_state->gen);
  rng_state->gen = NULL;
}

/* Manually set the generator seed */
__host__ static void THCRandom_manualSeedGen(Generator* gen, unsigned long seed)
{
  gen->initial_seed = seed;
  createGeneratorState(gen, seed);
  gen->initf = 1;
}

/* Get the generator for the current device */
__host__ Generator* THCRandom_getGenerator(THCState* state)
{
  THCRNGState* rng_state = THCState_getRngState(state);

  int device;
  THCudaCheck(hipGetDevice(&device));
  if (device >= rng_state->num_devices) THError("Invalid device index.");

  Generator* gen = &rng_state->gen[device];
  if (gen->initf == 0)
  {
    initializeGenerator(state, gen);
    THCRandom_manualSeedGen(gen, (unsigned long)time(0));
  }
  return gen;
}

__host__ struct hiprandStateMtgp32* THCRandom_generatorStates(struct THCState* state)
{
  return THCRandom_getGenerator(state)->gen_states;
}

/* Random seed */
__host__ unsigned long THCRandom_seed(THCState* state)
{
  unsigned long s = (unsigned long)time(0);
  THCRandom_manualSeed(state, s);
  return s;
}

__host__ unsigned long THCRandom_seedAll(THCState* state)
{
  unsigned long s = (unsigned long)time(0);
  THCRandom_manualSeedAll(state, s);
  return s;
}

/* Manually set the seed */
__host__ void THCRandom_manualSeed(THCState* state, unsigned long seed)
{
  Generator* gen = THCRandom_getGenerator(state);
  THCRandom_manualSeedGen(gen, seed);
}

__host__ void THCRandom_manualSeedAll(THCState* state, unsigned long seed)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  int currentDevice;
  THCudaCheck(hipGetDevice(&currentDevice));
  for (int i = 0; i < rng_state->num_devices; ++i) {
    THCudaCheck(hipSetDevice(i));
    THCRandom_manualSeed(state, seed);
  }
  THCudaCheck(hipSetDevice(currentDevice));
}

/* Get the initial seed */
__host__ unsigned long THCRandom_initialSeed(THCState* state)
{
  return THCRandom_getGenerator(state)->initial_seed;
}

__host__ void THCRandom_getRNGState(THCState* state, THByteTensor *rng_state)
{
  Generator* gen = THCRandom_getGenerator(state);

  // The RNG state comprises the MTPG32 states and the seed.
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(unsigned long);
  static const size_t total_size = states_size + seed_size;
  THByteTensor_resize1d(rng_state, total_size);
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");
  THCudaCheck(hipMemcpy(THByteTensor_data(rng_state), gen->gen_states,
                         states_size, hipMemcpyDeviceToHost));
  memcpy(THByteTensor_data(rng_state) + states_size, &gen->initial_seed, seed_size);
}

__global__ void set_rngstate_kernel(hiprandStateMtgp32 *state, mtgp32_kernel_params *kernel)
{
  state[threadIdx.x].k = kernel;
}

__host__ void THCRandom_setRNGState(THCState* state, THByteTensor *rng_state)
{
  Generator* gen = THCRandom_getGenerator(state);

  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(unsigned long);
  static const size_t total_size = states_size + seed_size;
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");

  THCudaCheck(hipMemcpy(gen->gen_states, THByteTensor_data(rng_state),
                         states_size, hipMemcpyHostToDevice));
  set_rngstate_kernel<<<1, MAX_NUM_BLOCKS, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, gen->kernel_params);
  memcpy(&gen->initial_seed, THByteTensor_data(rng_state) + states_size, seed_size);
}

#define GENERATE_KERNEL1(NAME, T, ARG1, CURAND_T, CURAND_FUNC, TRANSFORM)               \
__global__ void NAME(hiprandStateMtgp32 *state, int size, T *result, ARG1)  \
{                                                                              \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                             \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                     \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {      \
    CURAND_T x = CURAND_FUNC(&state[blockIdx.x]);                                 \
    if (i < size) {                                                            \
      T y = TRANSFORM;                                                           \
      result[i] = y;                                                           \
    }                                                                          \
  }                                                                            \
}

#define GENERATE_KERNEL2(NAME, T, ARG1, ARG2, CURAND_T, CURAND_FUNC, TRANSFORM)                \
__global__ void NAME(hiprandStateMtgp32 *state, int size, T *result, ARG1, ARG2)  \
{                                                                                    \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                   \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                           \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {            \
    CURAND_T x = CURAND_FUNC(&state[blockIdx.x]);                                       \
    if (i < size) {                                                                  \
      T y = TRANSFORM;                                                                 \
      result[i] = y;                                                                 \
    }                                                                                \
  }                                                                                  \
}

GENERATE_KERNEL2(generate_uniform, float, double a, double b, float, hiprand_uniform, x * (b-a) + a)
GENERATE_KERNEL2(generate_uniform, double, double a, double b, double, hiprand_uniform_double, x * (b-a) + a)

GENERATE_KERNEL2(generate_normal, float, double mean, double stdv, float, hiprand_normal, (x * stdv) + mean)
GENERATE_KERNEL2(generate_normal, double, double mean, double stdv, double, hiprand_normal_double, (x * stdv) + mean)

GENERATE_KERNEL1(generate_exponential, float, double lambda, float, hiprand_uniform, (float)(-1. / lambda * log(1-x)))
GENERATE_KERNEL1(generate_exponential, double, double lambda, double, hiprand_uniform_double, (double)(-1. / lambda * log(1-x)))

GENERATE_KERNEL2(generate_cauchy, float, double median, double sigma, float, hiprand_uniform, (float)(median + sigma * tan(M_PI*(x-0.5))))
GENERATE_KERNEL2(generate_cauchy, double, double median, double sigma, double, hiprand_uniform_double, (double)(median + sigma * tan(M_PI*(x-0.5))))

#ifdef CUDA_HALF_TENSOR
GENERATE_KERNEL2(generate_uniform, half, double a, double b, float, hiprand_uniform, (ScalarConvert<float, half>::to(x * (b-a) + a)))
GENERATE_KERNEL2(generate_normal, half, double mean, double stdv, float, hiprand_normal, (ScalarConvert<float, half>::to((x * stdv) + mean)))
GENERATE_KERNEL1(generate_exponential, half, double lambda, float, hiprand_uniform, (ScalarConvert<float, half>::to((float)(-1. / lambda * log(1-x)))))
GENERATE_KERNEL2(generate_cauchy, half, double median, double sigma, float, hiprand_uniform, (ScalarConvert<float, half>::to((float)(median + sigma * tan(M_PI*(x-0.5))))))
#endif // CUDA_HALF_TENSOR

#include "generic/THCTensorRandom.cu"
#include "THCGenerateAllTypes.h"

#undef GENERATE_KERNEL1
#undef GENERATE_KERNEL2

