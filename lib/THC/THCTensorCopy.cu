#include "hip/hip_runtime.h"
#include "THGeneral.h"
#include "THCGeneral.h"
#include "THCTensor.h"

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

static void THCudaTensor_computesz(THCudaTensor *self, long **sz_, long **st_)
{
  long *sz, *st, *szh;
  int i;
  
  THCudaCheck(hipMalloc(&sz, sizeof(long)*self->nDimension));
  THCudaCheck(hipMalloc(&st, sizeof(long)*self->nDimension));
  szh = (long*)THAlloc(sizeof(long)*self->nDimension);

  for(i = self->nDimension-1; i >= 0; i--)
  {
    if(i == self->nDimension-1)
      szh[i] = 1;
    else
      szh[i] = szh[i+1]*self->size[i+1];
  }

  THCudaCheck(hipMemcpy(sz, szh, self->nDimension * sizeof(long), hipMemcpyHostToDevice));
  THCudaCheck(hipMemcpy(st, self->stride, self->nDimension * sizeof(long), hipMemcpyHostToDevice));
  THFree(szh);

  *sz_ = sz;
  *st_ = st;
}

__global__ void THCudaTensor_kernel_copy(float *dst, 
                                         long *dst_sz, long *dst_st, int dst_dim,
                                         float *src,
                                         long *src_sz, long *src_st, int src_dim,
                                         long n_elem, long innerdim)
{
  long k = (blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x)*blockDim.y + threadIdx.y;

  long i_start = threadIdx.x * src_st[src_dim-1];
  long i_step = blockDim.x * src_st[src_dim-1];

  long o_start = threadIdx.x * dst_st[dst_dim-1];
  long o_step = blockDim.x * dst_st[dst_dim-1];
  long o_end = innerdim * dst_st[dst_dim-1];

  if ( ((k+1) * innerdim) <= n_elem) // too safe
  {
    long dst_idx = 0;
    long dst_rest = k * innerdim;
    for(int dim = 0; dim < dst_dim; dim++)
    {
      dst_idx += (dst_rest/dst_sz[dim])*dst_st[dim];
      dst_rest = dst_rest % dst_sz[dim];
    }

    long src_idx = 0;
    long src_rest = k * innerdim;
    for(int dim = 0; dim < src_dim; dim++)
    {
      src_idx += (src_rest/src_sz[dim])*src_st[dim];
      src_rest = src_rest % src_sz[dim];
    }

    for (int i=i_start, o=o_start; o<o_end; i+=i_step, o+=o_step) {
      dst[dst_idx + o] = src[src_idx + i];
    }
  }
}

THC_API void THCudaTensor_copy(THCudaTensor *self, THCudaTensor *src)
{
  THArgCheck(THCudaTensor_nElement(self) == THCudaTensor_nElement(src), 2, "sizes do not match"); 

  if(THCudaTensor_isContiguous(self) && THCudaTensor_isContiguous(src))
    THCudaCheck(hipMemcpy(self->storage->data + self->storageOffset, src->storage->data + src->storageOffset, THCudaTensor_nElement(src) * sizeof(float), hipMemcpyDeviceToDevice));
  else
  {    
    long *d_self_sz, *d_self_st, *d_src_sz, *d_src_st;
    long size = THCudaTensor_nElement(self);

    long ndims = self->nDimension;
    long innermostdim = self->size[ndims-1];

    THCudaTensor_computesz(self, &d_self_sz, &d_self_st);
    THCudaTensor_computesz(src, &d_src_sz, &d_src_st);

    dim3 threads(16,16);

    int nblocks = ceil((float)size / (16 * innermostdim ));

    // if nblocks greater than 65535 then we need to open a second dimension
#define __MAX_NUM_BLOCKS_PER_GRID_DIM__ 65535

    /* The configuration below can deal with Tensors 
    * of size up to 65535 * 65535 * 65535 * 16 elements.
    */
    int nblocks_x = (nblocks > __MAX_NUM_BLOCKS_PER_GRID_DIM__) ? __MAX_NUM_BLOCKS_PER_GRID_DIM__ : nblocks;
    int number_blocks_dim_x = DIVUP(nblocks, nblocks_x);
    int nblocks_y = (number_blocks_dim_x > __MAX_NUM_BLOCKS_PER_GRID_DIM__) ? __MAX_NUM_BLOCKS_PER_GRID_DIM__ : number_blocks_dim_x;
    int number_blocks_dim_y = DIVUP(nblocks, nblocks_x * nblocks_y);
    int nblocks_z = number_blocks_dim_y;

    // printf("%d %d %d \n", nblocks_x, nblocks_y, nblocks_z);

    dim3 grid(nblocks_x, nblocks_y, nblocks_z);

    THCudaTensor_kernel_copy<<<grid, threads>>>(THCudaTensor_data(self),
                                                d_self_sz, d_self_st, ndims,
                                                THCudaTensor_data(src),
                                                d_src_sz, d_src_st, src->nDimension,
                                                size, innermostdim);

    hipError_t errcode = hipGetLastError();
    if(errcode != hipSuccess)
      THError(hipGetErrorString(errcode));

    THCudaCheck(hipFree(d_self_sz));
    THCudaCheck(hipFree(d_self_st));
    THCudaCheck(hipFree(d_src_sz));
    THCudaCheck(hipFree(d_src_st));
  }
}
