#include "hip/hip_runtime.h"
#include "THCApply.cuh"
#include "THCHalf.h"
#include "THCNumerics.cuh"

inline int curGPU() {
  int curDev;
  THCudaCheck(hipGetDevice(&curDev));
  return curDev;
}

// Copy operator for the pointwise apply kernel
template <typename TypeDst, typename TypeSrc>
struct CopyOp {
  __device__ __forceinline__ void operator()(TypeDst* dst, TypeSrc* src) {
#if __CUDA_ARCH__ >= 350
    *dst = ScalarConvert<TypeSrc, TypeDst>::to(__ldg(src));
#else
    *dst = ScalarConvert<TypeSrc, TypeDst>::to(*src);
#endif
  }
};

// Copy for the same type to the same type
template <typename TensorTypeDst, typename TensorTypeSrc>
void
THC_copyTensor(THCState* state, TensorTypeDst* dst, TensorTypeSrc* src) {
  long totalElements = TensorUtils<TensorTypeDst>::getNumElements(state, dst);

  THArgCheck(totalElements ==
             TensorUtils<TensorTypeSrc>::getNumElements(state, src),
             2, "sizes do not match");

  if (TensorUtils<TensorTypeDst>::getDims(state, dst) == 0) {
    // Zero-dim tensor; copy nothing
    return;
  }

  // We can memcpy the memory if:
  // -both tensors are contiguous; or,
  // -there is only one element to copy; or,
  // -FIXME: if both tensors have matching size and stride arrays, and no
  // holes within (in other words, there is some permutation that can be applied
  // to the size/strides such that the resulting tensor is
  // contiguous).
  // -AND: both tensors have the same type.
  bool sameType = isSameType<TensorTypeSrc, TensorTypeDst>();
  bool srcContig = TensorUtils<TensorTypeSrc>::isContiguous(state, src);
  bool dstContig = TensorUtils<TensorTypeDst>::isContiguous(state, dst);
  bool memcpyEligible =
    ((srcContig && dstContig) || (totalElements == 1)) && sameType;


  int srcDev = TensorUtils<TensorTypeSrc>::getDevice(state, src);
  int dstDev = TensorUtils<TensorTypeDst>::getDevice(state, dst);
  int oldDev = curGPU();

  // We always perform the copy on the source device, using the
  // current stream on the source device.
  // If the copy is on the default stream, then we fully synchronize
  // both src and dst's default streams for completion of the
  // copy. We have to explicitly do this for non-contig copies.
  // This mimics the behavior of cross-device hipMemcpyAsync on
  // the default stream.
  // If the copy is not on the default stream, then it is up to the
  // user to add needed synchronization on the dst device, since the
  // stream on the dst device that wishes to synchronize may not be
  // the same index as the one on the src device.
  int copyStreamIndex =
    THCState_getCurrentStreamIndex(state);
  hipStream_t copyStream =
    THCState_getDeviceStream(state, srcDev, copyStreamIndex);

  if (srcDev != dstDev && copyStreamIndex == 0) {
    // This is a cross-device copy on the default stream. We perform a
    // two-way barrier between both devices' default streams before
    // the copy. This ensures that any write-after-write and
    // write-after-read dependencies on the destination side are
    // handled, so that no one is operating on the dst memory when
    // we perform the copy.
    // src waits on dst barrier (src already waits on src)
    hipEvent_t dstReady;
    THCudaCheck(hipSetDevice(dstDev));
    THCudaCheck(hipEventCreateWithFlags(&dstReady, hipEventDisableTiming));
    THCudaCheck(hipEventRecord(dstReady, NULL));

    THCudaCheck(hipSetDevice(srcDev));
    THCudaCheck(hipStreamWaitEvent(NULL, dstReady, 0));
    THCudaCheck(hipEventDestroy(dstReady));
  } else if (srcDev != oldDev) {
    THCudaCheck(hipSetDevice(srcDev));
  }

  // We are now on srcDev
  if (memcpyEligible) {
    // Perform the copy
    THCudaCheck(hipMemcpyAsync(
                  TensorUtils<TensorTypeDst>::getData(state, dst),
                  TensorUtils<TensorTypeSrc>::getData(state, src),
                  totalElements *
                  sizeof(typename TensorUtils<TensorTypeDst>::DataType),
                  hipMemcpyDeviceToDevice,
                  copyStream));
  } else {
    // Non-contiguous copy or a type-conversion copy

    // We avoid creating temporary memory copies if possible.
    // If both src and dst are on the same device, or if they are on
    // different devices and p2p access is enabled, perform the copy
    // by a pointwise copy kernel.
    // Otherwise, we'll have to make contiguous (which will in fact
    // invoke copy() again), and then perform the copy.
    // FIXME: might want to consider only running the pointwise kernel
    // if both src and dst innermost dimensions are contiguous. If
    // they are not, then taking the hit of the memory allocation/free
    // might be worth it to avoid non-coalesced reads or writes.

    // A device always has access to itself, so this also handles the
    // case srcDev == dstDev
    if (THCState_getPeerToPeerAccess(state, srcDev, dstDev)) {
      // Make sure we have the current stream set in THCState, since
      // pointwise uses that
      if (srcDev != oldDev) {
        THCState_setStream(state, srcDev, copyStreamIndex);
      }

      bool succ =
        THC_pointwiseApply2(
          state, dst, src,
          CopyOp<typename TensorUtils<TensorTypeDst>::DataType,
                 typename TensorUtils<TensorTypeSrc>::DataType>());

      // Restore prior THCState stream
      if (srcDev != oldDev) {
        THCState_setStream(state, oldDev, copyStreamIndex);
      }

      THArgCheck(succ, 2, CUTORCH_DIM_WARNING);
    } else {
      // GPUs can't access each other directly, but the tensors
      // involved are non-contiguous and/or are different types.

      // Make sure the src is contiguous and in the same type as dst
      THCudaCheck(hipSetDevice(srcDev));
      TensorTypeDst* srcContig = NULL;

      if (sameType) {
        srcContig =
          (TensorTypeDst*) // this is actually the same type as src
          TensorUtils<TensorTypeSrc>::newContiguous(state, src);

      } else {
        // Types are different
        // Copy into the new format, contiguous, on the source device
        srcContig = TensorUtils<TensorTypeDst>::newTensor(state);
        TensorUtils<TensorTypeDst>::resizeAs(state, srcContig, dst);

        if (srcDev != oldDev) {
          THCState_setStream(state, srcDev, copyStreamIndex);
        }

        bool succ =
          THC_pointwiseApply2(
            state, srcContig, src,
            CopyOp<typename TensorUtils<TensorTypeDst>::DataType,
                   typename TensorUtils<TensorTypeSrc>::DataType>());

        // Restore prior THCState stream
        if (srcDev != oldDev) {
          THCState_setStream(state, oldDev, copyStreamIndex);
        }

        THArgCheck(succ, 2, CUTORCH_DIM_WARNING);
      }

      // Make sure the dst is contiguous
      THCudaCheck(hipSetDevice(dstDev));
      TensorTypeDst* dstContig =
        TensorUtils<TensorTypeDst>::newContiguous(state, dst);

      // Now, we are ready for a cross-device memcpy of contiguous
      // data, of the same layout and type
      THCudaCheck(hipSetDevice(srcDev));

      THCudaCheck(hipMemcpyAsync(
                    TensorUtils<TensorTypeDst>::getData(state, dstContig),
                    TensorUtils<TensorTypeDst>::getData(state, srcContig),
                    totalElements *
                    sizeof(typename TensorUtils<TensorTypeDst>::DataType),
                    hipMemcpyDeviceToDevice,
                    copyStream));

      // We are done with the src
      TensorUtils<TensorTypeDst>::free(state, srcContig);

      if (dst != dstContig) {
        TensorUtils<TensorTypeDst>::freeCopyTo(state, dstContig, dst);
      } else {
        TensorUtils<TensorTypeDst>::free(state, dstContig);
      }

      // We're still on srcDev at this point
    }
  }

  if (srcDev != dstDev && copyStreamIndex == 0) {
    // dst waits on src barrier (dst already waits on dst). We cannot
    // operate on dst's copy until the copy is complete.

    // Still on srcDev, record default stream event
    hipEvent_t srcReady;
    THCudaCheck(hipEventCreateWithFlags(&srcReady, hipEventDisableTiming));
    THCudaCheck(hipEventRecord(srcReady, NULL));

    THCudaCheck(hipSetDevice(dstDev));
    THCudaCheck(hipStreamWaitEvent(NULL, srcReady, 0));
    THCudaCheck(hipEventDestroy(srcReady));

    // We are now on dstDev (right above). Restore prior device from dst
    if (dstDev != oldDev) {
      THCudaCheck(hipSetDevice(oldDev));
    }
  } else {
    // We are still on srcDev. Restore prior device from src
    if (srcDev != oldDev) {
      THCudaCheck(hipSetDevice(oldDev));
    }
  }

  THCudaCheck(hipGetLastError());
}

#include "generic/THCTensorCopy.cu"
#include "THCGenerateAllTypes.h"
